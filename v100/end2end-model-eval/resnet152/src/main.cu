#include "../inc/common.h"
#include "../inc/nvml_helper.hpp"
int main(int argc,char *argv[]){
    float *input = (float *)malloc(224*224*3*sizeof(float));
    string imagePath = argv[1];
    float *dInput;
    hipMalloc(&dInput,224*224*3*sizeof(float));
    Conv conv1_conv;
    conv1_conv.initialize(1,3,224,224,64,3,7,7,2,"../../../weights/resnet152/weights/conv1_conv.bin");
    BatchNorm conv1_bn;
    conv1_bn.initialize(1,64,112,112,"../../../weights/resnet152/weights/conv1_bn.bin");
    Activation conv1_relu;
    conv1_relu.initialize(1,64,112,112);
    Pool conv1_max_pool;
    conv1_max_pool.initialize(1,64,112,112,1,3,3,HIPDNN_POOLING_MAX,2);

    ConvBlk1 conv2_block1(1,64,56,56,"../../../weights/resnet152/weights/conv2_block1_0_conv.bin","../../../weights/resnet152/weights/conv2_block1_0_bn.bin",
                          "../../../weights/resnet152/weights/conv2_block1_1_conv.bin","../../../weights/resnet152/weights/conv2_block1_1_bn.bin",
                          "../../../weights/resnet152/weights/conv2_block1_2_conv.bin","../../../weights/resnet152/weights/conv2_block1_2_bn.bin",
                          "../../../weights/resnet152/weights/conv2_block1_3_conv.bin","../../../weights/resnet152/weights/conv2_block1_3_bn.bin");
    ConvBlk2 conv2_block2(1,256,56,56,"../../../weights/resnet152/weights/conv2_block2_1_conv.bin","../../../weights/resnet152/weights/conv2_block2_1_bn.bin",
                          "../../../weights/resnet152/weights/conv2_block2_2_conv.bin","../../../weights/resnet152/weights/conv2_block2_2_bn.bin",
                          "../../../weights/resnet152/weights/conv2_block2_3_conv.bin","../../../weights/resnet152/weights/conv2_block2_3_bn.bin");
    ConvBlk2 conv2_block3(1,256,56,56,"../../../weights/resnet152/weights/conv2_block3_1_conv.bin","../../../weights/resnet152/weights/conv2_block3_1_bn.bin",
                          "../../../weights/resnet152/weights/conv2_block3_2_conv.bin","../../../weights/resnet152/weights/conv2_block3_2_bn.bin",
                          "../../../weights/resnet152/weights/conv2_block3_3_conv.bin","../../../weights/resnet152/weights/conv2_block3_3_bn.bin");
    ConvBlk3 conv3_block1(1,256,56,56,"../../../weights/resnet152/weights/conv3_block1_0_conv.bin","../../../weights/resnet152/weights/conv3_block1_0_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block1_1_conv.bin","../../../weights/resnet152/weights/conv3_block1_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block1_2_conv.bin","../../../weights/resnet152/weights/conv3_block1_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block1_3_conv.bin","../../../weights/resnet152/weights/conv3_block1_3_bn.bin");
    ConvBlk4 conv3_block2(1,512,28,28,"../../../weights/resnet152/weights/conv3_block2_1_conv.bin","../../../weights/resnet152/weights/conv3_block2_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block2_2_conv.bin","../../../weights/resnet152/weights/conv3_block2_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block2_3_conv.bin","../../../weights/resnet152/weights/conv3_block2_3_bn.bin");
    ConvBlk4 conv3_block3(1,512,28,28,"../../../weights/resnet152/weights/conv3_block3_1_conv.bin","../../../weights/resnet152/weights/conv3_block3_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block3_2_conv.bin","../../../weights/resnet152/weights/conv3_block3_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block3_3_conv.bin","../../../weights/resnet152/weights/conv3_block3_3_bn.bin");
    ConvBlk4 conv3_block4(1,512,28,28,"../../../weights/resnet152/weights/conv3_block4_1_conv.bin","../../../weights/resnet152/weights/conv3_block4_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block4_2_conv.bin","../../../weights/resnet152/weights/conv3_block4_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block4_3_conv.bin","../../../weights/resnet152/weights/conv3_block4_3_bn.bin");
    ConvBlk4 conv3_block5(1,512,28,28,"../../../weights/resnet152/weights/conv3_block5_1_conv.bin","../../../weights/resnet152/weights/conv3_block5_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block5_2_conv.bin","../../../weights/resnet152/weights/conv3_block5_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block5_3_conv.bin","../../../weights/resnet152/weights/conv3_block5_3_bn.bin");
    ConvBlk4 conv3_block6(1,512,28,28,"../../../weights/resnet152/weights/conv3_block6_1_conv.bin","../../../weights/resnet152/weights/conv3_block6_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block6_2_conv.bin","../../../weights/resnet152/weights/conv3_block6_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block6_3_conv.bin","../../../weights/resnet152/weights/conv3_block6_3_bn.bin");
    ConvBlk4 conv3_block7(1,512,28,28,"../../../weights/resnet152/weights/conv3_block7_1_conv.bin","../../../weights/resnet152/weights/conv3_block7_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block7_2_conv.bin","../../../weights/resnet152/weights/conv3_block7_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block7_3_conv.bin","../../../weights/resnet152/weights/conv3_block7_3_bn.bin");
    ConvBlk4 conv3_block8(1,512,28,28,"../../../weights/resnet152/weights/conv3_block8_1_conv.bin","../../../weights/resnet152/weights/conv3_block8_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block8_2_conv.bin","../../../weights/resnet152/weights/conv3_block8_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block8_3_conv.bin","../../../weights/resnet152/weights/conv3_block8_3_bn.bin");

    ConvBlk5 conv4_block1(1,512,28,28,"../../../weights/resnet152/weights/conv4_block1_0_conv.bin","../../../weights/resnet152/weights/conv4_block1_0_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block1_1_conv.bin","../../../weights/resnet152/weights/conv4_block1_1_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block1_2_conv.bin","../../../weights/resnet152/weights/conv4_block1_2_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block1_3_conv.bin","../../../weights/resnet152/weights/conv4_block1_3_bn.bin");
    ConvBlk6 conv4_block2(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block2_1_conv.bin","../../../weights/resnet152/weights/conv4_block2_1_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block2_2_conv.bin","../../../weights/resnet152/weights/conv4_block2_2_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block2_3_conv.bin","../../../weights/resnet152/weights/conv4_block2_3_bn.bin");
    ConvBlk6 conv4_block3(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block3_1_conv.bin","../../../weights/resnet152/weights/conv4_block3_1_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block3_2_conv.bin","../../../weights/resnet152/weights/conv4_block3_2_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block3_3_conv.bin","../../../weights/resnet152/weights/conv4_block3_3_bn.bin");
    ConvBlk6 conv4_block4(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block4_1_conv.bin","../../../weights/resnet152/weights/conv4_block4_1_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block4_2_conv.bin","../../../weights/resnet152/weights/conv4_block4_2_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block4_3_conv.bin","../../../weights/resnet152/weights/conv4_block4_3_bn.bin");
    ConvBlk6 conv4_block5(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block5_1_conv.bin","../../../weights/resnet152/weights/conv4_block5_1_bn.bin","../../../weights/resnet152/weights/conv4_block5_2_conv.bin","../../../weights/resnet152/weights/conv4_block5_2_bn.bin","../../../weights/resnet152/weights/conv4_block5_3_conv.bin","../../../weights/resnet152/weights/conv4_block5_3_bn.bin");
    ConvBlk6 conv4_block6(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block6_1_conv.bin","../../../weights/resnet152/weights/conv4_block6_1_bn.bin","../../../weights/resnet152/weights/conv4_block6_2_conv.bin","../../../weights/resnet152/weights/conv4_block6_2_bn.bin","../../../weights/resnet152/weights/conv4_block6_3_conv.bin","../../../weights/resnet152/weights/conv4_block6_3_bn.bin");
    ConvBlk6 conv4_block7(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block7_1_conv.bin","../../../weights/resnet152/weights/conv4_block7_1_bn.bin","../../../weights/resnet152/weights/conv4_block7_2_conv.bin","../../../weights/resnet152/weights/conv4_block7_2_bn.bin","../../../weights/resnet152/weights/conv4_block7_3_conv.bin","../../../weights/resnet152/weights/conv4_block7_3_bn.bin");
    ConvBlk6 conv4_block8(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block8_1_conv.bin","../../../weights/resnet152/weights/conv4_block8_1_bn.bin","../../../weights/resnet152/weights/conv4_block8_2_conv.bin","../../../weights/resnet152/weights/conv4_block8_2_bn.bin","../../../weights/resnet152/weights/conv4_block8_3_conv.bin","../../../weights/resnet152/weights/conv4_block8_3_bn.bin");
    ConvBlk6 conv4_block9(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block9_1_conv.bin","../../../weights/resnet152/weights/conv4_block9_1_bn.bin","../../../weights/resnet152/weights/conv4_block9_2_conv.bin","../../../weights/resnet152/weights/conv4_block9_2_bn.bin","../../../weights/resnet152/weights/conv4_block9_3_conv.bin","../../../weights/resnet152/weights/conv4_block9_3_bn.bin");
    ConvBlk6 conv4_block10(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block10_1_conv.bin","../../../weights/resnet152/weights/conv4_block10_1_bn.bin","../../../weights/resnet152/weights/conv4_block10_2_conv.bin","../../../weights/resnet152/weights/conv4_block10_2_bn.bin","../../../weights/resnet152/weights/conv4_block10_3_conv.bin","../../../weights/resnet152/weights/conv4_block10_3_bn.bin");
    ConvBlk6 conv4_block11(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block11_1_conv.bin","../../../weights/resnet152/weights/conv4_block11_1_bn.bin","../../../weights/resnet152/weights/conv4_block11_2_conv.bin","../../../weights/resnet152/weights/conv4_block11_2_bn.bin","../../../weights/resnet152/weights/conv4_block11_3_conv.bin","../../../weights/resnet152/weights/conv4_block11_3_bn.bin");
    ConvBlk6 conv4_block12(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block12_1_conv.bin","../../../weights/resnet152/weights/conv4_block12_1_bn.bin","../../../weights/resnet152/weights/conv4_block12_2_conv.bin","../../../weights/resnet152/weights/conv4_block12_2_bn.bin","../../../weights/resnet152/weights/conv4_block12_3_conv.bin","../../../weights/resnet152/weights/conv4_block12_3_bn.bin");
    ConvBlk6 conv4_block13(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block13_1_conv.bin","../../../weights/resnet152/weights/conv4_block13_1_bn.bin","../../../weights/resnet152/weights/conv4_block13_2_conv.bin","../../../weights/resnet152/weights/conv4_block13_2_bn.bin","../../../weights/resnet152/weights/conv4_block13_3_conv.bin","../../../weights/resnet152/weights/conv4_block13_3_bn.bin");
    ConvBlk6 conv4_block14(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block14_1_conv.bin","../../../weights/resnet152/weights/conv4_block14_1_bn.bin","../../../weights/resnet152/weights/conv4_block14_2_conv.bin","../../../weights/resnet152/weights/conv4_block14_2_bn.bin","../../../weights/resnet152/weights/conv4_block14_3_conv.bin","../../../weights/resnet152/weights/conv4_block14_3_bn.bin");
    ConvBlk6 conv4_block15(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block15_1_conv.bin","../../../weights/resnet152/weights/conv4_block15_1_bn.bin","../../../weights/resnet152/weights/conv4_block15_2_conv.bin","../../../weights/resnet152/weights/conv4_block15_2_bn.bin","../../../weights/resnet152/weights/conv4_block15_3_conv.bin","../../../weights/resnet152/weights/conv4_block15_3_bn.bin");
    ConvBlk6 conv4_block16(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block16_1_conv.bin","../../../weights/resnet152/weights/conv4_block16_1_bn.bin","../../../weights/resnet152/weights/conv4_block16_2_conv.bin","../../../weights/resnet152/weights/conv4_block16_2_bn.bin","../../../weights/resnet152/weights/conv4_block16_3_conv.bin","../../../weights/resnet152/weights/conv4_block16_3_bn.bin");
    ConvBlk6 conv4_block17(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block17_1_conv.bin","../../../weights/resnet152/weights/conv4_block17_1_bn.bin","../../../weights/resnet152/weights/conv4_block17_2_conv.bin","../../../weights/resnet152/weights/conv4_block17_2_bn.bin","../../../weights/resnet152/weights/conv4_block17_3_conv.bin","../../../weights/resnet152/weights/conv4_block17_3_bn.bin");
    ConvBlk6 conv4_block18(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block18_1_conv.bin","../../../weights/resnet152/weights/conv4_block18_1_bn.bin","../../../weights/resnet152/weights/conv4_block18_2_conv.bin","../../../weights/resnet152/weights/conv4_block18_2_bn.bin","../../../weights/resnet152/weights/conv4_block18_3_conv.bin","../../../weights/resnet152/weights/conv4_block18_3_bn.bin");
    ConvBlk6 conv4_block19(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block19_1_conv.bin","../../../weights/resnet152/weights/conv4_block19_1_bn.bin","../../../weights/resnet152/weights/conv4_block19_2_conv.bin","../../../weights/resnet152/weights/conv4_block19_2_bn.bin","../../../weights/resnet152/weights/conv4_block19_3_conv.bin","../../../weights/resnet152/weights/conv4_block19_3_bn.bin");
    ConvBlk6 conv4_block20(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block20_1_conv.bin","../../../weights/resnet152/weights/conv4_block20_1_bn.bin","../../../weights/resnet152/weights/conv4_block20_2_conv.bin","../../../weights/resnet152/weights/conv4_block20_2_bn.bin","../../../weights/resnet152/weights/conv4_block20_3_conv.bin","../../../weights/resnet152/weights/conv4_block20_3_bn.bin");
    ConvBlk6 conv4_block21(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block21_1_conv.bin","../../../weights/resnet152/weights/conv4_block21_1_bn.bin","../../../weights/resnet152/weights/conv4_block21_2_conv.bin","../../../weights/resnet152/weights/conv4_block21_2_bn.bin","../../../weights/resnet152/weights/conv4_block21_3_conv.bin","../../../weights/resnet152/weights/conv4_block21_3_bn.bin");
    ConvBlk6 conv4_block22(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block22_1_conv.bin","../../../weights/resnet152/weights/conv4_block22_1_bn.bin","../../../weights/resnet152/weights/conv4_block22_2_conv.bin","../../../weights/resnet152/weights/conv4_block22_2_bn.bin","../../../weights/resnet152/weights/conv4_block22_3_conv.bin","../../../weights/resnet152/weights/conv4_block22_3_bn.bin");
    ConvBlk6 conv4_block23(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block23_1_conv.bin","../../../weights/resnet152/weights/conv4_block23_1_bn.bin","../../../weights/resnet152/weights/conv4_block23_2_conv.bin","../../../weights/resnet152/weights/conv4_block23_2_bn.bin","../../../weights/resnet152/weights/conv4_block23_3_conv.bin","../../../weights/resnet152/weights/conv4_block23_3_bn.bin");
    ConvBlk6 conv4_block24(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block24_1_conv.bin","../../../weights/resnet152/weights/conv4_block24_1_bn.bin","../../../weights/resnet152/weights/conv4_block24_2_conv.bin","../../../weights/resnet152/weights/conv4_block24_2_bn.bin","../../../weights/resnet152/weights/conv4_block24_3_conv.bin","../../../weights/resnet152/weights/conv4_block24_3_bn.bin");
    ConvBlk6 conv4_block25(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block25_1_conv.bin","../../../weights/resnet152/weights/conv4_block25_1_bn.bin","../../../weights/resnet152/weights/conv4_block25_2_conv.bin","../../../weights/resnet152/weights/conv4_block25_2_bn.bin","../../../weights/resnet152/weights/conv4_block25_3_conv.bin","../../../weights/resnet152/weights/conv4_block25_3_bn.bin");
    ConvBlk6 conv4_block26(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block26_1_conv.bin","../../../weights/resnet152/weights/conv4_block26_1_bn.bin","../../../weights/resnet152/weights/conv4_block26_2_conv.bin","../../../weights/resnet152/weights/conv4_block26_2_bn.bin","../../../weights/resnet152/weights/conv4_block26_3_conv.bin","../../../weights/resnet152/weights/conv4_block26_3_bn.bin");
    ConvBlk6 conv4_block27(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block27_1_conv.bin","../../../weights/resnet152/weights/conv4_block27_1_bn.bin","../../../weights/resnet152/weights/conv4_block27_2_conv.bin","../../../weights/resnet152/weights/conv4_block27_2_bn.bin","../../../weights/resnet152/weights/conv4_block27_3_conv.bin","../../../weights/resnet152/weights/conv4_block27_3_bn.bin");
    ConvBlk6 conv4_block28(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block28_1_conv.bin","../../../weights/resnet152/weights/conv4_block28_1_bn.bin","../../../weights/resnet152/weights/conv4_block28_2_conv.bin","../../../weights/resnet152/weights/conv4_block28_2_bn.bin","../../../weights/resnet152/weights/conv4_block28_3_conv.bin","../../../weights/resnet152/weights/conv4_block28_3_bn.bin");
    ConvBlk6 conv4_block29(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block29_1_conv.bin","../../../weights/resnet152/weights/conv4_block29_1_bn.bin","../../../weights/resnet152/weights/conv4_block29_2_conv.bin","../../../weights/resnet152/weights/conv4_block29_2_bn.bin","../../../weights/resnet152/weights/conv4_block29_3_conv.bin","../../../weights/resnet152/weights/conv4_block29_3_bn.bin");
    ConvBlk6 conv4_block30(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block30_1_conv.bin","../../../weights/resnet152/weights/conv4_block30_1_bn.bin","../../../weights/resnet152/weights/conv4_block30_2_conv.bin","../../../weights/resnet152/weights/conv4_block30_2_bn.bin","../../../weights/resnet152/weights/conv4_block30_3_conv.bin","../../../weights/resnet152/weights/conv4_block30_3_bn.bin");
    ConvBlk6 conv4_block31(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block31_1_conv.bin","../../../weights/resnet152/weights/conv4_block31_1_bn.bin","../../../weights/resnet152/weights/conv4_block31_2_conv.bin","../../../weights/resnet152/weights/conv4_block31_2_bn.bin","../../../weights/resnet152/weights/conv4_block31_3_conv.bin","../../../weights/resnet152/weights/conv4_block31_3_bn.bin");
    ConvBlk6 conv4_block32(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block32_1_conv.bin","../../../weights/resnet152/weights/conv4_block32_1_bn.bin","../../../weights/resnet152/weights/conv4_block32_2_conv.bin","../../../weights/resnet152/weights/conv4_block32_2_bn.bin","../../../weights/resnet152/weights/conv4_block32_3_conv.bin","../../../weights/resnet152/weights/conv4_block32_3_bn.bin");
    ConvBlk6 conv4_block33(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block33_1_conv.bin","../../../weights/resnet152/weights/conv4_block33_1_bn.bin","../../../weights/resnet152/weights/conv4_block33_2_conv.bin","../../../weights/resnet152/weights/conv4_block33_2_bn.bin","../../../weights/resnet152/weights/conv4_block33_3_conv.bin","../../../weights/resnet152/weights/conv4_block33_3_bn.bin");
    ConvBlk6 conv4_block34(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block34_1_conv.bin","../../../weights/resnet152/weights/conv4_block34_1_bn.bin","../../../weights/resnet152/weights/conv4_block34_2_conv.bin","../../../weights/resnet152/weights/conv4_block34_2_bn.bin","../../../weights/resnet152/weights/conv4_block34_3_conv.bin","../../../weights/resnet152/weights/conv4_block34_3_bn.bin");
    ConvBlk6 conv4_block35(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block35_1_conv.bin","../../../weights/resnet152/weights/conv4_block35_1_bn.bin","../../../weights/resnet152/weights/conv4_block35_2_conv.bin","../../../weights/resnet152/weights/conv4_block35_2_bn.bin","../../../weights/resnet152/weights/conv4_block35_3_conv.bin","../../../weights/resnet152/weights/conv4_block35_3_bn.bin");
    ConvBlk6 conv4_block36(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block36_1_conv.bin","../../../weights/resnet152/weights/conv4_block36_1_bn.bin","../../../weights/resnet152/weights/conv4_block36_2_conv.bin","../../../weights/resnet152/weights/conv4_block36_2_bn.bin","../../../weights/resnet152/weights/conv4_block36_3_conv.bin","../../../weights/resnet152/weights/conv4_block36_3_bn.bin");
    ConvBlk7 conv5_block1(1,1024,14,14,"../../../weights/resnet152/weights/conv5_block1_0_conv.bin","../../../weights/resnet152/weights/conv5_block1_0_bn.bin",
                          "../../../weights/resnet152/weights/conv5_block1_1_conv.bin","../../../weights/resnet152/weights/conv5_block1_1_bn.bin",
                          "../../../weights/resnet152/weights/conv5_block1_2_conv.bin","../../../weights/resnet152/weights/conv5_block1_2_bn.bin",
                          "../../../weights/resnet152/weights/conv5_block1_3_conv.bin","../../../weights/resnet152/weights/conv5_block1_3_bn.bin");
    ConvBlk8 conv5_block2(1,2048,7,7,"../../../weights/resnet152/weights/conv5_block2_1_conv.bin","../../../weights/resnet152/weights/conv5_block2_1_bn.bin",
                          "../../../weights/resnet152/weights/conv5_block2_2_conv.bin","../../../weights/resnet152/weights/conv5_block2_2_bn.bin",
                          "../../../weights/resnet152/weights/conv5_block2_3_conv.bin","../../../weights/resnet152/weights/conv5_block2_3_bn.bin");
    ConvBlk8 conv5_block3(1,2048,7,7,"../../../weights/resnet152/weights/conv5_block3_1_conv.bin","../../../weights/resnet152/weights/conv5_block3_1_bn.bin",
                          "../../../weights/resnet152/weights/conv5_block3_2_conv.bin","../../../weights/resnet152/weights/conv5_block3_2_bn.bin",
                          "../../../weights/resnet152/weights/conv5_block3_3_conv.bin","../../../weights/resnet152/weights/conv5_block3_3_bn.bin");
    Pool avg_pool;
    avg_pool.initialize(1,2048,7,7,0,7,7,HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING,1);
    Conv predict;
    predict.initialize(1,2048,1,1,1000,0,1,1,1,"../../../weights/resnet152/weights/predictions.bin");

    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    hipMemcpy(dInput, input, 224 * 224 * 3 * sizeof(float), hipMemcpyHostToDevice);
    float *output;
    output = conv1_conv.forward(dInput);
    output = conv1_bn.forward(output);
    output = conv1_relu.forward(output);
    output = conv1_max_pool.forward(output);
    output = conv2_block1.forward(output);
    output = conv2_block2.forward(output);
    output = conv2_block3.forward(output);
    output = conv3_block1.forward(output);
    output = conv3_block2.forward(output);
    output = conv3_block3.forward(output);
    output = conv3_block4.forward(output);
    output = conv3_block5.forward(output);
    output = conv3_block6.forward(output);
    output = conv3_block7.forward(output);
    output = conv3_block8.forward(output);
    output = conv4_block1.forward(output);
    output = conv4_block2.forward(output);
    output = conv4_block3.forward(output);
    output = conv4_block4.forward(output);
    output = conv4_block5.forward(output);
    output = conv4_block6.forward(output);
    output = conv4_block7.forward(output);
    output = conv4_block8.forward(output);
    output = conv4_block9.forward(output);
    output = conv4_block10.forward(output);
    output = conv4_block11.forward(output);
    output = conv4_block12.forward(output);
    output = conv4_block13.forward(output);
    output = conv4_block14.forward(output);
    output = conv4_block15.forward(output);
    output = conv4_block16.forward(output);
    output = conv4_block17.forward(output);
    output = conv4_block18.forward(output);
    output = conv4_block19.forward(output);
    output = conv4_block20.forward(output);
    output = conv4_block21.forward(output);
    output = conv4_block22.forward(output);
    output = conv4_block23.forward(output);
    output = conv4_block24.forward(output);
    output = conv4_block25.forward(output);
    output = conv4_block26.forward(output);
    output = conv4_block27.forward(output);
    output = conv4_block28.forward(output);
    output = conv4_block29.forward(output);
    output = conv4_block30.forward(output);
    output = conv4_block31.forward(output);
    output = conv4_block32.forward(output);
    output = conv4_block33.forward(output);
    output = conv4_block34.forward(output);
    output = conv4_block35.forward(output);
    output = conv4_block36.forward(output);
    output = conv5_block1.forward(output);
    output = conv5_block2.forward(output);
    output = conv5_block3.forward(output);
    output = avg_pool.forward(output);
    NVML_INIT;
    NVML_DEV_t dev0;
    NVML_GET_HANDLE(0, &dev0);
    unsigned long long start,end;
    unsigned long long energy = 0;
    float inference_time = 0.0f;
    for(int i=0;i<100;++i){
        string image = imagePath + "/" + to_string(i) + ".bin";
        load_input(image, 3 * 224 * 224, input);
        hipMemcpy(dInput,input,224*224*3*sizeof(float),hipMemcpyHostToDevice);
        hipEventRecord(event_start);
        output = conv1_conv.forward(dInput);
        output = conv1_bn.forward(output);
        output = conv1_relu.forward(output);
        output = conv1_max_pool.forward(output);
        output = conv2_block1.forward(output);
        output = conv2_block2.forward(output);
        output = conv2_block3.forward(output);
        output = conv3_block1.forward(output);
        output = conv3_block2.forward(output);
        output = conv3_block3.forward(output);
        output = conv3_block4.forward(output);
        output = conv3_block5.forward(output);
        output = conv3_block6.forward(output);
        output = conv3_block7.forward(output);
        output = conv3_block8.forward(output);
        output = conv4_block1.forward(output);
        output = conv4_block2.forward(output);
        output = conv4_block3.forward(output);
        output = conv4_block4.forward(output);
        output = conv4_block5.forward(output);
        output = conv4_block6.forward(output);
        output = conv4_block7.forward(output);
        output = conv4_block8.forward(output);
        output = conv4_block9.forward(output);
        output = conv4_block10.forward(output);
        output = conv4_block11.forward(output);
        output = conv4_block12.forward(output);
        output = conv4_block13.forward(output);
        output = conv4_block14.forward(output);
        output = conv4_block15.forward(output);
        output = conv4_block16.forward(output);
        output = conv4_block17.forward(output);
        output = conv4_block18.forward(output);
        output = conv4_block19.forward(output);
        output = conv4_block20.forward(output);
        output = conv4_block21.forward(output);
        output = conv4_block22.forward(output);
        output = conv4_block23.forward(output);
        output = conv4_block24.forward(output);
        output = conv4_block25.forward(output);
        output = conv4_block26.forward(output);
        output = conv4_block27.forward(output);
        output = conv4_block28.forward(output);
        output = conv4_block29.forward(output);
        output = conv4_block30.forward(output);
        output = conv4_block31.forward(output);
        output = conv4_block32.forward(output);
        output = conv4_block33.forward(output);
        output = conv4_block34.forward(output);
        output = conv4_block35.forward(output);
        output = conv4_block36.forward(output);
        output = conv5_block1.forward(output);
        output = conv5_block2.forward(output);
        output = conv5_block3.forward(output);
        output = avg_pool.forward(output);
        output = predict.forward(output);
        hipDeviceSynchronize();
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<"resnet152 cuDNN costs "<<inference_time/100<<" ms"<<endl;
    cout<<endl;
    unsigned int outputSize = 1000;
    float *hOutput = (float *)malloc(outputSize*sizeof(float));
    hipMemcpy(hOutput,output,outputSize*sizeof(float),hipMemcpyDeviceToHost);
    vector<float> out_bin;
    for(int i=0;i<outputSize;++i){
        out_bin.push_back(hOutput[i]);
    }
    std::ofstream ofp("d.bin", std::ios::out | std::ios::binary);
    ofp.write(reinterpret_cast<const char*>(out_bin.data()), out_bin.size() * sizeof(float));
    return 0;
}
