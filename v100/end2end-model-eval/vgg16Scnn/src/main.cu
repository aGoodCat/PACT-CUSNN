
#include "../inc/conv_and_pool.h"
#include "../inc/nvml_helper.hpp"
int main(int argc, char *argv[]){
    float *input = (float *)malloc(224*224*3*sizeof(float));
    string imagePath = argv[1];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    float *dInput;
    hipMalloc(&dInput,224*224*3*sizeof(float));

    Conv block1_conv1;
    block1_conv1.initialize(1,3,224,224,64,1,3,3,1,"../../../weights/vgg16/weights/block1_conv1.bin");
    Relu relu1;
    relu1.initialize(1,64,224,224,0.6);

    Conv_1_64_224_224_64 block1_conv2;
    block1_conv2.initialize(1,64,224,224,64,1,3,3,1,"../../../weights/vgg16/weights/block1_conv2.bin");
    Conv block1_conv2_cudnn;
    block1_conv2_cudnn.initialize(1,64,224,224,64,1,3,3,1,"../../../weights/vgg16/weights/block1_conv2.bin");
    Activation relu2;
    relu2.initialize(1,64,224,224);

    Pool block1_pool;
    block1_pool.initialize(1,64,224,224,1,3,3,HIPDNN_POOLING_MAX,2);
    Conv block2_conv1;
    block2_conv1.initialize(1,64,112,112,128,1,3,3,1,"../../../weights/vgg16/weights/block2_conv1.bin");
    Relu relu3;
    relu3.initialize(1,128,112,112,0.75);

    Conv_1_128_112_112_128 block2_conv2;
    block2_conv2.initialize(1,128,112,112,128,1,3,3,1,"../../../weights/vgg16/weights/block2_conv2.bin");
    Conv block2_conv2_cudnn;
    block2_conv2_cudnn.initialize(1,128,112,112,128,1,3,3,1,"../../../weights/vgg16/weights/block2_conv2.bin");
    Activation relu4;
    relu4.initialize(1,128,112,112);

    Pool block2_pool;
    block2_pool.initialize(1,128,112,112,1,3,3,HIPDNN_POOLING_MAX,2);
    Conv block3_conv1;
    block3_conv1.initialize(1,128,56,56,256,1,3,3,1,"../../../weights/vgg16/weights/block3_conv1.bin");
    Relu relu5;
    relu5.initialize(1,256,56,56,0.6);

    Conv_1_256_56_56_256 block3_conv2;
    block3_conv2.initialize(1,256,56,56,256,1,3,3,1,"../../../weights/vgg16/weights/block3_conv2.bin");
    Conv block3_conv2_cudnn;
    block3_conv2_cudnn.initialize(1,256,56,56,256,1,3,3,1,"../../../weights/vgg16/weights/block3_conv2.bin");
    Relu relu6;
    relu6.initialize(1,256,56,56,0.6);

    Conv_1_256_56_56_256 block3_conv3;
    block3_conv3.initialize(1,256,56,56,256,1,3,3,1,"../../../weights/vgg16/weights/block3_conv3.bin");
    Conv block3_conv3_cudnn;
    block3_conv3_cudnn.initialize(1,256,56,56,256,1,3,3,1,"../../../weights/vgg16/weights/block3_conv3.bin");

    Activation relu6_cudnn;
    relu6_cudnn.initialize(1,256,56,56);

    Pool block3_pool;
    block3_pool.initialize(1,256,56,56,1,3,3,HIPDNN_POOLING_MAX,2);
    Conv block4_conv1;
    block4_conv1.initialize(1,256,28,28,512,1,3,3,1,"../../../weights/vgg16/weights/block4_conv1.bin");
    Relu relu7;
    relu7.initialize(1,512,28,28,0.4);

    Conv_1_512_28_28_512 block4_conv2;
    block4_conv2.initialize(1,512,28,28,512,1,3,3,1,"../../../weights/vgg16/weights/block4_conv2.bin");
    Conv block4_conv2_cudnn;
    block4_conv2_cudnn.initialize(1,512,28,28,512,1,3,3,1,"../../../weights/vgg16/weights/block4_conv2.bin");
    Relu relu8;
    relu8.initialize(1,512,28,28,0.4);

    Conv_1_512_28_28_512 block4_conv3;
    block4_conv3.initialize(1,512,28,28,512,1,3,3,1,"../../../weights/vgg16/weights/block4_conv3.bin");
    Conv block4_conv3_cudnn;
    block4_conv3_cudnn.initialize(1,512,28,28,512,1,3,3,1,"../../../weights/vgg16/weights/block4_conv3.bin");
    Activation relu9;
    relu9.initialize(1,512,28,28);

    Pool block4_pool;
    block4_pool.initialize(1,512,28,28,1,3,3,HIPDNN_POOLING_MAX,2);
    Conv block5_conv1;
    block5_conv1.initialize(1,512,14,14,512,1,3,3,1,"../../../weights/vgg16/weights/block5_conv1.bin");
    Activation relu10;
    relu10.initialize(1,512,14,14);

    Conv_1_512_14_14_512 block5_conv2;
    Conv block5_conv2_cudnn;
    block5_conv2.initialize(1,512,14,14,512,1,3,3,1,"../../../weights/vgg16/weights/block5_conv2.bin");
    block5_conv2_cudnn.initialize(1,512,14,14,512,1,3,3,1,"../../../weights/vgg16/weights/block5_conv2.bin");
    Activation relu11;
    relu11.initialize(1,512,14,14);

    Conv_1_512_14_14_512 block5_conv3;
    block5_conv3.initialize(1,512,14,14,512,1,3,3,1,"../../../weights/vgg16/weights/block5_conv3.bin");
    Conv block5_conv3_cudnn;
    block5_conv3_cudnn.initialize(1,512,14,14,512,1,3,3,1,"../../../weights/vgg16/weights/block5_conv3.bin");
    Activation relu12;
    relu12.initialize(1,512,14,14);


    Pool block5_pool;
    block5_pool.initialize(1,512,14,14,1,3,3,HIPDNN_POOLING_MAX,2);
    FC fc1;
    fc1.initialize(1,25088,1,1,4096,0,1,1,1,"../../../weights/vgg16/weights/fc1.bin");
    FC fc2;
    fc2.initialize(1,4096,1,1,4096,0,1,1,1,"../../../weights/vgg16/weights/fc2.bin");
    FC fc3;
    fc3.initialize(1,4096,1,1,1000,0,1,1,1,"../../../weights/vgg16/weights/predictions.bin");
    hipMemcpy(dInput,input,224*224*3*sizeof(float),hipMemcpyHostToDevice);
    float *output;
    output = block1_conv1.forward(dInput);
    output = relu1.forward(output);
    output = block1_conv2.forward(output);
    output = relu2.forward(output);
    output = block1_pool.forward(output);
    output = block2_conv1.forward(output);
    output = relu3.forward(output);
    if(relu3.sparse){
        output = block2_conv2.forward(output);
    }else{
        output = block2_conv2_cudnn.forward(output);
    }
    output = relu4.forward(output);
    output = block2_pool.forward(output);
    output = block3_conv1.forward(output);
    output = relu5.forward(output);
    if(relu5.sparse){
        output = block3_conv2.forward(output);
    }else{
        output = block3_conv2_cudnn.forward(output);
    }
    output = relu6.forward(output);
    if(relu6.sparse){
        output = block3_conv3.forward(output);
    }else{
        output = block3_conv3_cudnn.forward(output);
    }
    output = relu6_cudnn.forward(output);
    output = block3_pool.forward(output);
    output = block4_conv1.forward(output);
    output = relu7.forward(output);
    if(relu7.sparse){
        output = block4_conv2.forward(output);
    }else{
        output = block4_conv2_cudnn.forward(output);
    }
    output = relu8.forward(output);
    if(relu8.sparse){
        output = block4_conv3.forward(output);
    }else{
        output = block4_conv3_cudnn.forward(output);
    }
    output = relu9.forward(output);
    output = block4_pool.forward(output);
    output = block5_conv1.forward(output);
    output = relu10.forward(output);
    output = block5_conv2.forward(output);
    output = relu11.forward(output);
    output = block5_conv3.forward(output);
    output = relu12.forward(output);
    output = block5_pool.forward(output);
    output = fc1.forward(output);
    output = fc2.forward(output);
    output = fc3.forward(output);
    hipDeviceSynchronize();
    float inference_time = 0.0f;
    for(int i=0;i<100;++i){
        string image = imagePath + "/" + to_string(i) + ".bin";
        load_input(image, 3 * 224 * 224, input);
        hipMemcpy(dInput,input,224*224*3*sizeof(float),hipMemcpyHostToDevice);
        hipEventRecord(event_start);
        output = block1_conv1.forward(dInput);
        output = relu1.forward(output);
        output = block1_conv2.forward(output);
        output = relu2.forward(output);
        output = block1_pool.forward(output);
        output = block2_conv1.forward(output);
        output = relu3.forward(output);
        if(relu3.sparse){
            output = block2_conv2.forward(output);
        }else{
            output = block2_conv2_cudnn.forward(output);
        }
        output = relu4.forward(output);
        output = block2_pool.forward(output);
        output = block3_conv1.forward(output);
        output = relu5.forward(output);
        if(relu5.sparse){
            output = block3_conv2.forward(output);
        }else{
            output = block3_conv2_cudnn.forward(output);
        }
        output = relu6.forward(output);
        if(relu6.sparse){
            output = block3_conv3.forward(output);
        }else{
            output = block3_conv3_cudnn.forward(output);
        }
        output = relu6_cudnn.forward(output);
        output = block3_pool.forward(output);
        output = block4_conv1.forward(output);
        output = relu7.forward(output);
        if(relu7.sparse){
            output = block4_conv2.forward(output);
        }else{
            output = block4_conv2_cudnn.forward(output);
        }
        output = relu8.forward(output);
        if(relu8.sparse){
            output = block4_conv3.forward(output);
        }else{
            output = block4_conv3_cudnn.forward(output);
        }
        output = relu9.forward(output);
        output = block4_pool.forward(output);
        output = block5_conv1.forward(output);
        output = relu10.forward(output);
        output = block5_conv2.forward(output);
        output = relu11.forward(output);
        output = block5_conv3.forward(output);
        output = relu12.forward(output);
        output = block5_pool.forward(output);
        output = fc1.forward(output);
        output = fc2.forward(output);
        output = fc3.forward(output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<"vgg16 cuSNN costs "<<inference_time/100<<" ms"<<endl;
    cout<<endl;
    unsigned int outputSize = 1000;
    float *hOutput = (float *)malloc(outputSize*sizeof(float));
    hipMemcpy(hOutput,output,outputSize*sizeof(float),hipMemcpyDeviceToHost);
    vector<float> out_bin;
    for(int i=0;i<outputSize;++i){
        out_bin.push_back(hOutput[i]);
    }
    std::ofstream ofp("s.bin", std::ios::out | std::ios::binary);
    ofp.write(reinterpret_cast<const char*>(out_bin.data()), out_bin.size() * sizeof(float));
    return 0;
}
