


#include <hip/hip_runtime.h>
#include <iostream>
#include "cudnn.h"
#include <fstream>
#include <vector>
using namespace std;
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
void load_input(string input_path,unsigned int dataSize,float *input){
    std::ifstream fin(input_path, std::ios::binary);
    if (!fin) {
        cerr<<"load weight failed "<<input_path<<endl;
        exit(-1);
    }
    vector<float> temp_vector;
    float f;
    while (fin.read(reinterpret_cast<char*>(&f), sizeof(float))){
        temp_vector.push_back(f);
    }
    cout<<dataSize<<" "<<temp_vector.size()<<endl;
    for(int i=0;i<dataSize;++i){
        input[i] = temp_vector[i];
    }
}
float diff(float *a,float *b,unsigned int c){
    float result = 0.0f;
    for(unsigned int i=0;i<c;++i){
        result += abs(a[i] - b[i]);
    }
    return result;
}
class Conv{
public:
    unsigned int H;
    unsigned int W;
    unsigned int C;
    unsigned int B;
    unsigned int N;
    unsigned int PAD;
    unsigned int hOut;
    unsigned int wOut;
    unsigned int R;
    unsigned int S;
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    float beta2 = 1.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    cudnnTensorDescriptor_t biasDescriptor;
    float *output;
    float *kernel;
    float *bias;
    void initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                    unsigned int pad,unsigned int r,unsigned int s,unsigned int stride,string weightFile);
    float *forward(float *input);
};
void Conv::initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                      unsigned int pad,unsigned int r,unsigned int s,unsigned int stride,string weightFile){
    this->B = b;
    this->C = c;
    this->H = h;
    this->W = w;
    this->N = n;
    this->R = r;
    this->S = s;
    this->hOut = (H+2*pad - r)/stride + 1;
    this->wOut = (W+2*pad - s)/stride + 1;
    hipMalloc(&kernel,sizeof(float)*C*N*R*S);
    hipMalloc(&bias,sizeof(float)*N);
    hipMalloc(&this->output,sizeof(float)*B*hOut*wOut*N);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnCreateTensorDescriptor(&biasDescriptor);
    cudnnSetTensor4dDescriptor(biasDescriptor,
            /*format=*/CUDNN_TENSOR_NHWC,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/1,
            /*image_width=*/1);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NHWC,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/pad,
            /*pad_width=*/pad,
            /*vertical_stride=*/stride,
            /*horizontal_stride=*/stride,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NHWC,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/N,
            /*image_height=*/hOut,
            /*image_width=*/wOut);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N + N;//kernel + bias
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    try{
        load_input(weightFile,kernelSize,cpuKernel);
    }catch (const char* msg) {
        cerr << msg << endl;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(bias,&cpuKernel[R*S*C*N],N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * Conv::forward(float *input) {
    hipMemset(output, 0, B*N*hOut*wOut*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    checkCUDNN(cudnnAddTensor(convCudnn,&alpha,biasDescriptor,bias,&beta2,convOutputDescriptor,output));
    return output;
}
__device__ void conv_1_512_28_28_512_switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
    switch (switch_condition) {
        case 0:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(0-s)] += result;
                }
            }
            break;
        case 1:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(1-s)] += result;
                }
            }
            break;
        case 2:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(2-s)] += result;
                }
            }
            break;
        case 3:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(3-s)] += result;
                }
            }
            break;
        case 4:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(4-s)] += result;
                }
            }
            break;
        case 5:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(5-s)] += result;
                }
            }
            break;
        case 6:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(6-s)] += result;
                }
            }
            break;
        case 7:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(7-s)] += result;
                }
            }
            break;
        case 8:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(8-s)] += result;
                }
            }
            break;
        case 9:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(0-s)] += result;
                }
            }
            break;
        case 10:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(1-s)] += result;
                }
            }
            break;
        case 11:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(2-s)] += result;
                }
            }
            break;
        case 12:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(3-s)] += result;
                }
            }
            break;
        case 13:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(4-s)] += result;
                }
            }
            break;
        case 14:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(5-s)] += result;
                }
            }
            break;
        case 15:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(6-s)] += result;
                }
            }
            break;
        case 16:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(7-s)] += result;
                }
            }
            break;
        case 17:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(8-s)] += result;
                }
            }
            break;
        case 18:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(0-s)] += result;
                }
            }
            break;
        case 19:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(1-s)] += result;
                }
            }
            break;
        case 20:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(2-s)] += result;
                }
            }
            break;
        case 21:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(3-s)] += result;
                }
            }
            break;
        case 22:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(4-s)] += result;
                }
            }
            break;
        case 23:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(5-s)] += result;
                }
            }
            break;
        case 24:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(6-s)] += result;
                }
            }
            break;
        case 25:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(7-s)] += result;
                }
            }
            break;
        case 26:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(8-s)] += result;
                }
            }
            break;
        case 27:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(0-s)] += result;
                }
            }
            break;
        case 28:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(1-s)] += result;
                }
            }
            break;
        case 29:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(2-s)] += result;
                }
            }
            break;
        case 30:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(3-s)] += result;
                }
            }
            break;
        case 31:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(4-s)] += result;
                }
            }
            break;
        case 32:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(5-s)] += result;
                }
            }
            break;
        case 33:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(6-s)] += result;
                }
            }
            break;
        case 34:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(7-s)] += result;
                }
            }
            break;
        case 35:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(8-s)] += result;
                }
            }
            break;
        case 36:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(0-s)] += result;
                }
            }
            break;
        case 37:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(1-s)] += result;
                }
            }
            break;
        case 38:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(2-s)] += result;
                }
            }
            break;
        case 39:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(3-s)] += result;
                }
            }
            break;
        case 40:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(4-s)] += result;
                }
            }
            break;
        case 41:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(5-s)] += result;
                }
            }
            break;
        case 42:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(6-s)] += result;
                }
            }
            break;
        case 43:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(7-s)] += result;
                }
            }
            break;
        case 44:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(8-s)] += result;
                }
            }
            break;
        case 45:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(0-s)] += result;
                }
            }
            break;
        case 46:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(1-s)] += result;
                }
            }
            break;
        case 47:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(2-s)] += result;
                }
            }
            break;
        case 48:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(3-s)] += result;
                }
            }
            break;
        case 49:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(4-s)] += result;
                }
            }
            break;
        case 50:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(5-s)] += result;
                }
            }
            break;
        case 51:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(6-s)] += result;
                }
            }
            break;
        case 52:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(7-s)] += result;
                }
            }
            break;
        case 53:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(8-s)] += result;
                }
            }
            break;
        case 54:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(0-s)] += result;
                }
            }
            break;
        case 55:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(1-s)] += result;
                }
            }
            break;
        case 56:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(2-s)] += result;
                }
            }
            break;
        case 57:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(3-s)] += result;
                }
            }
            break;
        case 58:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(4-s)] += result;
                }
            }
            break;
        case 59:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(5-s)] += result;
                }
            }
            break;
        case 60:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(6-s)] += result;
                }
            }
            break;
        case 61:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(7-s)] += result;
                }
            }
            break;
        case 62:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(8-s)] += result;
                }
            }
            break;

    }
}
__global__ void conv_1_512_28_28_512_transform(float *matrix, float *d_nnz, char *ids, unsigned int *c_lens){
    unsigned int global_id = blockIdx.x * 512 + threadIdx.x;
    unsigned int batch_id = global_id/(512*28*28);
    if(global_id >= 1*28*28*512){
        return ;
    }
    const float v = matrix[global_id];
    if(v == 0.0f){
        return;
    }
    unsigned int c = global_id % 512;

    int h = ((global_id - batch_id * 512 * 28 * 28)/512)/28+1;
    int w = ((global_id - batch_id * 512 * 28 * 28)/512)%28+1;
    int th_start = min(h/5,6-1);
    int tw_start = min(w/7,4-1);
    for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
        if((tile_h_id*5+5+3-1)<=h){
            break;
        }
        for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
            if((tile_w_id*7+7+3-1)<=w){
                break;
            }
            unsigned int tile_id = tile_h_id * 4 + tile_w_id;
            unsigned int index = atomicAdd(&c_lens[batch_id*512*24+c*24+tile_id],1);
            unsigned int abs_h = h - tile_h_id*5;
            unsigned int abs_w = w - tile_w_id*7;
            d_nnz[batch_id*512*24*(5+3-1)*(7+3-1)+
                  c*24*(5+3-1)*(7+3-1)+tile_id*(5+3-1)*(7+3-1)+index] = v;
            ids[batch_id*512*24*(5+3-1)*(7+3-1)+
                c*24*(5+3-1)*(7+3-1)+tile_id*(5+3-1)*(7+3-1)+index] = abs_h*(7+3-1)+abs_w;
        }
    }
}
__device__ void conv_1_512_28_28_512_load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<3;++r){
        for(unsigned int s=0;s<3;++s){
            data_array[r*3+s] = kernel[c_index*512*3*3+r*3*512+s*512+n_id];
        }
    }
}
__device__ void conv_1_512_28_28_512_load_input_2_shared_memory(float *values,unsigned int *c_lens,char *ids,
                                           float *shared_input,char *shared_ids,unsigned int *shared_lens,
                                           unsigned int warp_id,unsigned int lane_id,unsigned int batch_id,
                                           unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<16&&tile_c_id+c_id<512;c_id+=16){
        unsigned int end_index = c_lens[batch_id*512*24+(tile_c_id+c_id)*24+tile_id];
        if(lane_id ==0){
            shared_lens[c_id] = end_index;
        }
        for(unsigned int id = lane_id;id<end_index;id+=32){
            shared_input[c_id*(5+3-1)*(7+3-1)+id] = values[batch_id*512*24*(5+3-1)*(7+3-1)+
                                                           (tile_c_id+c_id)*24*(5+3-1)*(7+3-1)+tile_id*(5+3-1)*(7+3-1)+id];
            shared_ids[c_id*(5+3-1)*(7+3-1)+id] = ids[batch_id*512*24*(5+3-1)*(7+3-1)+
                                                      (tile_c_id+c_id)*24*(5+3-1)*(7+3-1)+tile_id*(5+3-1)*(7+3-1)+id];
        }
    }
}
__global__ void conv_1_512_28_28_512_conv2d(float * __restrict__ values, unsigned int * __restrict__ c_lens,
                       char * __restrict__ ids,
                       const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[16*(5+3-1)*(7+3-1)];
    __shared__ char input_ids[16*(5+3-1)*(7+3-1)];
    __shared__ unsigned int channel_lens[(16)];

    const unsigned int batch_id = (blockIdx.x/(32*24));
    const unsigned int t_id = (blockIdx.x - batch_id*32*24)/32;
    const unsigned int tile_h_id = (t_id / 4)*5;
    const unsigned int tile_w_id = (t_id % 4)*7;
    const unsigned int index = blockIdx.x % (32);
    const unsigned int start_channel_index = index*16;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[5*7] = {0.0f};
    conv_1_512_28_28_512_load_input_2_shared_memory(values,c_lens,ids,input,input_ids,channel_lens,warp_id,lane_id,batch_id,t_id,start_channel_index);
    __syncthreads();
    float v;
    unsigned int id;
    for(unsigned int n = threadIdx.x;n<512;n+=512){
        for(unsigned int c=start_channel_index;c<start_channel_index+16&&c<512;c++){
            unsigned int abs_c = c - start_channel_index;
            unsigned int start_index = abs_c*(5+3-1)*(7+3-1);
            unsigned int end_index = start_index+channel_lens[abs_c];
            if(start_index == end_index){
                continue;
            }
            conv_1_512_28_28_512_load_data_2_register(data_array,(c),kernel,n);
            unsigned int iters = end_index - start_index;
            for(unsigned int iter=0;iter<iters;iter++) {
                v = input[iter+start_index];
                id = input_ids[iter+start_index];
                conv_1_512_28_28_512_switch_function(id,data_array,v,temp_result);
            }
        }
        for (unsigned int th = 0; th < 5; ++th) {
            for (unsigned int tw = 0; tw < 7; ++tw) {
                if (tile_h_id + th >= 28 || tile_w_id + tw >= 28) {
                    continue;
                }
                atomicAdd(&outputs[batch_id * 512 * 28 * 28 + (tile_h_id + th) * 28 * 512 + (tile_w_id + tw) * 512 +
                                   n],temp_result[(th * 7 + tw)]);
            }
        }
        for(unsigned int i=0;i<5*7;++i){
            temp_result[i] = 0.0f;
        }
    }
}
__device__ void conv_1_512_14_14_512_switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
    switch (switch_condition) {
        case 0:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(0-s)] += result;
                }
            }
            break;
        case 1:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(1-s)] += result;
                }
            }
            break;
        case 2:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(2-s)] += result;
                }
            }
            break;
        case 3:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(3-s)] += result;
                }
            }
            break;
        case 4:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(4-s)] += result;
                }
            }
            break;
        case 5:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(5-s)] += result;
                }
            }
            break;
        case 6:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(6-s)] += result;
                }
            }
            break;
        case 7:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(7-s)] += result;
                }
            }
            break;
        case 8:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(8-s)] += result;
                }
            }
            break;
        case 9:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(0-s)] += result;
                }
            }
            break;
        case 10:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(1-s)] += result;
                }
            }
            break;
        case 11:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(2-s)] += result;
                }
            }
            break;
        case 12:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(3-s)] += result;
                }
            }
            break;
        case 13:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(4-s)] += result;
                }
            }
            break;
        case 14:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(5-s)] += result;
                }
            }
            break;
        case 15:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(6-s)] += result;
                }
            }
            break;
        case 16:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(7-s)] += result;
                }
            }
            break;
        case 17:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(8-s)] += result;
                }
            }
            break;
        case 18:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(0-s)] += result;
                }
            }
            break;
        case 19:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(1-s)] += result;
                }
            }
            break;
        case 20:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(2-s)] += result;
                }
            }
            break;
        case 21:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(3-s)] += result;
                }
            }
            break;
        case 22:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(4-s)] += result;
                }
            }
            break;
        case 23:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(5-s)] += result;
                }
            }
            break;
        case 24:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(6-s)] += result;
                }
            }
            break;
        case 25:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(7-s)] += result;
                }
            }
            break;
        case 26:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(8-s)] += result;
                }
            }
            break;
        case 27:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(0-s)] += result;
                }
            }
            break;
        case 28:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(1-s)] += result;
                }
            }
            break;
        case 29:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(2-s)] += result;
                }
            }
            break;
        case 30:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(3-s)] += result;
                }
            }
            break;
        case 31:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(4-s)] += result;
                }
            }
            break;
        case 32:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(5-s)] += result;
                }
            }
            break;
        case 33:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(6-s)] += result;
                }
            }
            break;
        case 34:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(7-s)] += result;
                }
            }
            break;
        case 35:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(8-s)] += result;
                }
            }
            break;
        case 36:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(0-s)] += result;
                }
            }
            break;
        case 37:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(1-s)] += result;
                }
            }
            break;
        case 38:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(2-s)] += result;
                }
            }
            break;
        case 39:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(3-s)] += result;
                }
            }
            break;
        case 40:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(4-s)] += result;
                }
            }
            break;
        case 41:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(5-s)] += result;
                }
            }
            break;
        case 42:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(6-s)] += result;
                }
            }
            break;
        case 43:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(7-s)] += result;
                }
            }
            break;
        case 44:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(8-s)] += result;
                }
            }
            break;
        case 45:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(0-s)] += result;
                }
            }
            break;
        case 46:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(1-s)] += result;
                }
            }
            break;
        case 47:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(2-s)] += result;
                }
            }
            break;
        case 48:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(3-s)] += result;
                }
            }
            break;
        case 49:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(4-s)] += result;
                }
            }
            break;
        case 50:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(5-s)] += result;
                }
            }
            break;
        case 51:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(6-s)] += result;
                }
            }
            break;
        case 52:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(7-s)] += result;
                }
            }
            break;
        case 53:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(8-s)] += result;
                }
            }
            break;
        case 54:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(0-s)] += result;
                }
            }
            break;
        case 55:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(1-s)] += result;
                }
            }
            break;
        case 56:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(2-s)] += result;
                }
            }
            break;
        case 57:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(3-s)] += result;
                }
            }
            break;
        case 58:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(4-s)] += result;
                }
            }
            break;
        case 59:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(5-s)] += result;
                }
            }
            break;
        case 60:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(6-s)] += result;
                }
            }
            break;
        case 61:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(7-s)] += result;
                }
            }
            break;
        case 62:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(8-s)] += result;
                }
            }
            break;

    }
}
__global__ void conv_1_512_14_14_512_transform(float *matrix, float *d_nnz, char *ids, unsigned int *c_lens){
    unsigned int global_id = blockIdx.x * 512 + threadIdx.x;
    unsigned int batch_id = global_id/(512*14*14);
    if(global_id >= 1*14*14*512){
        return ;
    }
    const float v = matrix[global_id];
    if(v == 0.0f){
        return;
    }
    unsigned int c = global_id % 512;

    int h = ((global_id - batch_id * 512 * 14 * 14)/512)/14+1;
    int w = ((global_id - batch_id * 512 * 14 * 14)/512)%14+1;
    int th_start = min(h/5,3-1);
    int tw_start = min(w/7,2-1);
    for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
        if((tile_h_id*5+5+3-1)<=h){
            break;
        }
        for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
            if((tile_w_id*7+7+3-1)<=w){
                break;
            }
            unsigned int tile_id = tile_h_id * 2 + tile_w_id;
            unsigned int index = atomicAdd(&c_lens[batch_id*512*6+c*6+tile_id],1);
            unsigned int abs_h = h - tile_h_id*5;
            unsigned int abs_w = w - tile_w_id*7;
            d_nnz[batch_id*512*6*(5+3-1)*(7+3-1)+
                  c*6*(5+3-1)*(7+3-1)+tile_id*(5+3-1)*(7+3-1)+index] = v;
            ids[batch_id*512*6*(5+3-1)*(7+3-1)+
                c*6*(5+3-1)*(7+3-1)+tile_id*(5+3-1)*(7+3-1)+index] = abs_h*(7+3-1)+abs_w;
        }
    }
}
__device__ void conv_1_512_14_14_512_load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<3;++r){
        for(unsigned int s=0;s<3;++s){
            data_array[r*3+s] = kernel[c_index*512*3*3+r*3*512+s*512+n_id];
        }
    }
}
__device__ void conv_1_512_14_14_512_load_input_2_shared_memory(float *values,unsigned int *c_lens,char *ids,
                                           float *shared_input,char *shared_ids,unsigned int *shared_lens,
                                           unsigned int warp_id,unsigned int lane_id,unsigned int batch_id,
                                           unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<8&&tile_c_id+c_id<512;c_id+=16){
        unsigned int end_index = c_lens[batch_id*512*6+(tile_c_id+c_id)*6+tile_id];
        if(lane_id ==0){
            shared_lens[c_id] = end_index;
        }
        for(unsigned int id = lane_id;id<end_index;id+=32){
            shared_input[c_id*(5+3-1)*(7+3-1)+id] = values[batch_id*512*6*(5+3-1)*(7+3-1)+
                                                           (tile_c_id+c_id)*6*(5+3-1)*(7+3-1)+tile_id*(5+3-1)*(7+3-1)+id];
            shared_ids[c_id*(5+3-1)*(7+3-1)+id] = ids[batch_id*512*6*(5+3-1)*(7+3-1)+
                                                      (tile_c_id+c_id)*6*(5+3-1)*(7+3-1)+tile_id*(5+3-1)*(7+3-1)+id];
        }
    }
}
__global__ void conv_1_512_14_14_512_conv2d(float * __restrict__ values, unsigned int * __restrict__ c_lens,
                       char * __restrict__ ids,
                       const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[8*(5+3-1)*(7+3-1)];
    __shared__ char input_ids[8*(5+3-1)*(7+3-1)];
    __shared__ unsigned int channel_lens[(8)];

    const unsigned int batch_id = (blockIdx.x/(64*6));
    const unsigned int t_id = (blockIdx.x - batch_id*64*6)/64;
    const unsigned int tile_h_id = (t_id / 2)*5;
    const unsigned int tile_w_id = (t_id % 2)*7;
    const unsigned int index = blockIdx.x % (64);
    const unsigned int start_channel_index = index*8;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[5*7] = {0.0f};
    conv_1_512_14_14_512_load_input_2_shared_memory(values,c_lens,ids,input,input_ids,channel_lens,warp_id,lane_id,batch_id,t_id,start_channel_index);
    __syncthreads();
    float v;
    unsigned int id;
    for(unsigned int n = threadIdx.x;n<512;n+=512){
        for(unsigned int c=start_channel_index;c<start_channel_index+8&&c<512;c++){
            unsigned int abs_c = c - start_channel_index;
            unsigned int start_index = abs_c*(5+3-1)*(7+3-1);
            unsigned int end_index = start_index+channel_lens[abs_c];
            if(start_index == end_index){
                continue;
            }
            conv_1_512_14_14_512_load_data_2_register(data_array,(c),kernel,n);
            unsigned int iters = end_index - start_index;
            for(unsigned int iter=0;iter<iters;iter++) {
                v = input[iter+start_index];
                id = input_ids[iter+start_index];
                conv_1_512_14_14_512_switch_function(id,data_array,v,temp_result);
            }
        }
        for (unsigned int th = 0; th < 5; ++th) {
            for (unsigned int tw = 0; tw < 7; ++tw) {
                if (tile_h_id + th >= 14 || tile_w_id + tw >= 14) {
                    continue;
                }
                atomicAdd(&outputs[batch_id * 512 * 14 * 14 + (tile_h_id + th) * 14 * 512 + (tile_w_id + tw) * 512 +
                                   n],temp_result[(th * 7 + tw)]);
            }
        }
        for(unsigned int i=0;i<5*7;++i){
            temp_result[i] = 0.0f;
        }
    }
}
class Conv_1_512_28_28_512{
public:
    unsigned int H;
    unsigned int W;
    unsigned int C;
    unsigned int B;
    unsigned int N;
    unsigned int hOut;
    unsigned int wOut;
    unsigned int TH = 5;
    unsigned int TW = 7;
    unsigned int TC = 16;
    unsigned int TCS;
    float alpha = 1.0f;
    float beta = 0.0f;
    float beta2 = 1.0f;
    cudnnHandle_t convCudnn;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnTensorDescriptor_t biasDescriptor;
    float *cpuKernel;
    float *output;
    float *kernel;
    float *bias;
    float * nnz;
    unsigned int TILES_EACH_CHANNEL;
    unsigned int * c_lens;
    char * ids;
    void initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                    unsigned int pad,unsigned int r,unsigned int s,unsigned int stride,string weightFile);
    float * forward(float *input);
};
class Conv_1_512_14_14_512{
public:
    unsigned int H;
    unsigned int W;
    unsigned int C;
    unsigned int B;
    unsigned int N;
    unsigned int hOut;
    unsigned int wOut;
    unsigned int TH = 5;
    unsigned int TW = 7;
    unsigned int TC = 8;
    unsigned int TCS;
    float alpha = 1.0f;
    float beta = 0.0f;
    float beta2 = 1.0f;
    cudnnHandle_t convCudnn;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnTensorDescriptor_t biasDescriptor;
    float *cpuKernel;
    float *output;
    float *kernel;
    float *bias;
    float * nnz;
    unsigned int TILES_EACH_CHANNEL;
    unsigned int * c_lens;
    char * ids;
    void initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                    unsigned int pad,unsigned int r,unsigned int s,unsigned int stride,string weightFile);
    float * forward(float *input);
};
void Conv_1_512_28_28_512::initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                                      unsigned int pad,unsigned int r,unsigned int s,unsigned int stride,string weightFile) {
    this->B = b;
    this->C = c;
    this->H = h;
    this->W = w;
    this->N = n;
    this->hOut = h;
    this->wOut = w;
    cudnnCreate(&convCudnn);
    unsigned int kernelSize = 3*3*C*N + N;//kernel + bias
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    try{
        load_input(weightFile,kernelSize,cpuKernel);
    }catch (const char* msg) {
        cerr << msg << endl;
    }
    float *temp_kernel = (float *)malloc(kernelSize*sizeof(float));
    for(unsigned int i=0;i<N;++i){
        for(unsigned int l=0;l<C;++l){
            for(unsigned int j=0;j<3;++j){
                for(unsigned int k=0;k<3;++k){
                    temp_kernel[l*N*3*3+j*3*N+k*N+i] = cpuKernel[i*3*3*C+l*9+j*3+k];
                }
            }
        }
    }
    hipMalloc(&kernel,9*C*N*sizeof(float));
    hipMemcpy(kernel,temp_kernel,9*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(temp_kernel);
    hipMalloc(&bias,N*sizeof(float));
    hipMemcpy(bias,&cpuKernel[9*C*N],N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
    TILES_EACH_CHANNEL = ((H-1)/TH+1)*((W-1)/TW+1);
    TCS = (C-1)/TC + 1;
    hipMalloc(&nnz,b*c*TILES_EACH_CHANNEL*(TH+3-1)*(TW+3-1)*sizeof(float));
    hipMalloc(&ids,b*c*TILES_EACH_CHANNEL*(TH+3-1)*(TW+3-1)*sizeof(char));
    hipMalloc(&c_lens,b*c*TILES_EACH_CHANNEL*sizeof(unsigned int));
    hipMalloc(&output,b*n*h*w*sizeof(float));
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NHWC,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/N,
            /*image_height=*/hOut,
            /*image_width=*/wOut);
    cudnnCreateTensorDescriptor(&biasDescriptor);
    cudnnSetTensor4dDescriptor(biasDescriptor,
            /*format=*/CUDNN_TENSOR_NHWC,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/1,
            /*image_width=*/1);
}
float * Conv_1_512_28_28_512::forward(float *input) {
    hipMemset(output, 0, B*N*hOut*wOut*sizeof(float));
    hipMemset(c_lens, 0, B*C*TILES_EACH_CHANNEL*sizeof(unsigned int));
    conv_1_512_28_28_512_transform<<<(B*C*H*W-1)/512+1,512>>>(input,this->nnz,this->ids,this->c_lens);
    conv_1_512_28_28_512_conv2d<<<B*TCS*TILES_EACH_CHANNEL,N>>>(this->nnz,this->c_lens,this->ids,this->kernel,this->output);

    chkerr(hipGetLastError());
    chkerr(hipDeviceSynchronize());
    checkCUDNN(cudnnAddTensor(convCudnn,&alpha,biasDescriptor,bias,&beta2,convOutputDescriptor,output));
    return output;
}
float * Conv_1_512_14_14_512::forward(float *input) {
    hipMemset(output, 0, B*N*hOut*wOut*sizeof(float));
    hipMemset(c_lens, 0, B*C*TILES_EACH_CHANNEL*sizeof(unsigned int));
    conv_1_512_14_14_512_transform<<<(B*C*H*W-1)/512+1,512>>>(input,this->nnz,this->ids,this->c_lens);
    conv_1_512_14_14_512_conv2d<<<B*TCS*TILES_EACH_CHANNEL,N>>>(this->nnz,this->c_lens,this->ids,this->kernel,this->output);
    chkerr(hipGetLastError());
    chkerr(hipDeviceSynchronize());
    checkCUDNN(cudnnAddTensor(convCudnn,&alpha,biasDescriptor,bias,&beta2,convOutputDescriptor,output));
    return output;
}
int main(void){
    Conv_1_512_28_28_512 conv_28;
    Conv conv_28_cudnn;
    conv_28.initialize(1,512,28,28,512,1,3,3,1,"/home/lizhi/cuSNN/2080ti/vgg16Scnn/weights/block4_conv2.bin");
    conv_28_cudnn.initialize(1,512,28,28,512,1,3,3,1,"/home/lizhi/cuSNN/2080ti/vgg16Scnn/weights/block4_conv2.bin");
    cout<<"123"<<endl;
    float *h1 = new float[512*28*28];
    float *h2 = new float[512*28*28];
    float *input;
    hipMalloc(&input,512*28*28*sizeof(float));
    conv_28_cudnn.forward(input);
    conv_28.forward(input);
    hipMemcpy(h1,conv_28.output,512*28*28*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(h2,conv_28_cudnn.output,512*28*28*sizeof(float),hipMemcpyDeviceToHost);
    cout<<diff(h1,h2,512*28*28)<<endl;
    return 0;
}
