

#include "../inc/scnnOps.h"
void Relu::initialize(unsigned int b, unsigned int c, unsigned int h, unsigned int w, float threshold) {
    B = b;
    C = c;
    H = h;
    W = w;
    sparse_threshold = threshold;
    hipMalloc(&counter,1*sizeof(unsigned int));
    h_counter = new unsigned int[1];
}
float * Relu::forward(float *input) {
    sparse = false;
    hipMemset(counter,0,sizeof(unsigned int));
    relu<<<68,1024>>>(input,B,C,H,W,counter);
    hipMemcpy(h_counter,counter,1*sizeof(unsigned int),hipMemcpyDeviceToHost);
    if(float(h_counter[0])/float(B*C*H*W) >=sparse_threshold){
        sparse = true;
    }
    return input;
}
