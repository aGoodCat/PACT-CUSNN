#include "hip/hip_runtime.h"
#include "../inc/cudnnOps.h"
#include "../inc/cudnnOps.h"
__global__ void concateX(unsigned int b,unsigned int c1,unsigned int c2,unsigned int h,unsigned int w,float *x,float *z){
    unsigned int gId = threadIdx.x + blockIdx.x*1024;
    if(gId>=b*c1*h*w){
        return ;
    }
    for(unsigned int i=gId;i<b*c1*h*w;i+=8*1024){
        unsigned int id = i/c1;
        unsigned int cId = i%c1;
        z[id*(c1+c2)+cId] = x[id*c1+cId];
    }
}
__global__ void concateY(unsigned int b,unsigned int c1,unsigned int c2,unsigned int h,unsigned int w,float *y,float *z){
    unsigned int gId = threadIdx.x + blockIdx.x*1024;
    if(gId>=b*c2*h*w){
        return ;
    }
    for(unsigned int i=gId;i<b*c2*h*w;i+=8*1024){
        unsigned int id = i/c2;
        unsigned int cId = i%c2;
        z[id*(c1+c2)+cId+c1] = y[id*c2+cId];
    }
}
void Concate::initialize(unsigned int b, unsigned int c1, unsigned int c2, unsigned int h, unsigned int w) {
    B = b;
    C1 = c1;
    C2 = c2;
    H = h;
    W = w;
    hipMalloc(&output,b*(c1+c2)*h*w*sizeof(float));
    for (int i = 0; i < 2; i ++){
        hipStreamCreate(&streams[i]);
    }
}
float * Concate::forward(float *x, float *y) {
    concateX<<<8,1024,0,streams[0]>>>(B,C1,C2,H,W,x,output);
    concateY<<<8,1024,0,streams[1]>>>(B,C1,C2,H,W,y,output);
    return output;
}