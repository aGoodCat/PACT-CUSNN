#include "./inc/conv_and_pool.h"
#include "./inc/nvml_helper.hpp"

int main(int argc, char *argv[]){
    float *input = (float *)malloc(224*224*3*sizeof(float));
    string imagePath = argv[1];
    load_input(imagePath+"/" + to_string(0) + ".bin",3*224*224,input);
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    float *dInput;
    hipMalloc(&dInput,224*224*3*sizeof(float));

    Conv block1_conv1;
    block1_conv1.initialize(1,3,224,224,64,1,3,3,1,
                            "../../../weights/vgg19/weights/block1_conv1.bin");
    Relu relu1;
    relu1.initialize(1,64,224,224,0.62);
    Conv_1_64_224_224_64 block1_conv2;
    block1_conv2.initialize(1,64,224,224,64,1,3,3,
                            1,"../../../weights/vgg19/weights/block1_conv2.bin");
    Conv block1_conv2_cudnn;
    block1_conv2_cudnn.initialize(1,64,224,224,64,1,3,3,1,
                                  "../../../weights/vgg19/weights/block1_conv2.bin");
    Activation relu2;
    relu2.initialize(1,64,224,224);

    Pool block1_pool;
    block1_pool.initialize(1,64,224,224,1,3,3,HIPDNN_POOLING_MAX,2);
    Conv block2_conv1;
    block2_conv1.initialize(1,64,112,112,128,1,3,3,1,
                            "../../../weights/vgg19/weights/block2_conv1.bin");
    Relu relu3;
    relu3.initialize(1,128,112,112,0.75);
    Conv_1_128_112_112_128 block2_conv2;
    Conv block2_conv2_cudnn;
    block2_conv2.initialize(1,128,112,112,128,1,3,3,1,
                            "../../../weights/vgg19/weights/block2_conv2.bin");
    block2_conv2_cudnn.initialize(1,128,112,112,128,1,3,3,1,
                                  "../../../weights/vgg19/weights/block2_conv2.bin");
    Activation relu4;
    relu4.initialize(1,128,112,112);

    Pool block2_pool;
    block2_pool.initialize(1,128,112,112,1,3,3,HIPDNN_POOLING_MAX,2);
    Conv block3_conv1;
    block3_conv1.initialize(1,128,56,56,256,1,3,3,1,
                            "../../../weights/vgg19/weights/block3_conv1.bin");
    Relu relu5;
    relu5.initialize(1,256,56,56,0.6);
    Conv_1_256_56_56_256 block3_conv2;
    Conv block3_conv2_cudnn;
    block3_conv2.initialize(1,256,56,56,256,1,3,3,1,
                            "../../../weights/vgg19/weights/block3_conv2.bin");
    block3_conv2_cudnn.initialize(1,256,56,56,256,1,3,3,
                                  1,"../../../weights/vgg19/weights/block3_conv2.bin");
    Relu relu6;
    relu6.initialize(1,256,56,56,0.6);
    Conv_1_256_56_56_256 block3_conv3;
    Conv block3_conv3_cudnn;
    block3_conv3.initialize(1,256,56,56,256,1,3,3,
                            1,"../../../weights/vgg19/weights/block3_conv3.bin");
    block3_conv3_cudnn.initialize(1,256,56,56,256,1,3,3,
                                  1,"../../../weights/vgg19/weights/block3_conv3.bin");
    Relu relu7;
    relu7.initialize(1,256,56,56,0.6);
    Conv block3_conv4_cudnn;
    Conv_1_256_56_56_256 block3_conv4;
    block3_conv4.initialize(1,256,56,56,256,1,3,3,
                            1,"../../../weights/vgg19/weights/block3_conv4.bin");
    block3_conv4_cudnn.initialize(1,256,56,56,256,1,3,3,
                                  1,"../../../weights/vgg19/weights/block3_conv4.bin");
    Activation relu8;
    relu8.initialize(1,256,56,56);

    Pool block3_pool;
    block3_pool.initialize(1,256,56,56,1,3,3,HIPDNN_POOLING_MAX,2);
    Conv block4_conv1;
    block4_conv1.initialize(1,256,28,28,512,1,3,3,
                            1,"../../../weights/vgg19/weights/block4_conv1.bin");
    Relu relu9;
    relu9.initialize(1,512,28,28,0.4);
    Conv block4_conv2_cudnn;
    Conv_1_512_28_28_512 block4_conv2;
    block4_conv2.initialize(1,512,28,28,512,1,3,3,
                            1,"../../../weights/vgg19/weights/block4_conv2.bin");
    block4_conv2_cudnn.initialize(1,512,28,28,512,1,3,3,
                                  1,"../../../weights/vgg19/weights/block4_conv2.bin");
    Relu relu10;
    relu10.initialize(1,512,28,28,0.4);
    Conv block4_conv3_cudnn;
    Conv_1_512_28_28_512 block4_conv3;
    block4_conv3.initialize(1,512,28,28,512,1,3,3,
                            1,"../../../weights/vgg19/weights/block4_conv3.bin");
    block4_conv3_cudnn.initialize(1,512,28,28,512,1,3,3,
                                  1,"../../../weights/vgg19/weights/block4_conv3.bin");
    Relu relu11;
    relu11.initialize(1,512,28,28,0.4);
    Conv block4_conv4_cudnn;
    Conv_1_512_28_28_512 block4_conv4;
    block4_conv4.initialize(1,512,28,28,512,1,3,3,
                            1,"../../../weights/vgg19/weights/block4_conv4.bin");
    block4_conv4_cudnn.initialize(1,512,28,28,512,1,3,3,
                                  1,"../../../weights/vgg19/weights/block4_conv4.bin");
    Activation relu12;
    relu12.initialize(1,512,28,28);


    Pool block4_pool;
    block4_pool.initialize(1,512,28,28,1,3,3,HIPDNN_POOLING_MAX,2);
    Conv block5_conv1;
    block5_conv1.initialize(1,512,14,14,512,1,3,3,
                            1,"../../../weights/vgg19/weights/block5_conv1.bin");
    Relu relu13;
    relu13.initialize(1,512,14,14,0.4);
    Conv block5_conv2_cudnn;
    Conv_1_512_14_14_512 block5_conv2;
    block5_conv2.initialize(1,512,14,14,512,1,3,3,
                            1,"../../../weights/vgg19/weights/block5_conv2.bin");
    block5_conv2_cudnn.initialize(1,512,14,14,512,1,3,3,
                                  1,"../../../weights/vgg19/weights/block5_conv2.bin");
    Activation relu14;
    relu14.initialize(1,512,14,14);
    Conv block5_conv3_cudnn;
    Conv_1_512_14_14_512 block5_conv3;
    block5_conv3.initialize(1,512,14,14,512,1,3,3,
                            1,"../../../weights/vgg19/weights/block5_conv3.bin");
    block5_conv3_cudnn.initialize(1,512,14,14,512,1,3,3,
                                  1,"../../../weights/vgg19/weights/block5_conv3.bin");
    Activation relu15;
    relu15.initialize(1,512,14,14);
    Conv block5_conv4_cudnn;
    Conv_1_512_14_14_512 block5_conv4;
    block5_conv4.initialize(1,512,14,14,512,1,3,3,
                            1,"../../../weights/vgg19/weights/block5_conv4.bin");
    block5_conv4_cudnn.initialize(1,512,14,14,512,1,3,3,
                                  1,"../../../weights/vgg19/weights/block5_conv4.bin");
    Activation relu16;
    relu16.initialize(1,512,14,14);

    Pool block5_pool;
    block5_pool.initialize(1,512,14,14,1,3,3,HIPDNN_POOLING_MAX,2);
    FC fc1;
    fc1.initialize(1,25088,1,1,4096,0,1,1,1,"../../../weights/vgg19/weights/fc1.bin");
    FC fc2;
    fc2.initialize(1,4096,1,1,4096,0,1,1,1,"../../../weights/vgg19/weights/fc2.bin");
    FC fc3;
    fc3.initialize(1,4096,1,1,1000,0,1,1,1,"../../../weights/vgg19/weights/predictions.bin");

    hipMemcpy(dInput,input,224*224*3*sizeof(float),hipMemcpyHostToDevice);
    float *output;
    output = block1_conv1.forward(dInput);
    output = relu1.forward(output);
    if(relu1.sparse){
        output = block1_conv2.forward(output);
    }else{
        output = block1_conv2_cudnn.forward(output);
    }
    output = relu2.forward(output);
    output = block1_pool.forward(output);
    output = block2_conv1.forward(output);
    output = relu3.forward(output);
    if(relu3.sparse){
        output = block2_conv2.forward(output);
    }else{
        output = block2_conv2_cudnn.forward(output);
    }
    output = relu4.forward(output);
    output = block2_pool.forward(output);
    output = block3_conv1.forward(output);
    output = relu5.forward(output);
    if(relu5.sparse){
        output = block3_conv2.forward(output);
    }else{
        output = block3_conv2_cudnn.forward(output);
    }
    output = relu6.forward(output);
    if(relu6.sparse){
        output = block3_conv3.forward(output);
    }else{
        output = block3_conv3_cudnn.forward(output);
    }
    output = relu7.forward(output);
    if(relu7.sparse){
        output = block3_conv4.forward(output);
    }else{
        output = block3_conv4_cudnn.forward(output);
    }
    output = relu8.forward(output);
    output = block3_pool.forward(output);
    output = block4_conv1.forward(output);
    output = relu9.forward(output);
    if(relu9.sparse){
        output = block4_conv2.forward(output);
    }else{
        output = block4_conv2_cudnn.forward(output);
    }
    output = relu10.forward(output);
    if(relu10.sparse){
        output = block4_conv3.forward(output);
    }else{
        output = block4_conv3_cudnn.forward(output);
    }
    output = relu11.forward(output);
    if(relu11.sparse){
        output = block4_conv4.forward(output);
    }else{
        output = block4_conv4_cudnn.forward(output);
    }
    output = relu12.forward(output);
    output = block4_pool.forward(output);
    output = block5_conv1.forward(output);
    output = relu13.forward(output);
    if(relu13.sparse){
        output = block5_conv2.forward(output);
    }else{
        output = block5_conv2_cudnn.forward(output);
    }
    output = relu14.forward(output);
    output = block5_conv3.forward(output);
    output = relu15.forward(output);
    output = block5_conv4.forward(output);
    output = relu16.forward(output);
    output = block5_pool.forward(output);
    output = fc1.forward(output);
    output = fc2.forward(output);
    output = fc3.forward(output);
    hipDeviceSynchronize();
    float inference_time = 0.0f;
    for(int i=0;i<100;++i){
        string image = imagePath + "/" + to_string(i) + ".bin";
        load_input(image, 3 * 224 * 224, input);
        hipMemcpy(dInput,input,224*224*3*sizeof(float),hipMemcpyHostToDevice);
        hipEventRecord(event_start);
        output = block1_conv1.forward(dInput);
        output = relu1.forward(output);
        if(relu1.sparse){
            output = block1_conv2.forward(output);
        }else{
            output = block1_conv2_cudnn.forward(output);
        }
        output = relu2.forward(output);
        output = block1_pool.forward(output);
        output = block2_conv1.forward(output);
        output = relu3.forward(output);
        if(relu3.sparse){
            output = block2_conv2.forward(output);
        }else{
            output = block2_conv2_cudnn.forward(output);
        }
        output = relu4.forward(output);
        output = block2_pool.forward(output);
        output = block3_conv1.forward(output);
        output = relu5.forward(output);
        if(relu5.sparse){
            output = block3_conv2.forward(output);
        }else{
            output = block3_conv2_cudnn.forward(output);
        }
        output = relu6.forward(output);
        if(relu6.sparse){
            output = block3_conv3.forward(output);
        }else{
            output = block3_conv3_cudnn.forward(output);
        }
        output = relu7.forward(output);
        if(relu7.sparse){
            output = block3_conv4.forward(output);
        }else{
            output = block3_conv4_cudnn.forward(output);
        }
        output = relu8.forward(output);
        output = block3_pool.forward(output);
        output = block4_conv1.forward(output);
        output = relu9.forward(output);
        if(relu9.sparse){
            output = block4_conv2.forward(output);
        }else{
            output = block4_conv2_cudnn.forward(output);
        }
        output = relu10.forward(output);
        if(relu10.sparse){
            output = block4_conv3.forward(output);
        }else{
            output = block4_conv3_cudnn.forward(output);
        }
        output = relu11.forward(output);
        if(relu11.sparse){
            output = block4_conv4.forward(output);
        }else{
            output = block4_conv4_cudnn.forward(output);
        }
        output = relu12.forward(output);
        output = block4_pool.forward(output);
        output = block5_conv1.forward(output);
        output = relu13.forward(output);
        if(relu13.sparse){
            output = block5_conv2.forward(output);
        }else{
            output = block5_conv2_cudnn.forward(output);
        }
        output = relu14.forward(output);
        output = block5_conv3.forward(output);
        output = relu15.forward(output);
        output = block5_conv4.forward(output);
        output = relu16.forward(output);
        output = block5_pool.forward(output);
        output = fc1.forward(output);
        output = fc2.forward(output);
        output = fc3.forward(output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<"vgg19 cuSNN costs "<<inference_time/100<<" ms"<<endl;
    unsigned int outputSize = 1000;
    float *hOutput = (float *)malloc(outputSize*sizeof(float));
    hipMemcpy(hOutput,output,outputSize*sizeof(float),hipMemcpyDeviceToHost);
    vector<float> out_bin;
    for(int i=0;i<outputSize;++i){
        out_bin.push_back(hOutput[i]);
    }
    std::ofstream ofp("s.bin", std::ios::out | std::ios::binary);
    ofp.write(reinterpret_cast<const char*>(out_bin.data()), out_bin.size() * sizeof(float));
    return 0;
}
