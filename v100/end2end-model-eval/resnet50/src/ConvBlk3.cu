#include "../inc/common.h"
ConvBlk3::ConvBlk3(unsigned int b, unsigned int c, unsigned int h, unsigned int w, string conv0Weight, string bn0Weight,
                   string conv1Weight, string bn1Weight, string conv2Weight, string bn2Weight,string conv3Weight,string bn3Weight) {
    B = b;
    H = h;
    C = c;
    W = w;
    conv0.initialize(B,C,H,W,512,0,1,1,2,conv0Weight);
    conv1.initialize(B,C,H,W,128,0,1,1,2,conv1Weight);
    conv2.initialize(B,128,H/2,W/2,128,1,3,3,1,conv2Weight);
    conv3.initialize(B,128,H/2,W/2,512,0,1,1,1,conv3Weight);
    bn0.initialize(B,512,H/2,W/2,bn0Weight);
    bn1.initialize(B,128,H/2,W/2,bn1Weight);
    bn2.initialize(B,128,H/2,W/2,bn2Weight);
    bn3.initialize(B,512,H/2,W/2,bn3Weight);
    add.initialize(B,512,H/2,W/2);
    activation64.initialize(B,128,H/2,W/2);
    activation.initialize(B,512,H/2,W/2);
}
float * ConvBlk3::forward(float *input){
    float *block3_1_conv = conv1.forward(input);
    float *block3_1_bn = bn1.forward(block3_1_conv);
    float *block3_1_relu = activation64.forward(block3_1_bn);

    float *block3_2_conv = conv2.forward(block3_1_relu);
    float *block3_2_bn = bn2.forward(block3_2_conv);
    float *block3_2_relu = activation64.forward(block3_2_bn);

    float *block3_3_conv = conv3.forward(block3_2_relu);
    float *block3_3_bn = bn3.forward(block3_3_conv);

    float *block3_0_conv = conv0.forward(input);
    float *block3_0_bn = bn0.forward(block3_0_conv);

    float *block3_add = add.forward(block3_3_bn,block3_0_bn);
    float *block3_out = activation.forward(block3_add);
    return block3_out;
}
/*int main(void){
    ConvBlk3 a(1,256,56,56,"../weights/conv3_block1_0_conv.bin","../weights/conv3_block1_0_bn.bin",
                          "../weights/conv3_block1_1_conv.bin","../weights/conv3_block1_1_bn.bin",
                          "../weights/conv3_block1_2_conv.bin","../weights/conv3_block1_2_bn.bin",
                          "../weights/conv3_block1_3_conv.bin","../weights/conv3_block1_3_bn.bin");
    float *input = (float *)malloc(256*56*56*sizeof(float));
    for(int i=0;i<256*56*56;++i){
        input[i] = 1.0f;
    }
    float *dInput;
    hipMalloc(&dInput,256*56*56*sizeof(float));
    hipMemcpy(dInput,input,256*56*56*sizeof(float),hipMemcpyHostToDevice);
    float *dOutput = a.forward(dInput);
    float *output = (float *)malloc(512*28*28*sizeof(float));
    hipMemcpy(output,dOutput,512*28*28*sizeof(float),hipMemcpyDeviceToHost);
    float *pyOut = load_input("/home/lizhi/research/resnet152/convblk1.bin",64*56*256);
    cout<<output[0]<<endl;
    return 0;
}*/