#include "../inc/common.h"
#include "../inc/nvml_helper.hpp"
int main(int argc,char *argv[]){
    float *input = (float *)malloc(224*224*3*sizeof(float));
    string imagePath = argv[1];
    float *dInput;
    hipMalloc(&dInput,224*224*3*sizeof(float));
    Conv conv1_conv;
    conv1_conv.initialize(1,3,224,224,64,3,7,7,2,"../../../weights/resnet50/weights/conv1_conv.bin");
    BatchNorm conv1_bn;
    conv1_bn.initialize(1,64,112,112,"../../../weights/resnet50/weights/conv1_bn.bin");
    Activation conv1_relu;
    conv1_relu.initialize(1,64,112,112);
    Pool conv1_max_pool;
    conv1_max_pool.initialize(1,64,112,112,1,3,3,HIPDNN_POOLING_MAX,2);

    ConvBlk1 conv2_block1(1,64,56,56,"../../../weights/resnet50/weights/conv2_block1_0_conv.bin","../../../weights/resnet50/weights/conv2_block1_0_bn.bin",
                          "../../../weights/resnet50/weights/conv2_block1_1_conv.bin","../../../weights/resnet50/weights/conv2_block1_1_bn.bin",
                          "../../../weights/resnet50/weights/conv2_block1_2_conv.bin","../../../weights/resnet50/weights/conv2_block1_2_bn.bin",
                          "../../../weights/resnet50/weights/conv2_block1_3_conv.bin","../../../weights/resnet50/weights/conv2_block1_3_bn.bin");
    ConvBlk2 conv2_block2(1,256,56,56,"../../../weights/resnet50/weights/conv2_block2_1_conv.bin","../../../weights/resnet50/weights/conv2_block2_1_bn.bin",
                          "../../../weights/resnet50/weights/conv2_block2_2_conv.bin","../../../weights/resnet50/weights/conv2_block2_2_bn.bin",
                          "../../../weights/resnet50/weights/conv2_block2_3_conv.bin","../../../weights/resnet50/weights/conv2_block2_3_bn.bin");
    ConvBlk2 conv2_block3(1,256,56,56,"../../../weights/resnet50/weights/conv2_block3_1_conv.bin","../../../weights/resnet50/weights/conv2_block3_1_bn.bin",
                          "../../../weights/resnet50/weights/conv2_block3_2_conv.bin","../../../weights/resnet50/weights/conv2_block3_2_bn.bin",
                          "../../../weights/resnet50/weights/conv2_block3_3_conv.bin","../../../weights/resnet50/weights/conv2_block3_3_bn.bin");

    ConvBlk3 conv3_block1(1,256,56,56,"../../../weights/resnet50/weights/conv3_block1_0_conv.bin","../../../weights/resnet50/weights/conv3_block1_0_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block1_1_conv.bin","../../../weights/resnet50/weights/conv3_block1_1_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block1_2_conv.bin","../../../weights/resnet50/weights/conv3_block1_2_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block1_3_conv.bin","../../../weights/resnet50/weights/conv3_block1_3_bn.bin");
    ConvBlk4 conv3_block2(1,512,28,28,"../../../weights/resnet50/weights/conv3_block2_1_conv.bin","../../../weights/resnet50/weights/conv3_block2_1_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block2_2_conv.bin","../../../weights/resnet50/weights/conv3_block2_2_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block2_3_conv.bin","../../../weights/resnet50/weights/conv3_block2_3_bn.bin");
    ConvBlk4 conv3_block3(1,512,28,28,"../../../weights/resnet50/weights/conv3_block3_1_conv.bin","../../../weights/resnet50/weights/conv3_block3_1_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block3_2_conv.bin","../../../weights/resnet50/weights/conv3_block3_2_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block3_3_conv.bin","../../../weights/resnet50/weights/conv3_block3_3_bn.bin");
    ConvBlk4 conv3_block4(1,512,28,28,"../../../weights/resnet50/weights/conv3_block4_1_conv.bin","../../../weights/resnet50/weights/conv3_block4_1_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block4_2_conv.bin","../../../weights/resnet50/weights/conv3_block4_2_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block4_3_conv.bin","../../../weights/resnet50/weights/conv3_block4_3_bn.bin");

    ConvBlk5 conv4_block1(1,512,28,28,"../../../weights/resnet50/weights/conv4_block1_0_conv.bin","../../../weights/resnet50/weights/conv4_block1_0_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block1_1_conv.bin","../../../weights/resnet50/weights/conv4_block1_1_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block1_2_conv.bin","../../../weights/resnet50/weights/conv4_block1_2_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block1_3_conv.bin","../../../weights/resnet50/weights/conv4_block1_3_bn.bin");

    ConvBlk6 conv4_block2(1,1024,14,14,"../../../weights/resnet50/weights/conv4_block2_1_conv.bin","../../../weights/resnet50/weights/conv4_block2_1_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block2_2_conv.bin","../../../weights/resnet50/weights/conv4_block2_2_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block2_3_conv.bin","../../../weights/resnet50/weights/conv4_block2_3_bn.bin");
    ConvBlk6 conv4_block3(1,1024,14,14,"../../../weights/resnet50/weights/conv4_block3_1_conv.bin","../../../weights/resnet50/weights/conv4_block3_1_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block3_2_conv.bin","../../../weights/resnet50/weights/conv4_block3_2_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block3_3_conv.bin","../../../weights/resnet50/weights/conv4_block3_3_bn.bin");
    ConvBlk6 conv4_block4(1,1024,14,14,"../../../weights/resnet50/weights/conv4_block4_1_conv.bin","../../../weights/resnet50/weights/conv4_block4_1_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block4_2_conv.bin","../../../weights/resnet50/weights/conv4_block4_2_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block4_3_conv.bin","../../../weights/resnet50/weights/conv4_block4_3_bn.bin");
    ConvBlk6 conv4_block5(1,1024,14,14,"../../../weights/resnet50/weights/conv4_block5_1_conv.bin","../../../weights/resnet50/weights/conv4_block5_1_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block5_2_conv.bin","../../../weights/resnet50/weights/conv4_block5_2_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block5_3_conv.bin","../../../weights/resnet50/weights/conv4_block5_3_bn.bin");
    ConvBlk6 conv4_block6(1,1024,14,14,"../../../weights/resnet50/weights/conv4_block6_1_conv.bin","../../../weights/resnet50/weights/conv4_block6_1_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block6_2_conv.bin","../../../weights/resnet50/weights/conv4_block6_2_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block6_3_conv.bin","../../../weights/resnet50/weights/conv4_block6_3_bn.bin");

    ConvBlk7 conv5_block1(1,1024,14,14,"../../../weights/resnet50/weights/conv5_block1_0_conv.bin","../../../weights/resnet50/weights/conv5_block1_0_bn.bin",
                          "../../../weights/resnet50/weights/conv5_block1_1_conv.bin","../../../weights/resnet50/weights/conv5_block1_1_bn.bin",
                          "../../../weights/resnet50/weights/conv5_block1_2_conv.bin","../../../weights/resnet50/weights/conv5_block1_2_bn.bin",
                          "../../../weights/resnet50/weights/conv5_block1_3_conv.bin",
                          "../../../weights/resnet50/weights/conv5_block1_3_bn.bin");
    ConvBlk8 conv5_block2(1,2048,7,7,"../../../weights/resnet50/weights/conv5_block2_1_conv.bin",
                          "../../../weights/resnet50/weights/conv5_block2_1_bn.bin",
                          "../../../weights/resnet50/weights/conv5_block2_2_conv.bin","../../../weights/resnet50/weights/conv5_block2_2_bn.bin",
                          "../../../weights/resnet50/weights/conv5_block2_3_conv.bin","../../../weights/resnet50/weights/conv5_block2_3_bn.bin");
    ConvBlk8 conv5_block3(1,2048,7,7,"../../../weights/resnet50/weights/conv5_block3_1_conv.bin","../../../weights/resnet50/weights/conv5_block3_1_bn.bin",
                          "../../../weights/resnet50/weights/conv5_block3_2_conv.bin","../../../weights/resnet50/weights/conv5_block3_2_bn.bin",
                          "../../../weights/resnet50/weights/conv5_block3_3_conv.bin",
                          "../../../weights/resnet50/weights/conv5_block3_3_bn.bin");
    Pool avg_pool;
    avg_pool.initialize(1,2048,7,7,0,7,7,HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING,1);
    Conv predict;
    predict.initialize(1,2048,1,1,1000,0,1,1,1,"../../../weights/resnet50/weights/predictions.bin");

    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    hipMemcpy(dInput, input, 224 * 224 * 3 * sizeof(float), hipMemcpyHostToDevice);
    float *output;
    output = conv1_conv.forward(dInput);
    output = conv1_bn.forward(output);
    output = conv1_relu.forward(output);
    output = conv1_max_pool.forward(output);
    output = conv2_block1.forward(output);
    output = conv2_block2.forward(output);
    output = conv2_block3.forward(output);
    output = conv3_block1.forward(output);
    output = conv3_block2.forward(output);
    output = conv3_block3.forward(output);
    output = conv3_block4.forward(output);
    output = conv4_block1.forward(output);
    output = conv4_block2.forward(output);
    output = conv4_block3.forward(output);
    output = conv4_block4.forward(output);
    output = conv4_block5.forward(output);
    output = conv4_block6.forward(output);
    output = conv5_block1.forward(output);
    output = conv5_block2.forward(output);
    output = conv5_block3.forward(output);
    output = avg_pool.forward(output);
    output = predict.forward(output);
    hipDeviceSynchronize();
    NVML_INIT;
    NVML_DEV_t dev0;
    NVML_GET_HANDLE(0, &dev0);
    unsigned long long start,end;
    unsigned long long energy = 0;
    float inference_time = 0.0f;
    for(int i=0;i<100;++i){
        string image = imagePath + "/" + to_string(i) + ".bin";
        load_input(image, 3 * 224 * 224, input);
        hipMemcpy(dInput,input,224*224*3*sizeof(float),hipMemcpyHostToDevice);
        hipEventRecord(event_start);
        output = conv1_conv.forward(dInput);
        output = conv1_bn.forward(output);
        output = conv1_relu.forward(output);
        output = conv1_max_pool.forward(output);
        output = conv2_block1.forward(output);
        output = conv2_block2.forward(output);
        output = conv2_block3.forward(output);
        output = conv3_block1.forward(output);
        output = conv3_block2.forward(output);
        output = conv3_block3.forward(output);
        output = conv3_block4.forward(output);
        output = conv4_block1.forward(output);
        output = conv4_block2.forward(output);
        output = conv4_block3.forward(output);
        output = conv4_block4.forward(output);
        output = conv4_block5.forward(output);
        output = conv4_block6.forward(output);
        output = conv5_block1.forward(output);
        output = conv5_block2.forward(output);
        output = conv5_block3.forward(output);
        output = avg_pool.forward(output);
        output = predict.forward(output);
        hipDeviceSynchronize();
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<"resnet50 cuDNN costs "<<inference_time/100<<" ms"<<endl;
    cout<<endl;
    unsigned int outputSize = 1000;
    float *hOutput = (float *)malloc(outputSize*sizeof(float));
    hipMemcpy(hOutput,output,outputSize*sizeof(float),hipMemcpyDeviceToHost);
    vector<float> out_bin;
    for(int i=0;i<outputSize;++i){
        out_bin.push_back(hOutput[i]);
    }
    std::ofstream ofp("d.bin", std::ios::out | std::ios::binary);
    ofp.write(reinterpret_cast<const char*>(out_bin.data()), out_bin.size() * sizeof(float));
    return 0;
}
