#include "hip/hip_runtime.h"
class Conv56_64_64{
public:
    __global__ void transform(float *matrix, float *d_nnz, char *ids, unsigned int *c_lens);
    __global__ void conv2d()
};
#define SC 64
#define SN 64
#define SH 56
#define SW 56
#define R 3
#define S 3
#define BLK_DIM 64
#define TC 4
#define TCS ((SC-1)/TC+1)
#define WARPS_CONV ((BLK_DIM-1)/32+1)
#define TH 4
#define TW 4
#define TILES_EACH_CHANNEL (((SHo-1)/TH+1)*((SWo-1)/TW+1))

__global__ void transform(float *matrix, float *d_nnz, char *ids, unsigned int *c_lens){
    unsigned int global_id = blockIdx.x * BLK_DIM_TRANS + threadIdx.x;
    unsigned int batch_id = global_id/(SC*SH*SW);
    if(global_id >= SB*SH*SW*SC){
        return ;
    }
    const float v = matrix[global_id];
    if(v == 0.0f){
        return;
    }
    unsigned int c = global_id % SC;

    int h = ((global_id - batch_id * SC * SH * SW)/SC)/SW+1;
    int w = ((global_id - batch_id * SC * SH * SW)/SC)%SW+1;
    int th_start = min(h/TH,THS-1);
    int tw_start = min(w/TW,TWS-1);
    for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
        if((tile_h_id*TH+TH+R-1)<=h){
            break;
        }
        for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
            if((tile_w_id*TW+TW+S-1)<=w){
                break;
            }
            unsigned int tile_id = tile_h_id * TWS + tile_w_id;
            unsigned int index = atomicAdd(&c_lens[batch_id*SC*TILES_EACH_CHANNEL+c*TILES_EACH_CHANNEL+tile_id],1);
            unsigned int abs_h = h - tile_h_id*TH;
            unsigned int abs_w = w - tile_w_id*TW;
            d_nnz[batch_id*SC*TILES_EACH_CHANNEL*(TH+R-1)*(TW+S-1)+
                  c*TILES_EACH_CHANNEL*(TH+R-1)*(TW+S-1)+tile_id*(TH+R-1)*(TW+S-1)+index] = v;
            ids[batch_id*SC*TILES_EACH_CHANNEL*(TH+R-1)*(TW+S-1)+
                c*TILES_EACH_CHANNEL*(TH+R-1)*(TW+S-1)+tile_id*(TH+R-1)*(TW+S-1)+index] = abs_h*(TW+S-1)+abs_w;
        }
    }
}
__device__ void load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<R;++r){
        for(unsigned int s=0;s<S;++s){
            data_array[r*S+s] = kernel[c_index*SN*R*S+r*S*SN+s*SN+n_id];
        }
    }
}
__device__ void switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
    switch (switch_condition) {
        case 0:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(0-r)*4+(0-s)] += result;
                }
            }
            break;
        case 1:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(0-r)*4+(1-s)] += result;
                }
            }
            break;
        case 2:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(0-r)*4+(2-s)] += result;
                }
            }
            break;
        case 3:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(0-r)*4+(3-s)] += result;
                }
            }
            break;
        case 4:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(0-r)*4+(4-s)] += result;
                }
            }
            break;
        case 5:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(0-r)*4+(5-s)] += result;
                }
            }
            break;
        case 6:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(1-r)*4+(0-s)] += result;
                }
            }
            break;
        case 7:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(1-r)*4+(1-s)] += result;
                }
            }
            break;
        case 8:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(1-r)*4+(2-s)] += result;
                }
            }
            break;
        case 9:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(1-r)*4+(3-s)] += result;
                }
            }
            break;
        case 10:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(1-r)*4+(4-s)] += result;
                }
            }
            break;
        case 11:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(1-r)*4+(5-s)] += result;
                }
            }
            break;
        case 12:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(2-r)*4+(0-s)] += result;
                }
            }
            break;
        case 13:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(2-r)*4+(1-s)] += result;
                }
            }
            break;
        case 14:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(2-r)*4+(2-s)] += result;
                }
            }
            break;
        case 15:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(2-r)*4+(3-s)] += result;
                }
            }
            break;
        case 16:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(2-r)*4+(4-s)] += result;
                }
            }
            break;
        case 17:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(2-r)*4+(5-s)] += result;
                }
            }
            break;
        case 18:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(3-r)*4+(0-s)] += result;
                }
            }
            break;
        case 19:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(3-r)*4+(1-s)] += result;
                }
            }
            break;
        case 20:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(3-r)*4+(2-s)] += result;
                }
            }
            break;
        case 21:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(3-r)*4+(3-s)] += result;
                }
            }
            break;
        case 22:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(3-r)*4+(4-s)] += result;
                }
            }
            break;
        case 23:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(3-r)*4+(5-s)] += result;
                }
            }
            break;
        case 24:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(4-r)*4+(0-s)] += result;
                }
            }
            break;
        case 25:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(4-r)*4+(1-s)] += result;
                }
            }
            break;
        case 26:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(4-r)*4+(2-s)] += result;
                }
            }
            break;
        case 27:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(4-r)*4+(3-s)] += result;
                }
            }
            break;
        case 28:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(4-r)*4+(4-s)] += result;
                }
            }
            break;
        case 29:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(4-r)*4+(5-s)] += result;
                }
            }
            break;
        case 30:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(5-r)*4+(0-s)] += result;
                }
            }
            break;
        case 31:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(5-r)*4+(1-s)] += result;
                }
            }
            break;
        case 32:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(5-r)*4+(2-s)] += result;
                }
            }
            break;
        case 33:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(5-r)*4+(3-s)] += result;
                }
            }
            break;
        case 34:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(5-r)*4+(4-s)] += result;
                }
            }
            break;
        case 35:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*S+s];
                    temp_result[(5-r)*4+(5-s)] += result;
                }
            }
            break;

    }
}
__device__ void load_input_2_shared_memory(float *values,unsigned int *c_lens,char *ids,
                                           float *shared_input,char *shared_ids,unsigned int *shared_lens,
                                           unsigned int warp_id,unsigned int lane_id,unsigned int batch_id,
                                           unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<TC&&tile_c_id+c_id<SC;c_id+=WARPS_CONV){
        unsigned int end_index = c_lens[batch_id*SC*TILES_EACH_CHANNEL+(tile_c_id+c_id)*TILES_EACH_CHANNEL+tile_id];
        if(lane_id ==0){
            shared_lens[c_id] = end_index;
        }
        for(unsigned int id = lane_id;id<end_index;id+=32){
            shared_input[c_id*(TH+R-1)*(TW+S-1)+id] = values[batch_id*SC*TILES_EACH_CHANNEL*(TH+R-1)*(TW+S-1)+
                                                             (tile_c_id+c_id)*TILES_EACH_CHANNEL*(TH+R-1)*(TW+S-1)+tile_id*(TH+R-1)*(TW+S-1)+id];
            shared_ids[c_id*(TH+R-1)*(TW+S-1)+id] = ids[batch_id*SC*TILES_EACH_CHANNEL*(TH+R-1)*(TW+S-1)+
                                                        (tile_c_id+c_id)*TILES_EACH_CHANNEL*(TH+R-1)*(TW+S-1)+tile_id*(TH+R-1)*(TW+S-1)+id];
        }
    }
}
__global__ void conv2d(float * __restrict__ values, unsigned int * __restrict__ c_lens,
                       char * __restrict__ ids,
                       const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[TC*(TH+R-1)*(TW+S-1)];
    __shared__ char input_ids[TC*(TH+R-1)*(TW+S-1)];
    __shared__ unsigned int channel_lens[(TC)];

    const unsigned int batch_id = (blockIdx.x/(TCS*TILES_EACH_CHANNEL));
    const unsigned int t_id = (blockIdx.x - batch_id*TCS*TILES_EACH_CHANNEL)/TCS;
    const unsigned int tile_h_id = (t_id / TWS)*TH;
    const unsigned int tile_w_id = (t_id % TWS)*TW;
    const unsigned int index = blockIdx.x % (TCS);
    const unsigned int start_channel_index = index*TC;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    load_input_2_shared_memory(values,c_lens,ids,input,input_ids,channel_lens,warp_id,lane_id,batch_id,t_id,start_channel_index);
    __syncthreads();
    float v;
    unsigned int id;
    for(unsigned int n = threadIdx.x;n<SN;n+=BLK_DIM){
        for(unsigned int c=start_channel_index;c<start_channel_index+TC&&c<SC;c++){
            unsigned int abs_c = c - start_channel_index;
            unsigned int start_index = abs_c*(TH+R-1)*(TW+S-1);
            unsigned int end_index = start_index+channel_lens[abs_c];
            if(start_index == end_index){
                continue;
            }
            load_data_2_register(data_array,(c),kernel,n);
            unsigned int iters = end_index - start_index;
            for(unsigned int iter=0;iter<iters;iter++) {
                v = input[iter+start_index];
                id = input_ids[iter+start_index];
                switch_function(id,data_array,v,temp_result);
            }
        }
        for (unsigned int th = 0; th < TH; ++th) {
            for (unsigned int tw = 0; tw < TW; ++tw) {
                if (tile_h_id + th >= SHo || tile_w_id + tw >= SWo) {
                    continue;
                }
                atomicAdd(&outputs[batch_id * SN * SHo * SWo + (tile_h_id + th) * SWo * SN + (tile_w_id + tw) * SN +
                                   n],temp_result[(th * TW + tw)]);
            }
        }
        for(unsigned int i=0;i<TH*TW;++i){
            temp_result[i] = 0.0f;
        }
    }
}







