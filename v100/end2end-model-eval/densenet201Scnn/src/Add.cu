#include "../inc/cudnnOps.h"
void Add::initialize(unsigned int b, unsigned int c, unsigned int h, unsigned int w) {
    B = b;
    C = c;
    H = h;
    W = w;
    checkCUDNN(hipdnnCreate(&addCudnn));
    checkCUDNN(hipdnnCreateTensorDescriptor(&addInputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(addInputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
    checkCUDNN(hipdnnCreateTensorDescriptor(&addOutputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(addOutputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
}
float *Add::forward(float *x, float *y) {
    hipdnnAddTensor(addCudnn,&alpha,addInputDescriptor,x,&beta,addOutputDescriptor,y);
    return y;
}