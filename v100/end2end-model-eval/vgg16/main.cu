
#include "./inc/conv_and_pool.h"
#include "./inc/nvml_helper.hpp"
int main(int argc, char *argv[]){
    float *input = (float *)malloc(224*224*3*sizeof(float));
    string imagePath = argv[1];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    float *dInput;
    hipMalloc(&dInput,224*224*3*sizeof(float));

    Conv_and_pool block1(1,3,224,224,64,64,"../../../weights/vgg16/weights/block1_conv1.bin",
                         "../../../weights/vgg16/weights/block1_conv2.bin");
    Conv_and_pool block2(1,64,112,112,128,128,"../../../weights/vgg16/weights/block2_conv1.bin",
                         "../../../weights/vgg16/weights/block2_conv2.bin");
    Conv_and_pool3 block3(1,128,56,56,256,256,256,
                          "../../../weights/vgg16/weights/block3_conv1.bin",
                          "../../../weights/vgg16/weights/block3_conv2.bin",
                          "../../../weights/vgg16/weights/block3_conv3.bin");
    Conv_and_pool3 block4(1,256,28,28,512,512,512,
                          "../../../weights/vgg16/weights/block4_conv1.bin",
                          "../../../weights/vgg16/weights/block4_conv2.bin",
                          "../../../weights/vgg16/weights/block4_conv3.bin");
    Conv_and_pool3 block5(1,512,14,14,512,512,512,
                          "../../../weights/vgg16/weights/block5_conv1.bin",
                          "../../../weights/vgg16/weights/block5_conv2.bin",
                          "../../../weights/vgg16/weights/block5_conv3.bin");
    FC fc1;
    fc1.initialize(1,25088,1,1,4096,0,1,1,1,"../../../weights/vgg16/weights/fc1.bin");
    FC fc2;
    fc2.initialize(1,4096,1,1,4096,0,1,1,1,"../../../weights/vgg16/weights/fc2.bin");
    FC fc3;
    fc3.initialize(1,4096,1,1,1000,0,1,1,1,"../../../weights/vgg16/weights/predictions.bin");
    hipMemcpy(dInput,input,224*224*3*sizeof(float),hipMemcpyHostToDevice);
    float *output;
    output = block1.forward(dInput);
    output = block2.forward(output);
    output = block3.forward(output);
    output = block4.forward(output);
    output = block5.forward(output);
    output = fc1.forward(output);
    output = fc2.forward(output);
    output = fc3.forward(output);
    hipDeviceSynchronize();

    NVML_INIT;
    NVML_DEV_t dev0;
    NVML_GET_HANDLE(0, &dev0);
    unsigned long long start,end;
    unsigned long long energy = 0;
    float inference_time = 0.0f;
    for(int i=0;i<100;++i){
        string image = imagePath + "/" + to_string(i) + ".bin";
        load_input(image, 3 * 224 * 224, input);
        hipMemcpy(dInput,input,224*224*3*sizeof(float),hipMemcpyHostToDevice);
        hipEventRecord(event_start);
        output = block1.forward(dInput);
        output = block2.forward(output);
        output = block3.forward(output);
        output = block4.forward(output);
        output = block5.forward(output);
        output = fc1.forward(output);
        output = fc2.forward(output);
        output = fc3.forward(output);
        hipDeviceSynchronize();
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<"vgg16 cuDNN costs "<<inference_time/100<<" ms"<<endl;
    cout<<endl;
    unsigned int outputSize = 1000;
    float *hOutput = (float *)malloc(outputSize*sizeof(float));
    hipMemcpy(hOutput,output,outputSize*sizeof(float),hipMemcpyDeviceToHost);
    vector<float> out_bin;
    for(int i=0;i<outputSize;++i){
        out_bin.push_back(hOutput[i]);
    }
    std::ofstream ofp("d.bin", std::ios::out | std::ios::binary);
    ofp.write(reinterpret_cast<const char*>(out_bin.data()), out_bin.size() * sizeof(float));
    return 0;
}
