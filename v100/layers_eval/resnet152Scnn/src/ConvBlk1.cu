#include "../inc/convBlk.h"
#define MEASUE_CUSNN true
ConvBlk1::ConvBlk1(unsigned int b, unsigned int c, unsigned int h, unsigned int w, string conv0Weight, string bn0Weight,
                   string conv1Weight, string bn1Weight, string conv2Weight,
                   string bn2Weight,string conv3Weight,string bn3Weight, bool sparse,int index, float *t1, float *t2){
    B = b;
    H = h;
    C = c;
    W = w;
    conv0.initialize(B,C,H,W,256,0,1,1,1,conv0Weight);
    conv1.initialize(B,C,H,W,64,0,1,1,1,conv1Weight);
    conv2.initialize(B,C,H,W,64,1,3,3,1,conv2Weight);
    conv2_cudnn.initialize(B,C,H,W,64,1,3,3,1,conv2Weight);
    conv3.initialize(B,C,H,W,256,0,1,1,1,conv3Weight);
    bn0.initialize(B,256,H,W,bn0Weight);
    bn1.initialize(B,64,H,W,bn1Weight);
    bn2.initialize(B,64,H,W,bn2Weight);
    bn3.initialize(B,256,H,W,bn3Weight);
    add.initialize(B,256,H,W);
    activation64.initialize(B,64,H,W);
    activation64_cudnn.initialize(B,64,H,W);
    activation.initialize(B,256,H,W);
    this->sparse = sparse;
    this->index = index;
    this->t1 = t1;
    this->t2 = t2;

}
float * ConvBlk1::forward(float *input){
    float *block1_1_conv = conv1.forward(input);
    float *block1_1_bn = bn1.forward(block1_1_conv);
    float *block1_1_relu;
    float *block1_2_conv;
    if(sparse){
        block1_1_relu = activation64.forward(block1_1_bn);
        hipEvent_t event_start;
        hipEvent_t event_stop;
        hipEventCreate(&event_start);
        hipEventCreate(&event_stop);

        float temp_time;
        if(MEASUE_CUSNN){
            hipEventRecord(event_start);
            block1_2_conv = conv2.forward(block1_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t1[index - 1] +=temp_time;
        }else{
            hipEventRecord(event_start);
            block1_2_conv = conv2_cudnn.forward(block1_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t2[index - 1] +=temp_time;
        }
    }else{
        block1_1_relu = activation64_cudnn.forward(block1_1_bn);
        block1_2_conv = conv2_cudnn.forward(block1_1_relu);
    }

    float *block1_2_bn = bn2.forward(block1_2_conv);
    float *block1_2_relu = activation64_cudnn.forward(block1_2_bn);

    float *block1_3_conv = conv3.forward(block1_2_relu);
    float *block1_3_bn = bn3.forward(block1_3_conv);

    float *block1_0_conv = conv0.forward(input);
    float *block1_0_bn = bn0.forward(block1_0_conv);

    float *block1_add = add.forward(block1_3_bn,block1_0_bn);
    float *relu = activation.forward(block1_add);
    return relu;
}
ConvBlk2::ConvBlk2(unsigned int b, unsigned int c, unsigned int h, unsigned int w,string conv1Weight, string bn1Weight,
                   string conv2Weight, string bn2Weight,string conv3Weight,string bn3Weight, bool sparse,int index, float *t1, float *t2) {
    B = b;
    H = h;
    C = c;
    W = w;
    conv1.initialize(B,C,H,W,64,0,1,1,1,conv1Weight);
    conv2.initialize(B,conv1.N,conv1.hOut,conv1.wOut,64,1,3,3,1,conv2Weight);
    conv2_cudnn.initialize(B,conv1.N,conv1.hOut,conv1.wOut,64,1,3,3,1,conv2Weight);
    conv3.initialize(B,conv2.N,conv2.hOut,conv2.wOut,256,0,1,1,1,conv3Weight);
    bn1.initialize(B,64,conv1.hOut,conv1.wOut,bn1Weight);
    bn2.initialize(B,64,conv2.hOut,conv2.wOut,bn2Weight);
    bn3.initialize(B,256,conv3.hOut,conv3.wOut,bn3Weight);
    add.initialize(B,256,H,W);
    activation64.initialize(B,64,conv1.hOut,conv1.wOut);
    activation64_cudnn.initialize(B,64,conv1.hOut,conv1.wOut);
    activation.initialize(B,256,conv3.hOut,conv3.wOut);
    this->sparse = sparse;
    this->index = index;
    this->t1 = t1;
    this->t2 = t2;

}
float * ConvBlk2::forward(float *input){
    float *block2_1_conv = conv1.forward(input);
    float *block2_1_bn = bn1.forward(block2_1_conv);
    float *block2_1_relu;
    float *block2_2_conv;
    if(sparse){
        block2_1_relu = activation64.forward(block2_1_bn);
        hipEvent_t event_start;
        hipEvent_t event_stop;
        hipEventCreate(&event_start);
        hipEventCreate(&event_stop);

        float temp_time;
        if(MEASUE_CUSNN){
            hipEventRecord(event_start);
            block2_2_conv = conv2.forward(block2_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t1[index - 1] +=temp_time;
        }else{
            hipEventRecord(event_start);
            block2_2_conv = conv2_cudnn.forward(block2_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t2[index - 1] +=temp_time;
        }

    }else{
        block2_1_relu = activation64_cudnn.forward(block2_1_bn);
        block2_2_conv = conv2_cudnn.forward(block2_1_relu);
    }
    float *block2_2_bn = bn2.forward(block2_2_conv);
    float *block2_2_relu = activation64_cudnn.forward(block2_2_bn);

    float *block2_3_conv = conv3.forward(block2_2_relu);
    float *block2_3_bn = bn3.forward(block2_3_conv);

    float *block2_add = add.forward(block2_3_bn,input);
    float *block2_out = activation.forward(block2_add);
    return block2_out;
}
ConvBlk3::ConvBlk3(unsigned int b, unsigned int c, unsigned int h, unsigned int w, string conv0Weight, string bn0Weight,
                   string conv1Weight, string bn1Weight, string conv2Weight,
                   string bn2Weight,string conv3Weight,string bn3Weight, bool sparse,int index, float *t1, float *t2) {
    B = b;
    H = h;
    C = c;
    W = w;
    conv0.initialize(B,C,H,W,512,0,1,1,2,conv0Weight);
    conv1.initialize(B,C,H,W,128,0,1,1,2,conv1Weight);
    conv2.initialize(B,128,H/2,W/2,128,1,3,3,1,conv2Weight);
    conv2_cudnn.initialize(B,128,H/2,W/2,128,1,3,3,1,conv2Weight);
    conv3.initialize(B,128,H/2,W/2,512,0,1,1,1,conv3Weight);
    bn0.initialize(B,512,H/2,W/2,bn0Weight);
    bn1.initialize(B,128,H/2,W/2,bn1Weight);
    bn2.initialize(B,128,H/2,W/2,bn2Weight);
    bn3.initialize(B,512,H/2,W/2,bn3Weight);
    add.initialize(B,512,H/2,W/2);
    activation64.initialize(B,128,H/2,W/2);
    activation64_cudnn.initialize(B,128,H/2,W/2);
    activation.initialize(B,512,H/2,W/2);
    this->sparse = sparse;
    this->index = index;
    this->t1 = t1;
    this->t2 = t2;

}
float * ConvBlk3::forward(float *input){
    float *block3_1_conv = conv1.forward(input);
    float *block3_1_bn = bn1.forward(block3_1_conv);
    float *block3_1_relu;
    float *block3_2_conv;
    if(sparse){
        block3_1_relu = activation64.forward(block3_1_bn);
        hipEvent_t event_start;
        hipEvent_t event_stop;
        hipEventCreate(&event_start);
        hipEventCreate(&event_stop);
        float temp_time;
        if(MEASUE_CUSNN){
            hipEventRecord(event_start);
            block3_2_conv = conv2.forward(block3_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t1[index - 1] +=temp_time;
        }else{
            hipEventRecord(event_start);
            block3_2_conv = conv2_cudnn.forward(block3_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t2[index - 1] +=temp_time;
        }
    }else{
        block3_1_relu = activation64_cudnn.forward(block3_1_bn);
        block3_2_conv = conv2_cudnn.forward(block3_1_relu);
    }
    float *block3_2_bn = bn2.forward(block3_2_conv);
    float *block3_2_relu = activation64_cudnn.forward(block3_2_bn);

    float *block3_3_conv = conv3.forward(block3_2_relu);
    float *block3_3_bn = bn3.forward(block3_3_conv);

    float *block3_0_conv = conv0.forward(input);
    float *block3_0_bn = bn0.forward(block3_0_conv);

    float *block3_add = add.forward(block3_3_bn,block3_0_bn);
    float *block3_out = activation.forward(block3_add);
    return block3_out;
}
ConvBlk4::ConvBlk4(unsigned int b, unsigned int c, unsigned int h, unsigned int w,string conv1Weight, string bn1Weight,
                   string conv2Weight, string bn2Weight,string conv3Weight,string bn3Weight, bool sparse,int index, float *t1, float *t2) {
    B = b;
    H = h;
    C = c;
    W = w;
    conv1.initialize(B,C,H,W,128,0,1,1,1,conv1Weight);
    conv2.initialize(B,128,H,W,128,1,3,3,1,conv2Weight);
    conv2_cudnn.initialize(B,128,H,W,128,1,3,3,1,conv2Weight);
    conv3.initialize(B,128,H,W,512,0,1,1,1,conv3Weight);
    bn1.initialize(B,128,H,W,bn1Weight);
    bn2.initialize(B,128,H,W,bn2Weight);
    bn3.initialize(B,512,H,W,bn3Weight);
    add.initialize(B,512,H,W);
    activation64.initialize(B,128,H,W);
    activation64_cudnn.initialize(B,128,H,W);
    activation.initialize(B,512,H,W);
    this->sparse = sparse;
    this->index = index;
    this->t1 = t1;
    this->t2 = t2;

}
float * ConvBlk4::forward(float *input){
    float *block2_1_conv = conv1.forward(input);
    float *block2_1_bn = bn1.forward(block2_1_conv);
    float *block2_1_relu;
    float *block2_2_conv;
    if(sparse){
        block2_1_relu = activation64.forward(block2_1_bn);
        hipEvent_t event_start;
        hipEvent_t event_stop;
        hipEventCreate(&event_start);
        hipEventCreate(&event_stop);

        float temp_time;
        if(MEASUE_CUSNN){
            hipEventRecord(event_start);
            block2_2_conv = conv2.forward(block2_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t1[index - 1] +=temp_time;
        }else{
            hipEventRecord(event_start);
            block2_2_conv = conv2_cudnn.forward(block2_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t2[index - 1] +=temp_time;
        }

    }else{
        block2_1_relu = activation64_cudnn.forward(block2_1_bn);
        block2_2_conv = conv2_cudnn.forward(block2_1_relu);
    }
    float *block2_2_bn = bn2.forward(block2_2_conv);
    float *block2_2_relu = activation64_cudnn.forward(block2_2_bn);

    float *block2_3_conv = conv3.forward(block2_2_relu);
    float *block2_3_bn = bn3.forward(block2_3_conv);

    float *block2_add = add.forward(block2_3_bn,input);
    float *block2_out = activation.forward(block2_add);
    return block2_out;
}
ConvBlk5::ConvBlk5(unsigned int b, unsigned int c, unsigned int h, unsigned int w, string conv0Weight, string bn0Weight,
                   string conv1Weight, string bn1Weight, string conv2Weight, string bn2Weight,
                   string conv3Weight,string bn3Weight, bool sparse,int index, float *t1, float *t2) {
    B = b;
    H = h;
    C = c;
    W = w;
    conv0.initialize(B,C,H,W,1024,0,1,1,2,conv0Weight);
    conv1.initialize(B,C,H,W,256,0,1,1,2,conv1Weight);
    conv2.initialize(B,256,H/2,W/2,256,1,3,3,1,conv2Weight);
    conv2_cudnn.initialize(B,256,H/2,W/2,256,1,3,3,1,conv2Weight);
    conv3.initialize(B,256,H/2,W/2,1024,0,1,1,1,conv3Weight);
    bn0.initialize(B,1024,H/2,W/2,bn0Weight);
    bn1.initialize(B,256,H/2,W/2,bn1Weight);
    bn2.initialize(B,256,H/2,W/2,bn2Weight);
    bn3.initialize(B,1024,H/2,W/2,bn3Weight);
    add.initialize(B,1024,H/2,W/2);
    activation64.initialize(B,256,H/2,W/2);
    activation64_cudnn.initialize(B,256,H/2,W/2);
    activation.initialize(B,1024,H/2,W/2);
    this->sparse = sparse;
    this->index = index;
    this->t1 = t1;
    this->t2 = t2;

}
float * ConvBlk5::forward(float *input){
    float *block3_1_conv = conv1.forward(input);
    float *block3_1_bn = bn1.forward(block3_1_conv);
    float *block3_1_relu;
    float *block3_2_conv;
    if(sparse){
        block3_1_relu = activation64.forward(block3_1_bn);
        hipEvent_t event_start;
        hipEvent_t event_stop;
        hipEventCreate(&event_start);
        hipEventCreate(&event_stop);

        float temp_time;
        if(MEASUE_CUSNN){
            hipEventRecord(event_start);
            block3_2_conv = conv2.forward(block3_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t1[index - 1] +=temp_time;
        }else{
            hipEventRecord(event_start);
            block3_2_conv = conv2_cudnn.forward(block3_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t2[index - 1] +=temp_time;
        }

    }else{
        block3_1_relu = activation64_cudnn.forward(block3_1_bn);
        block3_2_conv = conv2_cudnn.forward(block3_1_relu);
    }
    float *block3_2_bn = bn2.forward(block3_2_conv);
    float *block3_2_relu = activation64_cudnn.forward(block3_2_bn);

    float *block3_3_conv = conv3.forward(block3_2_relu);
    float *block3_3_bn = bn3.forward(block3_3_conv);

    float *block3_0_conv = conv0.forward(input);
    float *block3_0_bn = bn0.forward(block3_0_conv);

    float *block3_add = add.forward(block3_3_bn,block3_0_bn);
    float *block3_out = activation.forward(block3_add);
    return block3_out;
}
ConvBlk6::ConvBlk6(unsigned int b, unsigned int c, unsigned int h, unsigned int w,string conv1Weight, string bn1Weight,
                   string conv2Weight, string bn2Weight,string conv3Weight,string bn3Weight, bool sparse,int index, float *t1, float *t2) {
    B = b;
    H = h;
    C = c;
    W = w;
    conv1.initialize(B,C,H,W,256,0,1,1,1,conv1Weight);
    conv2.initialize(B,256,H,W,256,1,3,3,1,conv2Weight);
    conv2_cudnn.initialize(B,256,H,W,256,1,3,3,1,conv2Weight);
    conv3.initialize(B,256,H,W,1024,0,1,1,1,conv3Weight);
    bn1.initialize(B,256,H,W,bn1Weight);
    bn2.initialize(B,256,H,W,bn2Weight);
    bn3.initialize(B,1024,H,W,bn3Weight);
    add.initialize(B,1024,H,W);
    activation64.initialize(B,256,H,W);
    activation64_cudnn.initialize(B,256,H,W);
    activation.initialize(B,1024,H,W);
    this->sparse = sparse;
    this->index = index;
    this->t1 = t1;
    this->t2 = t2;

}
float * ConvBlk6::forward(float *input){
    float *block2_1_conv = conv1.forward(input);
    float *block2_1_bn = bn1.forward(block2_1_conv);
    float *block2_1_relu;
    float *block2_2_conv;
    if(sparse){
        block2_1_relu = activation64.forward(block2_1_bn);
        hipEvent_t event_start;
        hipEvent_t event_stop;
        hipEventCreate(&event_start);
        hipEventCreate(&event_stop);

        float temp_time;
        if(MEASUE_CUSNN){
            hipEventRecord(event_start);
            block2_2_conv = conv2.forward(block2_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t1[index - 1] +=temp_time;
        }else{
            hipEventRecord(event_start);
            block2_2_conv = conv2_cudnn.forward(block2_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t2[index - 1] +=temp_time;
        }
        /**/

    }else{
        block2_1_relu = activation64_cudnn.forward(block2_1_bn);
        block2_2_conv = conv2_cudnn.forward(block2_1_relu);
    }
    float *block2_2_bn = bn2.forward(block2_2_conv);
    float *block2_2_relu = activation64_cudnn.forward(block2_2_bn);

    float *block2_3_conv = conv3.forward(block2_2_relu);
    float *block2_3_bn = bn3.forward(block2_3_conv);

    float *block2_add = add.forward(block2_3_bn,input);
    float *block2_out = activation.forward(block2_add);
    return block2_out;
}
ConvBlk7::ConvBlk7(unsigned int b, unsigned int c, unsigned int h, unsigned int w, string conv0Weight, string bn0Weight,
                   string conv1Weight, string bn1Weight, string conv2Weight,
                   string bn2Weight,string conv3Weight,string bn3Weight, bool sparse,int index, float *t1, float *t2) {
    B = b;
    H = h;
    C = c;
    W = w;
    conv0.initialize(B,C,H,W,2048,0,1,1,2,conv0Weight);
    conv1.initialize(B,C,H,W,512,0,1,1,2,conv1Weight);
    conv2.initialize(B,512,H/2,W/2,512,1,3,3,1,conv2Weight);
    conv2_cudnn.initialize(B,512,H/2,W/2,512,1,3,3,1,conv2Weight);
    conv3.initialize(B,512,H/2,W/2,2048,0,1,1,1,conv3Weight);
    bn0.initialize(B,2048,H/2,W/2,bn0Weight);
    bn1.initialize(B,512,H/2,W/2,bn1Weight);
    bn2.initialize(B,512,H/2,W/2,bn2Weight);
    bn3.initialize(B,2048,H/2,W/2,bn3Weight);
    add.initialize(B,2048,H/2,W/2);
    activation64.initialize(B,512,H/2,W/2);
    activation64_cudnn.initialize(B,512,H/2,W/2);
    activation.initialize(B,2048,H/2,W/2);
    this->sparse = sparse;
    this->index = index;
    this->t1 = t1;
    this->t2 = t2;

}
float * ConvBlk7::forward(float *input){
    float *block3_1_conv = conv1.forward(input);
    float *block3_1_bn = bn1.forward(block3_1_conv);
    float *block3_1_relu;
    float *block3_2_conv;
    if(sparse){
        block3_1_relu = activation64.forward(block3_1_bn);
        hipEvent_t event_start;
        hipEvent_t event_stop;
        hipEventCreate(&event_start);
        hipEventCreate(&event_stop);

        float temp_time;
        if(MEASUE_CUSNN){
            hipEventRecord(event_start);
            block3_2_conv = conv2.forward(block3_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t1[index - 1] +=temp_time;
        }else{
            hipEventRecord(event_start);
            block3_2_conv = conv2_cudnn.forward(block3_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t2[index - 1] +=temp_time;
        }

    }else{
        block3_1_relu = activation64_cudnn.forward(block3_1_bn);
        block3_2_conv = conv2_cudnn.forward(block3_1_relu);
    }
    float *block3_2_bn = bn2.forward(block3_2_conv);
    float *block3_2_relu = activation64_cudnn.forward(block3_2_bn);

    float *block3_3_conv = conv3.forward(block3_2_relu);
    float *block3_3_bn = bn3.forward(block3_3_conv);

    float *block3_0_conv = conv0.forward(input);
    float *block3_0_bn = bn0.forward(block3_0_conv);

    float *block3_add = add.forward(block3_3_bn,block3_0_bn);
    float *block3_out = activation.forward(block3_add);
    return block3_out;
}
ConvBlk8::ConvBlk8(unsigned int b, unsigned int c, unsigned int h, unsigned int w,string conv1Weight, string bn1Weight,
                   string conv2Weight, string bn2Weight,string conv3Weight,string bn3Weight,bool sparse,int index, float *t1, float *t2) {
    B = b;
    H = h;
    C = c;
    W = w;
    conv1.initialize(B,C,H,W,512,0,1,1,1,conv1Weight);
    conv2.initialize(B,512,H,W,512,1,3,3,1,conv2Weight);
    conv2_cudnn.initialize(B,512,H,W,512,1,3,3,1,conv2Weight);
    conv3.initialize(B,512,H,W,2048,0,1,1,1,conv3Weight);
    bn1.initialize(B,512,H,W,bn1Weight);
    bn2.initialize(B,512,H,W,bn2Weight);
    bn3.initialize(B,2048,H,W,bn3Weight);
    add.initialize(B,2048,H,W);
    activation64.initialize(B,512,H,W);
    activation64_cudnn.initialize(B,512,H,W);
    activation.initialize(B,2048,H,W);
    this->sparse = sparse;
    this->index = index;
    this->t1 = t1;
    this->t2 = t2;

}
float * ConvBlk8::forward(float *input){
    float *block2_1_conv = conv1.forward(input);
    float *block2_1_bn = bn1.forward(block2_1_conv);
    float *block2_1_relu;
    float *block2_2_conv;
    if(sparse){
        block2_1_relu = activation64.forward(block2_1_bn);
        hipEvent_t event_start;
        hipEvent_t event_stop;
        hipEventCreate(&event_start);
        hipEventCreate(&event_stop);

        float temp_time;
        if(MEASUE_CUSNN){
            hipEventRecord(event_start);
            block2_2_conv = conv2.forward(block2_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t1[index - 1] +=temp_time;
        }else{
            hipEventRecord(event_start);
            block2_2_conv = conv2_cudnn.forward(block2_1_relu);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t2[index - 1] +=temp_time;
        }
    }else{
        block2_1_relu = activation64_cudnn.forward(block2_1_bn);
        block2_2_conv = conv2_cudnn.forward(block2_1_relu);
    }
    float *block2_2_bn = bn2.forward(block2_2_conv);
    float *block2_2_relu = activation64_cudnn.forward(block2_2_bn);

    float *block2_3_conv = conv3.forward(block2_2_relu);
    float *block2_3_bn = bn3.forward(block2_3_conv);

    float *block2_add = add.forward(block2_3_bn,input);
    float *block2_out = activation.forward(block2_add);
    return block2_out;
}
