#include "hip/hip_runtime.h"
#include "../inc/common.h"
#include "../inc/scnn.h"
__device__ void conv_1_512_7_7_512_switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
    switch (switch_condition) {
        case 0:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(0-s)] += result;
                }
            }
            break;
        case 1:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(1-s)] += result;
                }
            }
            break;
        case 2:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(2-s)] += result;
                }
            }
            break;
        case 3:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(3-s)] += result;
                }
            }
            break;
        case 4:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(4-s)] += result;
                }
            }
            break;
        case 5:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(5-s)] += result;
                }
            }
            break;
        case 6:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(6-s)] += result;
                }
            }
            break;
        case 7:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(7-s)] += result;
                }
            }
            break;
        case 8:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(8-s)] += result;
                }
            }
            break;
        case 9:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(0-s)] += result;
                }
            }
            break;
        case 10:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(1-s)] += result;
                }
            }
            break;
        case 11:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(2-s)] += result;
                }
            }
            break;
        case 12:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(3-s)] += result;
                }
            }
            break;
        case 13:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(4-s)] += result;
                }
            }
            break;
        case 14:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(5-s)] += result;
                }
            }
            break;
        case 15:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(6-s)] += result;
                }
            }
            break;
        case 16:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(7-s)] += result;
                }
            }
            break;
        case 17:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(8-s)] += result;
                }
            }
            break;
        case 18:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(0-s)] += result;
                }
            }
            break;
        case 19:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(1-s)] += result;
                }
            }
            break;
        case 20:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(2-s)] += result;
                }
            }
            break;
        case 21:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(3-s)] += result;
                }
            }
            break;
        case 22:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(4-s)] += result;
                }
            }
            break;
        case 23:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(5-s)] += result;
                }
            }
            break;
        case 24:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(6-s)] += result;
                }
            }
            break;
        case 25:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(7-s)] += result;
                }
            }
            break;
        case 26:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(8-s)] += result;
                }
            }
            break;
        case 27:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(0-s)] += result;
                }
            }
            break;
        case 28:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(1-s)] += result;
                }
            }
            break;
        case 29:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(2-s)] += result;
                }
            }
            break;
        case 30:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(3-s)] += result;
                }
            }
            break;
        case 31:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(4-s)] += result;
                }
            }
            break;
        case 32:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(5-s)] += result;
                }
            }
            break;
        case 33:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(6-s)] += result;
                }
            }
            break;
        case 34:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(7-s)] += result;
                }
            }
            break;
        case 35:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(8-s)] += result;
                }
            }
            break;

    }
}
__global__ void conv_1_512_7_7_512_transform(float *matrix, float *d_nnz, char *ids, unsigned int *c_lens){
    unsigned int global_id = blockIdx.x * 512 + threadIdx.x;
    unsigned int batch_id = global_id/(512*7*7);
    if(global_id >= 1*7*7*512){
        return ;
    }
    const float v = matrix[global_id];
    if(v == 0.0f){
        return;
    }
    unsigned int c = global_id % 512;

    int h = ((global_id - batch_id * 512 * 7 * 7)/512)/7+1;
    int w = ((global_id - batch_id * 512 * 7 * 7)/512)%7+1;
    int th_start = min(h/2,4-1);
    int tw_start = min(w/7,1-1);
    for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
        if((tile_h_id*2+2+3-1)<=h){
            break;
        }
        for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
            if((tile_w_id*7+7+3-1)<=w){
                break;
            }
            unsigned int tile_id = tile_h_id * 1 + tile_w_id;
            unsigned int index = atomicAdd(&c_lens[batch_id*512*4+c*4+tile_id],1);
            unsigned int abs_h = h - tile_h_id*2;
            unsigned int abs_w = w - tile_w_id*7;
            d_nnz[batch_id*512*4*(2+3-1)*(7+3-1)+
                  c*4*(2+3-1)*(7+3-1)+tile_id*(2+3-1)*(7+3-1)+index] = v;
            ids[batch_id*512*4*(2+3-1)*(7+3-1)+
                c*4*(2+3-1)*(7+3-1)+tile_id*(2+3-1)*(7+3-1)+index] = abs_h*(7+3-1)+abs_w;
        }
    }
}
__device__ void conv_1_512_7_7_512_load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<3;++r){
        for(unsigned int s=0;s<3;++s){
            data_array[r*3+s] = kernel[c_index*512*3*3+r*3*512+s*512+n_id];
        }
    }
}
__device__ void conv_1_512_7_7_512_load_input_2_shared_memory(float *values,unsigned int *c_lens,char *ids,
                                                              float *shared_input,char *shared_ids,unsigned int *shared_lens,
                                                              unsigned int warp_id,unsigned int lane_id,unsigned int batch_id,
                                                              unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<16&&tile_c_id+c_id<512;c_id+=16){
        unsigned int end_index = c_lens[batch_id*512*4+(tile_c_id+c_id)*4+tile_id];
        if(lane_id ==0){
            shared_lens[c_id] = end_index;
        }
        for(unsigned int id = lane_id;id<end_index;id+=32){
            shared_input[c_id*(2+3-1)*(7+3-1)+id] = values[batch_id*512*4*(2+3-1)*(7+3-1)+
                                                           (tile_c_id+c_id)*4*(2+3-1)*(7+3-1)+tile_id*(2+3-1)*(7+3-1)+id];
            shared_ids[c_id*(2+3-1)*(7+3-1)+id] = ids[batch_id*512*4*(2+3-1)*(7+3-1)+
                                                      (tile_c_id+c_id)*4*(2+3-1)*(7+3-1)+tile_id*(2+3-1)*(7+3-1)+id];
        }
    }
}
__global__ void conv_1_512_7_7_512_conv2d(float * __restrict__ values, unsigned int * __restrict__ c_lens,
                                          char * __restrict__ ids,
                                          const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[16*(2+3-1)*(7+3-1)];
    __shared__ char input_ids[16*(2+3-1)*(7+3-1)];
    __shared__ unsigned int channel_lens[(16)];

    const unsigned int batch_id = (blockIdx.x/(32*4));
    const unsigned int t_id = (blockIdx.x - batch_id*32*4)/32;
    const unsigned int tile_h_id = (t_id / 1)*2;
    const unsigned int tile_w_id = (t_id % 1)*7;
    const unsigned int index = blockIdx.x % (32);
    const unsigned int start_channel_index = index*16;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[2*7] = {0.0f};
    conv_1_512_7_7_512_load_input_2_shared_memory(values,c_lens,ids,input,input_ids,channel_lens,warp_id,lane_id,batch_id,t_id,start_channel_index);
    __syncthreads();
    float v;
    unsigned int id;
    for(unsigned int n = threadIdx.x;n<512;n+=512){
        for(unsigned int c=start_channel_index;c<start_channel_index+16&&c<512;c++){
            unsigned int abs_c = c - start_channel_index;
            unsigned int start_index = abs_c*(2+3-1)*(7+3-1);
            unsigned int end_index = start_index+channel_lens[abs_c];
            if(start_index == end_index){
                continue;
            }
            conv_1_512_7_7_512_load_data_2_register(data_array,(c),kernel,n);
            unsigned int iters = end_index - start_index;
            for(unsigned int iter=0;iter<iters;iter++) {
                v = input[iter+start_index];
                id = input_ids[iter+start_index];
                conv_1_512_7_7_512_switch_function(id,data_array,v,temp_result);
            }
        }
        for (unsigned int th = 0; th < 2; ++th) {
            for (unsigned int tw = 0; tw < 7; ++tw) {
                if (tile_h_id + th >= 7 || tile_w_id + tw >= 7) {
                    continue;
                }
                atomicAdd(&outputs[batch_id * 512 * 7 * 7 + (tile_h_id + th) * 7 * 512 + (tile_w_id + tw) * 512 +
                                   n],temp_result[(th * 7 + tw)]);
            }
        }
        for(unsigned int i=0;i<2*7;++i){
            temp_result[i] = 0.0f;
        }
    }
}