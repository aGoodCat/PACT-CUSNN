#include "hip/hip_runtime.h"
#include "../inc/common.h"
#include "../inc/scnn.h"
__global__ void relu(float * __restrict__ values, unsigned int B,unsigned int C,unsigned int H,
                     unsigned int W,unsigned int blks,unsigned int *counter){
    unsigned int local_count = 0;
    unsigned int lane_id = threadIdx.x % 32;
    for(unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;i<B*C*H*W;i+=blks*blockDim.x){
        float value = values[i];
        if(value==0.0f){
            local_count ++;
        }
        if(value <0.0f){
            values[i] = 0.0f;
            local_count ++;
        }
    }
    for(int j=16;j>0;j=j/2){
        local_count += __shfl_down_sync(0xFFFFFFFF,local_count,j);
    }
    if(lane_id == 0&&local_count>0){
        atomicAdd(&counter[0],local_count);
    }
}