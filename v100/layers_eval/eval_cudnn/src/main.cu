#include "hip/hip_runtime.h"
#include "../inc/cudnnOps.h"
float rand_FloatRange(float a, float b)
{
    return ((b - a) * ((float)rand() / RAND_MAX)) + a;
}
float compute_difference(float *output1, float *output2, unsigned int size){
    float diff = 0;
    #pragma omp parallel for reduction (+:diff)
    for(int i=0;i<size;++i){
        diff += abs(output1[i] - output2[i]);
    }
    return diff;
}
float compute_sparsity(float *input, unsigned int size){
    float diff = 0;
    #pragma omp parallel for reduction (+:diff)
    for(int i=0;i<size;++i){
        if(input[i] == 0.0f){
            diff +=1;
        }
    }
    diff = diff/size;
    return diff;
}
void generate_random_input(unsigned int array_size, float * array){
    #pragma omp parallel for
    for(unsigned int i=0; i<array_size; ++i){
        array[i] = rand_FloatRange(1,100);
    }
}
float get_minimum(float a, float b, float c){
    float answer = min(a,b);
    answer = min(answer,c);
    return answer;
}
int main(int argc, char *argv[]){
    unsigned int B = atoi(argv[1]);
    unsigned int C = atoi(argv[2]);
    unsigned int H = atoi(argv[3]);
    unsigned int W = atoi(argv[4]);
    unsigned int N = atoi(argv[5]);
    string network = argv[6];
    string id = argv[7];
    unsigned int inputSize = B*H*W*C;
    CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM_CONV conv1;
    conv1.initialize(B,C,H,W,N,1,3,3,1);
    CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM_CONV conv2;
    conv2.initialize(B,C,H,W,N,1,3,3,1);
    CUDNN_CONVOLUTION_FWD_ALGO_GEMM_CONV conv3;
    conv3.initialize(B,C,H,W,N,1,3,3,1);

    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);

    float *input = new float[20*H*W*C];
    generate_random_input(20*H*W*C, input);
    float *dInput;
    hipMalloc(&dInput,inputSize*sizeof(float));
    float times[4] = {0.0f};
    float inferenceTime;
    hipMemcpy(dInput,input,B*C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *output;
    float *output_cudnn;
    output_cudnn = conv1.forward(dInput);
    output_cudnn = conv2.forward(dInput);
    output_cudnn = conv3.forward(dInput);
    for(int i=0;i<20;i+=B){
        hipMemcpy(dInput,&input[i*C*H*W],inputSize*sizeof(float),hipMemcpyHostToDevice);
        hipEventRecord(event_start);
        output_cudnn = conv1.forward(dInput);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        hipEventElapsedTime(&inferenceTime, event_start, event_stop);
        times[0] += inferenceTime;

        hipEventRecord(event_start);
        output_cudnn = conv2.forward(dInput);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        hipEventElapsedTime(&inferenceTime, event_start, event_stop);
        times[1] += inferenceTime;

        hipEventRecord(event_start);
        output_cudnn = conv3.forward(dInput);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        hipEventElapsedTime(&inferenceTime, event_start, event_stop);
        times[2] += inferenceTime;
    }
    std::ofstream file_out;
    file_out.open ("cudnn_layers.txt", std::ofstream::out|std::ofstream::app);
    file_out<<network<<","<<id<<","<<get_minimum((times[0]*B)/20,(times[1]*B)/20,(times[2]*B)/20)<<endl;
    file_out.close();
    return 0;
}
