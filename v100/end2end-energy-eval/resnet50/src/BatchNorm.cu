#include "../inc/common.h"
void BatchNorm::initialize(unsigned int b, unsigned int c, unsigned int h, unsigned int w,string weight) {
    B = b;
    H = h;
    W = w;
    C = c;
    chkerr(hipMalloc(&scaleDev,C*sizeof(float)));
    chkerr(hipMalloc(&shiftDev,C*sizeof(float)));
    chkerr(hipMalloc(&meanDev,C*sizeof(float)));
    chkerr(hipMalloc(&varDev,C*sizeof(float)));
    checkCUDNN(hipdnnCreate(&batchNormCudnn));
    hipMalloc(&output,B*C*H*W*sizeof(float));
    checkCUDNN(hipdnnCreateTensorDescriptor(&batchNormInputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(batchNormInputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
    checkCUDNN(hipdnnCreateTensorDescriptor(&batchNormOutputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(batchNormOutputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
    checkCUDNN(hipdnnCreateTensorDescriptor(&bnScaleBiasMeanVarDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(bnScaleBiasMeanVarDesc,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/1,
            /*image_width=*/1));

    this->cpuKernel = (float *)malloc(4*C*sizeof(float));
    //load_input(weight,4*C,cpuKernel);
    try{
        load_input(weight,4*C,cpuKernel);
    }catch (const char* msg) {
        cerr << msg << endl;
    }
    chkerr(hipMemcpy(scaleDev,cpuKernel,C*sizeof(float),hipMemcpyHostToDevice));
    chkerr(hipMemcpy(shiftDev,&cpuKernel[C],C*sizeof(float),hipMemcpyHostToDevice));
    chkerr(hipMemcpy(meanDev,&cpuKernel[2*C],C*sizeof(float),hipMemcpyHostToDevice));
    chkerr(hipMemcpy(varDev,&cpuKernel[3*C],C*sizeof(float),hipMemcpyHostToDevice));
    free(cpuKernel);
}
float * BatchNorm::forward(float *input) {
    checkCUDNN(hipdnnBatchNormalizationForwardInference(
            batchNormCudnn,
            HIPDNN_BATCHNORM_SPATIAL,
            &alpha,
            &beta,
            batchNormInputDescriptor,
            input, //gpu上的
            batchNormOutputDescriptor,
            output, //gpu上的
            bnScaleBiasMeanVarDesc,
            scaleDev,  //gpu上的
            shiftDev,    //gpu上的
            meanDev,  //gpu上的
            varDev,//gpu上的
            HIPDNN_BN_MIN_EPSILON
    ));
    return output;
}
/*int main(void){
    BatchNorm batchNorm;
    batchNorm.initialize(1,112,112,64,
                        "../weights/conv1_bn_0.bin","../weights/conv1_bn_1.bin",
                        "../weights/conv1_bn_2.bin","../weights/conv1_bn_3.bin");
    float *input;
    float *hostInput = (float *)malloc((1*64*112*112)*sizeof(float));
    for(int i=0;i<1*64*112*112;++i){
        hostInput[i] = 1.0f;
    }
    hipMalloc(&input,1*64*112*112*sizeof(float));
    hipMemcpy(input,hostInput,1*64*112*112*sizeof(float),hipMemcpyHostToDevice);

    //conv.forward(input);
    //float *outputPython = load_input("../conv.bin",1*112*112*64);
    float *outputCudnn = (float *)malloc(1*112*112*64*sizeof(float));
    hipMemcpy(outputCudnn,batchNorm.forward(input),1*112*112*64*sizeof(float),hipMemcpyDeviceToHost);
    cout<<outputCudnn[63]<<endl;
    float diff = 0.0f;
    for(int i=0;i<112*112*64;i++){
        diff +=(outputCudnn[i] - outputPython[i]);
    }
    cout<<outputCudnn[63]<<" "<<outputPython[0]<<endl;
    cout<<diff<<endl;
    return 0;
}*/