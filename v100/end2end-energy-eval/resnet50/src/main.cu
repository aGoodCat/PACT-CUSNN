#include "../inc/common.h"
#include "../inc/nvml_helper.hpp"
void generate_random_input(float * array, unsigned int size){
    for(unsigned int i=0;i<size;++i){
        array[i] = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/1000));
    }
}
int main(int argc,char *argv[]){
    int test_images = 100;
    float *input = (float *)malloc(224*224*3*sizeof(float)*test_images);
    string imagePath = argv[1];
    float *dInput;
    hipMalloc(&dInput,224*224*3*sizeof(float)*test_images);
    Conv conv1_conv;
    conv1_conv.initialize(1,3,224,224,64,3,7,7,2,"../../../weights/resnet50/weights/conv1_conv.bin");
    BatchNorm conv1_bn;
    conv1_bn.initialize(1,64,112,112,"../../../weights/resnet50/weights/conv1_bn.bin");
    Activation conv1_relu;
    conv1_relu.initialize(1,64,112,112);
    Pool conv1_max_pool;
    conv1_max_pool.initialize(1,64,112,112,1,3,3,HIPDNN_POOLING_MAX,2);

    ConvBlk1 conv2_block1(1,64,56,56,"../../../weights/resnet50/weights/conv2_block1_0_conv.bin","../../../weights/resnet50/weights/conv2_block1_0_bn.bin",
                          "../../../weights/resnet50/weights/conv2_block1_1_conv.bin","../../../weights/resnet50/weights/conv2_block1_1_bn.bin",
                          "../../../weights/resnet50/weights/conv2_block1_2_conv.bin","../../../weights/resnet50/weights/conv2_block1_2_bn.bin",
                          "../../../weights/resnet50/weights/conv2_block1_3_conv.bin","../../../weights/resnet50/weights/conv2_block1_3_bn.bin");
    ConvBlk2 conv2_block2(1,256,56,56,"../../../weights/resnet50/weights/conv2_block2_1_conv.bin","../../../weights/resnet50/weights/conv2_block2_1_bn.bin",
                          "../../../weights/resnet50/weights/conv2_block2_2_conv.bin","../../../weights/resnet50/weights/conv2_block2_2_bn.bin",
                          "../../../weights/resnet50/weights/conv2_block2_3_conv.bin","../../../weights/resnet50/weights/conv2_block2_3_bn.bin");
    ConvBlk2 conv2_block3(1,256,56,56,"../../../weights/resnet50/weights/conv2_block3_1_conv.bin","../../../weights/resnet50/weights/conv2_block3_1_bn.bin",
                          "../../../weights/resnet50/weights/conv2_block3_2_conv.bin","../../../weights/resnet50/weights/conv2_block3_2_bn.bin",
                          "../../../weights/resnet50/weights/conv2_block3_3_conv.bin","../../../weights/resnet50/weights/conv2_block3_3_bn.bin");

    ConvBlk3 conv3_block1(1,256,56,56,"../../../weights/resnet50/weights/conv3_block1_0_conv.bin","../../../weights/resnet50/weights/conv3_block1_0_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block1_1_conv.bin","../../../weights/resnet50/weights/conv3_block1_1_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block1_2_conv.bin","../../../weights/resnet50/weights/conv3_block1_2_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block1_3_conv.bin","../../../weights/resnet50/weights/conv3_block1_3_bn.bin");
    ConvBlk4 conv3_block2(1,512,28,28,"../../../weights/resnet50/weights/conv3_block2_1_conv.bin","../../../weights/resnet50/weights/conv3_block2_1_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block2_2_conv.bin","../../../weights/resnet50/weights/conv3_block2_2_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block2_3_conv.bin","../../../weights/resnet50/weights/conv3_block2_3_bn.bin");
    ConvBlk4 conv3_block3(1,512,28,28,"../../../weights/resnet50/weights/conv3_block3_1_conv.bin","../../../weights/resnet50/weights/conv3_block3_1_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block3_2_conv.bin","../../../weights/resnet50/weights/conv3_block3_2_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block3_3_conv.bin","../../../weights/resnet50/weights/conv3_block3_3_bn.bin");
    ConvBlk4 conv3_block4(1,512,28,28,"../../../weights/resnet50/weights/conv3_block4_1_conv.bin","../../../weights/resnet50/weights/conv3_block4_1_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block4_2_conv.bin","../../../weights/resnet50/weights/conv3_block4_2_bn.bin",
                          "../../../weights/resnet50/weights/conv3_block4_3_conv.bin","../../../weights/resnet50/weights/conv3_block4_3_bn.bin");

    ConvBlk5 conv4_block1(1,512,28,28,"../../../weights/resnet50/weights/conv4_block1_0_conv.bin","../../../weights/resnet50/weights/conv4_block1_0_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block1_1_conv.bin","../../../weights/resnet50/weights/conv4_block1_1_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block1_2_conv.bin","../../../weights/resnet50/weights/conv4_block1_2_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block1_3_conv.bin","../../../weights/resnet50/weights/conv4_block1_3_bn.bin");

    ConvBlk6 conv4_block2(1,1024,14,14,"../../../weights/resnet50/weights/conv4_block2_1_conv.bin","../../../weights/resnet50/weights/conv4_block2_1_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block2_2_conv.bin","../../../weights/resnet50/weights/conv4_block2_2_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block2_3_conv.bin","../../../weights/resnet50/weights/conv4_block2_3_bn.bin");
    ConvBlk6 conv4_block3(1,1024,14,14,"../../../weights/resnet50/weights/conv4_block3_1_conv.bin","../../../weights/resnet50/weights/conv4_block3_1_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block3_2_conv.bin","../../../weights/resnet50/weights/conv4_block3_2_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block3_3_conv.bin","../../../weights/resnet50/weights/conv4_block3_3_bn.bin");
    ConvBlk6 conv4_block4(1,1024,14,14,"../../../weights/resnet50/weights/conv4_block4_1_conv.bin","../../../weights/resnet50/weights/conv4_block4_1_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block4_2_conv.bin","../../../weights/resnet50/weights/conv4_block4_2_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block4_3_conv.bin","../../../weights/resnet50/weights/conv4_block4_3_bn.bin");
    ConvBlk6 conv4_block5(1,1024,14,14,"../../../weights/resnet50/weights/conv4_block5_1_conv.bin","../../../weights/resnet50/weights/conv4_block5_1_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block5_2_conv.bin","../../../weights/resnet50/weights/conv4_block5_2_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block5_3_conv.bin","../../../weights/resnet50/weights/conv4_block5_3_bn.bin");
    ConvBlk6 conv4_block6(1,1024,14,14,"../../../weights/resnet50/weights/conv4_block6_1_conv.bin","../../../weights/resnet50/weights/conv4_block6_1_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block6_2_conv.bin","../../../weights/resnet50/weights/conv4_block6_2_bn.bin",
                          "../../../weights/resnet50/weights/conv4_block6_3_conv.bin","../../../weights/resnet50/weights/conv4_block6_3_bn.bin");

    ConvBlk7 conv5_block1(1,1024,14,14,"../../../weights/resnet50/weights/conv5_block1_0_conv.bin","../../../weights/resnet50/weights/conv5_block1_0_bn.bin",
                          "../../../weights/resnet50/weights/conv5_block1_1_conv.bin","../../../weights/resnet50/weights/conv5_block1_1_bn.bin",
                          "../../../weights/resnet50/weights/conv5_block1_2_conv.bin","../../../weights/resnet50/weights/conv5_block1_2_bn.bin",
                          "../../../weights/resnet50/weights/conv5_block1_3_conv.bin",
                          "../../../weights/resnet50/weights/conv5_block1_3_bn.bin");
    ConvBlk8 conv5_block2(1,2048,7,7,"../../../weights/resnet50/weights/conv5_block2_1_conv.bin",
                          "../../../weights/resnet50/weights/conv5_block2_1_bn.bin",
                          "../../../weights/resnet50/weights/conv5_block2_2_conv.bin","../../../weights/resnet50/weights/conv5_block2_2_bn.bin",
                          "../../../weights/resnet50/weights/conv5_block2_3_conv.bin","../../../weights/resnet50/weights/conv5_block2_3_bn.bin");
    ConvBlk8 conv5_block3(1,2048,7,7,"../../../weights/resnet50/weights/conv5_block3_1_conv.bin","../../../weights/resnet50/weights/conv5_block3_1_bn.bin",
                          "../../../weights/resnet50/weights/conv5_block3_2_conv.bin","../../../weights/resnet50/weights/conv5_block3_2_bn.bin",
                          "../../../weights/resnet50/weights/conv5_block3_3_conv.bin",
                          "../../../weights/resnet50/weights/conv5_block3_3_bn.bin");
    Pool avg_pool;
    avg_pool.initialize(1,2048,7,7,0,7,7,HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING,1);
    Conv predict;
    predict.initialize(1,2048,1,1,1000,0,1,1,1,"../../../weights/resnet50/weights/predictions.bin");

    hipMemcpy(dInput, input, 224 * 224 * 3 * sizeof(float), hipMemcpyHostToDevice);
    float *output;
    output = conv1_conv.forward(dInput);
    output = conv1_bn.forward(output);
    output = conv1_relu.forward(output);
    output = conv1_max_pool.forward(output);
    output = conv2_block1.forward(output);
    output = conv2_block2.forward(output);
    output = conv2_block3.forward(output);
    output = conv3_block1.forward(output);
    output = conv3_block2.forward(output);
    output = conv3_block3.forward(output);
    output = conv3_block4.forward(output);
    output = conv4_block1.forward(output);
    output = conv4_block2.forward(output);
    output = conv4_block3.forward(output);
    output = conv4_block4.forward(output);
    output = conv4_block5.forward(output);
    output = conv4_block6.forward(output);
    output = conv5_block1.forward(output);
    output = conv5_block2.forward(output);
    output = conv5_block3.forward(output);
    output = avg_pool.forward(output);
    output = predict.forward(output);
    hipDeviceSynchronize();
    NVML_INIT;
    NVML_DEV_t dev0;
    NVML_GET_HANDLE(0, &dev0);
    unsigned long long start,end;
    unsigned long long energy = 0;

    generate_random_input(input, test_images*3*224*224);
    hipMemcpy(dInput,input,test_images*3*224*224*sizeof(float),hipMemcpyHostToDevice);
    NVML_MEASURE(dev0,&start);
    for(int i=0;i<test_images;++i){
        output = conv1_conv.forward(&dInput[i*3*224*224]);
        output = conv1_bn.forward(output);
        output = conv1_relu.forward(output);
        output = conv1_max_pool.forward(output);
        output = conv2_block1.forward(output);
        output = conv2_block2.forward(output);
        output = conv2_block3.forward(output);
        output = conv3_block1.forward(output);
        output = conv3_block2.forward(output);
        output = conv3_block3.forward(output);
        output = conv3_block4.forward(output);
        output = conv4_block1.forward(output);
        output = conv4_block2.forward(output);
        output = conv4_block3.forward(output);
        output = conv4_block4.forward(output);
        output = conv4_block5.forward(output);
        output = conv4_block6.forward(output);
        output = conv5_block1.forward(output);
        output = conv5_block2.forward(output);
        output = conv5_block3.forward(output);
        output = avg_pool.forward(output);
        output = predict.forward(output);
        hipDeviceSynchronize();
    }
    NVML_MEASURE(dev0,&end);
    energy +=(end - start);
    cout<<"resnet50 cuDNN energy costs,"<<energy/test_images<<" mj"<<endl;
    cout<<endl;
    return 0;
}
