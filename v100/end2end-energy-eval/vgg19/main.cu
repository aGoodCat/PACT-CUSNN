
#include "./inc/conv_and_pool.h"
#include "./inc/nvml_helper.hpp"
void generate_random_input(float * array, unsigned int size){
    for(unsigned int i=0;i<size;++i){
        array[i] = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/1000));
    }
}
int main(int argc, char *argv[]){
    int test_images = 100;
    float *input = (float *)malloc(224*224*3*sizeof(float)*test_images);
    string imagePath = argv[1];
    float *dInput;
    hipMalloc(&dInput,224*224*3*sizeof(float)*test_images);

    Conv_and_pool block1(1,3,224,224,64,64,"../../../weights/vgg19/weights/block1_conv1.bin","../../../weights/vgg19/weights/block1_conv2.bin");
    Conv_and_pool block2(1,64,112,112,128,128,"../../../weights/vgg19/weights/block2_conv1.bin","../../../weights/vgg19/weights/block2_conv2.bin");
    Conv_and_pool4 block3(1,128,56,56,256,256,256,256,"../../../weights/vgg19/weights/block3_conv1.bin",
                          "../../../weights/vgg19/weights/block3_conv2.bin","../../../weights/vgg19/weights/block3_conv3.bin","../../../weights/vgg19/weights/block3_conv4.bin");
    Conv_and_pool4 block4(1,256,28,28,512,512,512,512,"../../../weights/vgg19/weights/block4_conv1.bin",
                          "../../../weights/vgg19/weights/block4_conv2.bin","../../../weights/vgg19/weights/block4_conv3.bin","../../../weights/vgg19/weights/block4_conv4.bin");
    Conv_and_pool4 block5(1,512,14,14,512,512,512,512,"../../../weights/vgg19/weights/block5_conv1.bin",
                          "../../../weights/vgg19/weights/block5_conv2.bin","../../../weights/vgg19/weights/block5_conv3.bin","../../../weights/vgg19/weights/block5_conv4.bin");
    FC fc1;
    fc1.initialize(1,25088,1,1,4096,0,1,1,1,"../../../weights/vgg19/weights/fc1.bin");
    FC fc2;
    fc2.initialize(1,4096,1,1,4096,0,1,1,1,"../../../weights/vgg19/weights/fc2.bin");
    FC fc3;
    fc3.initialize(1,4096,1,1,1000,0,1,1,1,"../../../weights/vgg19/weights/predictions.bin");

    hipMemcpy(dInput,input,224*224*3*sizeof(float),hipMemcpyHostToDevice);
    float *output;
    output = block1.forward(dInput);
    output = block2.forward(output);
    output = block3.forward(output);
    output = block4.forward(output);
    output = block5.forward(output);
    output = fc1.forward(output);
    output = fc2.forward(output);
    output = fc3.forward(output);
    hipDeviceSynchronize();
    NVML_INIT;
    NVML_DEV_t dev0;
    NVML_GET_HANDLE(0, &dev0);
    unsigned long long start,end;
    unsigned long long energy = 0;

    generate_random_input(input, test_images*3*224*224);
    hipMemcpy(dInput,input,test_images*3*224*224*sizeof(float),hipMemcpyHostToDevice);
    NVML_MEASURE(dev0,&start);
    for(int i=0;i<test_images;++i){
        output = block1.forward(dInput);
        output = block2.forward(output);
        output = block3.forward(output);
        output = block4.forward(output);
        output = block5.forward(output);
        output = fc1.forward(output);
        output = fc2.forward(output);
        output = fc3.forward(output);
        hipDeviceSynchronize();
    }

    NVML_MEASURE(dev0,&end);
    energy +=(end - start);
    cout<<"vgg19 cuDNN energy costs,"<<energy/test_images<<" mj"<<endl;
    cout<<endl;
    return 0;
}
