#include "hip/hip_runtime.h"
#include "../inc/scnnOps.h"
__device__ void conv_1_128_28_28_128_switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
    switch (switch_condition) {
        case 0:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(0-s)] += result;
                }
            }
            break;
        case 1:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(1-s)] += result;
                }
            }
            break;
        case 2:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(2-s)] += result;
                }
            }
            break;
        case 3:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(3-s)] += result;
                }
            }
            break;
        case 4:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(4-s)] += result;
                }
            }
            break;
        case 5:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(5-s)] += result;
                }
            }
            break;
        case 6:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(6-s)] += result;
                }
            }
            break;
        case 7:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(7-s)] += result;
                }
            }
            break;
        case 8:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(8-s)] += result;
                }
            }
            break;
        case 9:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(0-s)] += result;
                }
            }
            break;
        case 10:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(1-s)] += result;
                }
            }
            break;
        case 11:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(2-s)] += result;
                }
            }
            break;
        case 12:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(3-s)] += result;
                }
            }
            break;
        case 13:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(4-s)] += result;
                }
            }
            break;
        case 14:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(5-s)] += result;
                }
            }
            break;
        case 15:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(6-s)] += result;
                }
            }
            break;
        case 16:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(7-s)] += result;
                }
            }
            break;
        case 17:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(8-s)] += result;
                }
            }
            break;
        case 18:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(0-s)] += result;
                }
            }
            break;
        case 19:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(1-s)] += result;
                }
            }
            break;
        case 20:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(2-s)] += result;
                }
            }
            break;
        case 21:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(3-s)] += result;
                }
            }
            break;
        case 22:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(4-s)] += result;
                }
            }
            break;
        case 23:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(5-s)] += result;
                }
            }
            break;
        case 24:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(6-s)] += result;
                }
            }
            break;
        case 25:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(7-s)] += result;
                }
            }
            break;
        case 26:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(8-s)] += result;
                }
            }
            break;
        case 27:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(0-s)] += result;
                }
            }
            break;
        case 28:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(1-s)] += result;
                }
            }
            break;
        case 29:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(2-s)] += result;
                }
            }
            break;
        case 30:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(3-s)] += result;
                }
            }
            break;
        case 31:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(4-s)] += result;
                }
            }
            break;
        case 32:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(5-s)] += result;
                }
            }
            break;
        case 33:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(6-s)] += result;
                }
            }
            break;
        case 34:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(7-s)] += result;
                }
            }
            break;
        case 35:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(8-s)] += result;
                }
            }
            break;

    }
}
__global__ void conv_1_128_28_28_128_transform(float *matrix, float *d_nnz, char *ids, unsigned int *c_lens){
    unsigned int global_id = blockIdx.x * 512 + threadIdx.x;
    unsigned int batch_id = global_id/(128*28*28);
    if(global_id >= 1*28*28*128){
        return ;
    }
    const float v = matrix[global_id];
    if(v == 0.0f){
        return;
    }
    unsigned int c = global_id % 128;

    int h = ((global_id - batch_id * 128 * 28 * 28)/128)/28+1;
    int w = ((global_id - batch_id * 128 * 28 * 28)/128)%28+1;
    int th_start = min(h/2,14-1);
    int tw_start = min(w/7,4-1);
    for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
        if((tile_h_id*2+2+3-1)<=h){
            break;
        }
        for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
            if((tile_w_id*7+7+3-1)<=w){
                break;
            }
            unsigned int tile_id = tile_h_id * 4 + tile_w_id;
            unsigned int index = atomicAdd(&c_lens[batch_id*128*56+c*56+tile_id],1);
            unsigned int abs_h = h - tile_h_id*2;
            unsigned int abs_w = w - tile_w_id*7;
            d_nnz[batch_id*128*56*(2+3-1)*(7+3-1)+
                  c*56*(2+3-1)*(7+3-1)+tile_id*(2+3-1)*(7+3-1)+index] = v;
            ids[batch_id*128*56*(2+3-1)*(7+3-1)+
                c*56*(2+3-1)*(7+3-1)+tile_id*(2+3-1)*(7+3-1)+index] = abs_h*(7+3-1)+abs_w;
        }
    }
}
__device__ void conv_1_128_28_28_128_load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<3;++r){
        for(unsigned int s=0;s<3;++s){
            data_array[r*3+s] = kernel[c_index*128*3*3+r*3*128+s*128+n_id];
        }
    }
}
__device__ void conv_1_128_28_28_128_load_input_2_shared_memory(float *values,unsigned int *c_lens,char *ids,
                                                                float *shared_input,char *shared_ids,unsigned int *shared_lens,
                                                                unsigned int warp_id,unsigned int lane_id,unsigned int batch_id,
                                                                unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<8&&tile_c_id+c_id<128;c_id+=4){
        unsigned int end_index = c_lens[batch_id*128*56+(tile_c_id+c_id)*56+tile_id];
        if(lane_id ==0){
            shared_lens[c_id] = end_index;
        }
        for(unsigned int id = lane_id;id<end_index;id+=32){
            shared_input[c_id*(2+3-1)*(7+3-1)+id] = values[batch_id*128*56*(2+3-1)*(7+3-1)+
                                                           (tile_c_id+c_id)*56*(2+3-1)*(7+3-1)+tile_id*(2+3-1)*(7+3-1)+id];
            shared_ids[c_id*(2+3-1)*(7+3-1)+id] = ids[batch_id*128*56*(2+3-1)*(7+3-1)+
                                                      (tile_c_id+c_id)*56*(2+3-1)*(7+3-1)+tile_id*(2+3-1)*(7+3-1)+id];
        }
    }
}
__global__ void conv_1_128_28_28_128_conv2d(float * __restrict__ values, unsigned int * __restrict__ c_lens,
                                            char * __restrict__ ids,
                                            const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[8*(2+3-1)*(7+3-1)];
    __shared__ char input_ids[8*(2+3-1)*(7+3-1)];
    __shared__ unsigned int channel_lens[(8)];

    const unsigned int batch_id = (blockIdx.x/(16*56));
    const unsigned int t_id = (blockIdx.x - batch_id*16*56)/16;
    const unsigned int tile_h_id = (t_id / 4)*2;
    const unsigned int tile_w_id = (t_id % 4)*7;
    const unsigned int index = blockIdx.x % (16);
    const unsigned int start_channel_index = index*8;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[2*7] = {0.0f};
    conv_1_128_28_28_128_load_input_2_shared_memory(values,c_lens,ids,input,input_ids,channel_lens,warp_id,lane_id,batch_id,t_id,start_channel_index);
    __syncthreads();
    float v;
    unsigned int id;
    for(unsigned int n = threadIdx.x;n<128;n+=128){
        for(unsigned int c=start_channel_index;c<start_channel_index+8&&c<128;c++){
            unsigned int abs_c = c - start_channel_index;
            unsigned int start_index = abs_c*(2+3-1)*(7+3-1);
            unsigned int end_index = start_index+channel_lens[abs_c];
            if(start_index == end_index){
                continue;
            }
            conv_1_128_28_28_128_load_data_2_register(data_array,(c),kernel,n);
            unsigned int iters = end_index - start_index;
            for(unsigned int iter=0;iter<iters;iter++) {
                v = input[iter+start_index];
                id = input_ids[iter+start_index];
                conv_1_128_28_28_128_switch_function(id,data_array,v,temp_result);
            }
        }
        for (unsigned int th = 0; th < 2; ++th) {
            for (unsigned int tw = 0; tw < 7; ++tw) {
                if (tile_h_id + th >= 28 || tile_w_id + tw >= 28) {
                    continue;
                }
                atomicAdd(&outputs[batch_id * 128 * 28 * 28 + (tile_h_id + th) * 28 * 128 + (tile_w_id + tw) * 128 +
                                   n],temp_result[(th * 7 + tw)]);
            }
        }
        for(unsigned int i=0;i<2*7;++i){
            temp_result[i] = 0.0f;
        }
    }
}
void Conv_1_128_28_28_128::initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                                      unsigned int pad,unsigned int r,unsigned int s,unsigned int stride,string weightFile) {
    this->B = b;
    this->C = c;
    this->H = h;
    this->W = w;
    this->N = n;
    this->hOut = h;
    this->wOut = w;
    hipdnnCreate(&convCudnn);
    unsigned int kernelSize = 3*3*C*N + N;//kernel + bias
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    try{
        load_input(weightFile,kernelSize,cpuKernel);
    }catch (const char* msg) {
        cerr << msg << endl;
    }
    float *temp_kernel = (float *)malloc(kernelSize*sizeof(float));
    for(unsigned int i=0;i<N;++i){
        for(unsigned int l=0;l<C;++l){
            for(unsigned int j=0;j<3;++j){
                for(unsigned int k=0;k<3;++k){
                    temp_kernel[l*N*3*3+j*3*N+k*N+i] = cpuKernel[i*3*3*C+l*9+j*3+k];
                }
            }
        }
    }
    hipMalloc(&kernel,9*C*N*sizeof(float));
    hipMemcpy(kernel,temp_kernel,9*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(temp_kernel);
    hipMalloc(&bias,N*sizeof(float));
    hipMemcpy(bias,&cpuKernel[9*C*N],N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
    TILES_EACH_CHANNEL = ((H-1)/TH+1)*((W-1)/TW+1);
    TCS = (C-1)/TC + 1;
    hipMalloc(&nnz,b*c*TILES_EACH_CHANNEL*(TH+3-1)*(TW+3-1)*sizeof(float));
    hipMalloc(&ids,b*c*TILES_EACH_CHANNEL*(TH+3-1)*(TW+3-1)*sizeof(char));
    hipMalloc(&c_lens,b*c*TILES_EACH_CHANNEL*sizeof(unsigned int));
    hipMalloc(&output,b*n*h*w*sizeof(float));
    hipdnnCreateTensorDescriptor(&convOutputDescriptor);
    hipdnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/N,
            /*image_height=*/hOut,
            /*image_width=*/wOut);
    hipdnnCreateTensorDescriptor(&biasDescriptor);
    hipdnnSetTensor4dDescriptor(biasDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/1,
            /*image_width=*/1);
}
float * Conv_1_128_28_28_128::forward(float *input) {
    hipMemset(output, 0, B*N*hOut*wOut*sizeof(float));
    hipMemset(c_lens, 0, B*C*TILES_EACH_CHANNEL*sizeof(unsigned int));
    conv_1_128_28_28_128_transform<<<(B*C*H*W-1)/512+1,512>>>(input,this->nnz,this->ids,this->c_lens);
    conv_1_128_28_28_128_conv2d<<<B*TCS*TILES_EACH_CHANNEL,N>>>(this->nnz,this->c_lens,this->ids,this->kernel,this->output);

    //chkerr(hipGetLastError());
    //chkerr(hipDeviceSynchronize());
    checkCUDNN(hipdnnAddTensor(convCudnn,&alpha,biasDescriptor,bias,&beta2,convOutputDescriptor,output));
    return output;
}
