#include "../inc/common.h"
ConvBlk2::ConvBlk2(unsigned int b, unsigned int c, unsigned int h, unsigned int w,string conv1Weight, string bn1Weight,
                   string conv2Weight, string bn2Weight,string conv3Weight,string bn3Weight) {
    B = b;
    H = h;
    C = c;
    W = w;
    conv1.initialize(B,C,H,W,64,0,1,1,1,conv1Weight);
    conv2.initialize(B,conv1.N,conv1.hOut,conv1.wOut,64,1,3,3,1,conv2Weight);
    conv3.initialize(B,conv2.N,conv2.hOut,conv2.wOut,256,0,1,1,1,conv3Weight);
    bn1.initialize(B,64,conv1.hOut,conv1.wOut,bn1Weight);
    bn2.initialize(B,64,conv2.hOut,conv2.wOut,bn2Weight);
    bn3.initialize(B,256,conv3.hOut,conv3.wOut,bn3Weight);
    add.initialize(B,256,H,W);
    activation64.initialize(B,64,conv1.hOut,conv1.wOut);
    activation.initialize(B,256,conv3.hOut,conv3.wOut);
}
float * ConvBlk2::forward(float *input){
    float *block2_1_conv = conv1.forward(input);
    float *block2_1_bn = bn1.forward(block2_1_conv);
    float *block2_1_relu = activation64.forward(block2_1_bn);

    float *block2_2_conv = conv2.forward(block2_1_relu);
    float *block2_2_bn = bn2.forward(block2_2_conv);
    float *block2_2_relu = activation64.forward(block2_2_bn);

    float *block2_3_conv = conv3.forward(block2_2_relu);
    float *block2_3_bn = bn3.forward(block2_3_conv);

    float *block2_add = add.forward(block2_3_bn,input);
    float *block2_out = activation.forward(block2_add);
    return block2_out;
}
/*int main(void){
    ConvBlk2 a(1,256,56,56,"../weights/conv2_block3_1_conv.bin","../weights/conv2_block3_1_bn.bin",
                                   "../weights/conv2_block3_2_conv.bin","../weights/conv2_block3_2_bn.bin",
                                   "../weights/conv2_block3_3_conv.bin","../weights/conv2_block3_3_bn.bin");
    float *input = (float *)malloc(256*56*56*sizeof(float));
    for(int i=0;i<256*56*56;++i){
        input[i] = 1.0f;
    }
    float *dInput;
    hipMalloc(&dInput,256*56*56*sizeof(float));
    hipMemcpy(dInput,input,256*56*56*sizeof(float),hipMemcpyHostToDevice);
    float *dOutput = a.forward(dInput);
    float *output = (float *)malloc(256*56*56*sizeof(float));
    hipMemcpy(output,dOutput,256*56*56*sizeof(float),hipMemcpyDeviceToHost);
    cout<<output[0]<<endl;
    return 0;
}*/