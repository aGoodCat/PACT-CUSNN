#include "../inc/common.h"
/*
 * hipdnnNanPropagation_t : HIPDNN_NOT_PROPAGATE_NAN,HIPDNN_PROPAGATE_NAN
 * hipdnnActivationMode_t:
 * HIPDNN_ACTIVATION_SIGMOID,
    HIPDNN_ACTIVATION_RELU,
    HIPDNN_ACTIVATION_TANH,
    HIPDNN_ACTIVATION_CLIPPED_RELU,
    HIPDNN_ACTIVATION_ELU,
 *
 */
void Activation::initialize(unsigned int b, unsigned int c, unsigned int h, unsigned int w) {
    B = b;
    C = c;
    H = h;
    W = w;
    hipMalloc(&output,B*C*H*W*sizeof(float));
    checkCUDNN(hipdnnCreate(&activationCudnn));
    checkCUDNN(hipdnnCreateTensorDescriptor(&activationInputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(activationInputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
    checkCUDNN(hipdnnCreateTensorDescriptor(&activationOutputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(activationOutputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
    checkCUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(activationDesc,HIPDNN_ACTIVATION_RELU,HIPDNN_NOT_PROPAGATE_NAN,0.0f));
}
float * Activation::forward(float *input) {
    checkCUDNN(hipdnnActivationForward(activationCudnn,activationDesc,&alpha,activationInputDescriptor,input,&beta,activationOutputDescriptor,output));
    return output;
}
/*int main(void){
    Activation activation;
    activation.initialize(1,112,112,64);
    float *input;
    float *hostInput = (float *)malloc((1*64*112*112)*sizeof(float));
    for(int i=0;i<1*64*112*112;++i){
        hostInput[i] = 0.000001f;
    }
    hipMalloc(&input,1*64*112*112*sizeof(float));
    hipMemcpy(input,hostInput,1*64*112*112*sizeof(float),hipMemcpyHostToDevice);

    //conv.forward(input);
    //float *outputPython = load_input("../conv.bin",1*112*112*64);
    float *outputCudnn = (float *)malloc(1*112*112*64*sizeof(float));
    hipMemcpy(outputCudnn,activation.forward(input),1*112*112*64*sizeof(float),hipMemcpyDeviceToHost);
    cout<<outputCudnn[63]<<endl;
    float diff = 0.0f;
    cout<<outputCudnn[63]<<endl;
    cout<<diff<<endl;
    return 0;
}*/