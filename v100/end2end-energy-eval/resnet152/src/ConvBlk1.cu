#include "../inc/common.h"
// conv2_block1
ConvBlk1::ConvBlk1(unsigned int b, unsigned int c, unsigned int h, unsigned int w, string conv0Weight, string bn0Weight,
                   string conv1Weight, string bn1Weight, string conv2Weight, string bn2Weight,string conv3Weight,string bn3Weight){
    B = b;
    H = h;
    C = c;
    W = w;
    conv0.initialize(B,C,H,W,256,0,1,1,1,conv0Weight);
    conv1.initialize(B,C,H,W,64,0,1,1,1,conv1Weight);
    conv2.initialize(B,C,H,W,64,1,3,3,1,conv2Weight);
    conv3.initialize(B,C,H,W,256,0,1,1,1,conv3Weight);
    bn0.initialize(B,256,H,W,bn0Weight);
    bn1.initialize(B,64,H,W,bn1Weight);
    bn2.initialize(B,64,H,W,bn2Weight);
    bn3.initialize(B,256,H,W,bn3Weight);
    add.initialize(B,256,H,W);
    activation64.initialize(B,64,H,W);
    activation.initialize(B,256,H,W);
}
float * ConvBlk1::forward(float *input){
    float *block1_1_conv = conv1.forward(input);
    float *block1_1_bn = bn1.forward(block1_1_conv);
    float *block1_1_relu = activation64.forward(block1_1_bn);

    float *block1_2_conv = conv2.forward(block1_1_relu);
    float *block1_2_bn = bn2.forward(block1_2_conv);
    float *block1_2_relu = activation64.forward(block1_2_bn);

    float *block1_3_conv = conv3.forward(block1_2_relu);
    float *block1_3_bn = bn3.forward(block1_3_conv);

    float *block1_0_conv = conv0.forward(input);
    float *block1_0_bn = bn0.forward(block1_0_conv);

    float *block1_add = add.forward(block1_3_bn,block1_0_bn);
    float *relu = activation.forward(block1_add);
    return relu;
}
/*int main(void){
    ConvBlk1 a(1,64,56,56,"/home/lizhi/research/resnet152/weights/conv2_block1_0_conv.bin",
               "/home/lizhi/research/resnet152/weights/conv2_block1_0_bn.bin",
               "/home/lizhi/research/resnet152/weights/conv2_block1_1_conv.bin",
               "/home/lizhi/research/resnet152/weights/conv2_block1_1_bn.bin",
               "/home/lizhi/research/resnet152/weights/conv2_block1_2_conv.bin",
               "/home/lizhi/research/resnet152/weights/conv2_block1_2_bn.bin",
               "/home/lizhi/research/resnet152/weights/conv2_block1_3_conv.bin",
               "/home/lizhi/research/resnet152/weights/conv2_block1_3_bn.bin");
    float *input = (float *)malloc(64*56*56*sizeof(float));
    for(int i=0;i<64*56*56;++i){
        input[i] = 1.0f;
    }
    float *dInput;
    hipMalloc(&dInput,64*56*56*sizeof(float));
    hipMemcpy(dInput,input,64*56*56*sizeof(float),hipMemcpyHostToDevice);
    float *dOutput = a.forward(dInput);
    float *output = (float *)malloc(256*56*56*sizeof(float));
    hipMemcpy(output,dOutput,256*56*56*sizeof(float),hipMemcpyDeviceToHost);
    float *pyOut = load_input("/home/lizhi/research/resnet152/convblk1.bin",64*56*256);
    cout<<pyOut[1]<<" "<<output[1]<<endl;
    float diff = 0.0f;
    for(int i=0;i<56*56*256;i++){
        diff += pyOut[i] - output[i];
    }
    cout<<diff<<endl;
    return 0;
}*/