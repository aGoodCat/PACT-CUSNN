#include "../inc/common.h"
void Add::initialize(unsigned int b, unsigned int c, unsigned int h, unsigned int w) {
    B = b;
    C = c;
    H = h;
    W = w;
    checkCUDNN(hipdnnCreate(&addCudnn));
    checkCUDNN(hipdnnCreateTensorDescriptor(&addInputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(addInputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
    checkCUDNN(hipdnnCreateTensorDescriptor(&addOutputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(addOutputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
}
float *Add::forward(float *x, float *y) {
    hipdnnAddTensor(addCudnn,&alpha,addInputDescriptor,x,&beta,addOutputDescriptor,y);
    return y;
}
/*int main(void){
    Add add;
    add.initialize(1,112,112,64);
    float *input;
    float *input2;
    float *hostInput = (float *)malloc((1*64*112*112)*sizeof(float));
    for(int i=0;i<1*64*112*112;++i){
        hostInput[i] = 1.0f;
    }
    hipMalloc(&input,1*64*112*112*sizeof(float));
    hipMemcpy(input,hostInput,1*64*112*112*sizeof(float),hipMemcpyHostToDevice);
    hipMalloc(&input2,1*64*112*112*sizeof(float));
    hipMemcpy(input2,hostInput,1*64*112*112*sizeof(float),hipMemcpyHostToDevice);
    //conv.forward(input);
    //float *outputPython = load_input("../conv.bin",1*112*112*64);
    float *outputCudnn = (float *)malloc(1*112*112*64*sizeof(float));
    hipMemcpy(outputCudnn,add.forward(input,input2),1*112*112*64*sizeof(float),hipMemcpyDeviceToHost);
    cout<<outputCudnn[63]<<endl;
    float diff = 0.0f;
    for(int i=0;i<112*112*64;i++){
        diff +=(outputCudnn[i] - outputPython[i]);
    }
    cout<<outputCudnn[63]<<" "<<outputPython[0]<<endl;
    cout<<diff<<endl;
    return 0;
}*/