#include "../inc/cudnnOps.h"
void Conv::initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                      unsigned int pad,unsigned int r,unsigned int s,unsigned int stride,string weightFile){
    this->B = b;
    this->C = c;
    this->H = h;
    this->W = w;
    this->N = n;
    this->R = r;
    this->S = s;
    this->hOut = (H+2*pad - r)/stride + 1;
    this->wOut = (W+2*pad - s)/stride + 1;
    hipMalloc(&kernel,sizeof(float)*C*N*R*S);
    hipMalloc(&this->output,sizeof(float)*B*hOut*wOut*N);
    hipdnnCreate(&convCudnn);
    hipdnnCreateTensorDescriptor(&convInputDescriptor);
    hipdnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    hipdnnCreateFilterDescriptor(&convKernelDescriptor);
    hipdnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    hipdnnCreateConvolutionDescriptor(&convDesc);
    hipdnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/pad,
            /*pad_width=*/pad,
            /*vertical_stride=*/stride,
            /*horizontal_stride=*/stride,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/HIPDNN_CROSS_CORRELATION,
                                    HIPDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    hipdnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    hipdnnCreateTensorDescriptor(&convOutputDescriptor);
    hipdnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/N,
            /*image_height=*/hOut,
            /*image_width=*/wOut);
    hipdnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    try{
        load_input(weightFile,kernelSize,cpuKernel);
    }catch (const char* msg) {
        cerr << msg << endl;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * Conv::forward(float *input) {
    hipMemset(output, 0, B*N*hOut*wOut*sizeof(float));
    checkCUDNN(hipdnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}