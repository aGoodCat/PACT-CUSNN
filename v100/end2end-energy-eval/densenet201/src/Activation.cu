#include "../inc/cudnnOps.h"
void Activation::initialize(unsigned int b, unsigned int c, unsigned int h, unsigned int w) {
    B = b;
    C = c;
    H = h;
    W = w;
    hipMalloc(&output,B*C*H*W*sizeof(float));
    checkCUDNN(hipdnnCreate(&activationCudnn));
    checkCUDNN(hipdnnCreateTensorDescriptor(&activationInputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(activationInputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
    checkCUDNN(hipdnnCreateTensorDescriptor(&activationOutputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(activationOutputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
    checkCUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(activationDesc,HIPDNN_ACTIVATION_RELU,HIPDNN_NOT_PROPAGATE_NAN,0.0f));
}
float * Activation::forward(float *input) {
    checkCUDNN(hipdnnActivationForward(activationCudnn,activationDesc,&alpha,activationInputDescriptor,input,&beta,activationOutputDescriptor,output));
    return output;
}