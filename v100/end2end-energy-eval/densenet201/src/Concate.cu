#include "hip/hip_runtime.h"
#include "../inc/cudnnOps.h"
#include "../inc/cudnnOps.h"
__global__ void concate(unsigned int b,unsigned int c1,unsigned int c2,unsigned int h,unsigned int w,float *x,
                        float * y, float *z){
    unsigned int id = threadIdx.x + blockIdx.x*blockDim.x;
    if(id >=b*(c1+c2)*h*w){
        return ;
    }
    for(unsigned int i = id;i<b*h*w*(c1+c2);i+=gridDim.x*blockDim.x){
        unsigned int c = i % (c1 + c2);
        unsigned int c_id = i / (c1 + c2);
        if(c >= c1){
            float v = y[c_id * c2 + c - c1];
            z[c_id*(c1+c2)+c] = v;
        }else{
            float v = x[c_id * c1 + c];
            z[c_id*(c1+c2)+c] = v;
        }
    }
}
void Concate::initialize(unsigned int b, unsigned int c1, unsigned int c2, unsigned int h, unsigned int w) {
    B = b;
    C1 = c1;
    C2 = c2;
    H = h;
    W = w;
    hipMalloc(&output,b*(c1+c2)*h*w*sizeof(float));
}
float * Concate::forward(float *x, float *y) {
    concate<<<84,1024>>>(B,C1,C2,H,W,x,y,output);
    return output;
}