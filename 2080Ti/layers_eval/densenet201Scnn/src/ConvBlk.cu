#include "../inc/convBlk.h"
#define MEASUE_CUSNN true
ConvBlk56::ConvBlk56(unsigned int b, unsigned int c, unsigned int h, unsigned int w, unsigned int n1, unsigned int n2,
                     string bn0Weight, string conv1Weight, string bn1Weight, string conv2Weight,bool sparse, int index,float *t1, float *t2) {
    conv_block_0_bn.initialize(b,c,h,w,bn0Weight);
    conv_block_0_relu.initialize(b,c,h,w);
    conv_block_1_conv.initialize(b,c,h,w,n1,0,1,1,1,conv1Weight);
    conv_block_1_bn.initialize(b,n1,h,w,bn1Weight);
    conv_block_1_relu.initialize(b,n1,h,w);
    conv_block_1_relu_cudnn.initialize(b,n1,h,w);
    conv_block_2_conv_cudnn.initialize(b,n1,h,w,n2,1,3,3,1,conv2Weight);
    conv_block_2_conv.initialize(b,n1,h,w,n2,1,3,3,1,conv2Weight);
    conv_block_concat.initialize(b,c,n2,h,w);
    this->sparse = sparse;
    this->index = index;
    this->t1 = t1;
    this->t2 = t2;
}
float * ConvBlk56::forward(float *input) {
    float * conv_block_0_bn_out = conv_block_0_bn.forward(input);
    float * conv_block_0_relu_out = conv_block_0_relu.forward(conv_block_0_bn_out);
    float * conv_block_1_conv_out = conv_block_1_conv.forward(conv_block_0_relu_out);
    float * conv_block_1_bn_out = conv_block_1_bn.forward(conv_block_1_conv_out);
    float * conv_block_1_relu_out;
    float * conv_block_2_conv_out;
    conv_block_1_relu_out = conv_block_1_relu_cudnn.forward(conv_block_1_bn_out);
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);

    float temp_time;
    if(MEASUE_CUSNN){
        hipEventRecord(event_start);
        conv_block_2_conv_out = conv_block_2_conv.forward(conv_block_1_relu_out);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        t1[index - 1] +=temp_time;
    }else{
        hipEventRecord(event_start);
        conv_block_2_conv_out = conv_block_2_conv_cudnn.forward(conv_block_1_relu_out);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        t2[index - 1] +=temp_time;
    }
    float * conv_block_concat_out = conv_block_concat.forward(input,conv_block_2_conv_out);
    return conv_block_concat_out;
}
ConvBlk28::ConvBlk28(unsigned int b, unsigned int c, unsigned int h, unsigned int w, unsigned int n1, unsigned int n2,
                     string bn0Weight, string conv1Weight, string bn1Weight, string conv2Weight, bool sparse,int index, float *t1, float *t2) {
    conv_block_0_bn.initialize(b,c,h,w,bn0Weight);
    conv_block_0_relu.initialize(b,c,h,w);
    conv_block_1_conv.initialize(b,c,h,w,n1,0,1,1,1,conv1Weight);
    conv_block_1_bn.initialize(b,n1,h,w,bn1Weight);
    conv_block_1_relu.initialize(b,n1,h,w);
    conv_block_1_relu_cudnn.initialize(b,n1,h,w);
    conv_block_2_conv.initialize(b,n1,h,w,n2,1,3,3,1,conv2Weight);
    conv_block_2_conv_cudnn.initialize(b,n1,h,w,n2,1,3,3,1,conv2Weight);
    conv_block_concat.initialize(b,c,n2,h,w);
    this->sparse = sparse;
    this->index = index;
    this->t1 = t1;
    this->t2 = t2;
}
float * ConvBlk28::forward(float *input) {
    float * conv_block_0_bn_out = conv_block_0_bn.forward(input);
    float * conv_block_0_relu_out = conv_block_0_relu.forward(conv_block_0_bn_out);
    float * conv_block_1_conv_out = conv_block_1_conv.forward(conv_block_0_relu_out);
    float * conv_block_1_bn_out = conv_block_1_bn.forward(conv_block_1_conv_out);
    float * conv_block_1_relu_out;
    float * conv_block_2_conv_out;
    if(sparse){
        conv_block_1_relu_out = conv_block_1_relu.forward(conv_block_1_bn_out);
        hipEvent_t event_start;
        hipEvent_t event_stop;
        hipEventCreate(&event_start);
        hipEventCreate(&event_stop);

        float temp_time;
        if(MEASUE_CUSNN){
            hipEventRecord(event_start);
            conv_block_2_conv_out = conv_block_2_conv.forward(conv_block_1_relu_out);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t1[index - 1] +=temp_time;
        }else{
            hipEventRecord(event_start);
            conv_block_2_conv_out = conv_block_2_conv_cudnn.forward(conv_block_1_relu_out);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t2[index - 1] +=temp_time;
        }
    }else{
        conv_block_1_relu_out = conv_block_1_relu_cudnn.forward(conv_block_1_bn_out);
        conv_block_2_conv_out = conv_block_2_conv_cudnn.forward(conv_block_1_relu_out);
    }
    float * conv_block_concat_out = conv_block_concat.forward(input,conv_block_2_conv_out);
    return conv_block_concat_out;
}
ConvBlk14::ConvBlk14(unsigned int b, unsigned int c, unsigned int h, unsigned int w, unsigned int n1, unsigned int n2,
                     string bn0Weight, string conv1Weight, string bn1Weight, string conv2Weight, bool sparse, int index, float *t1, float *t2) {
    conv_block_0_bn.initialize(b,c,h,w,bn0Weight);
    conv_block_0_relu.initialize(b,c,h,w);
    conv_block_1_conv.initialize(b,c,h,w,n1,0,1,1,1,conv1Weight);
    conv_block_1_bn.initialize(b,n1,h,w,bn1Weight);
    conv_block_1_relu.initialize(b,n1,h,w);
    conv_block_1_relu_cudnn.initialize(b,n1,h,w);
    conv_block_2_conv.initialize(b,n1,h,w,n2,1,3,3,1,conv2Weight);
    conv_block_2_conv_cudnn.initialize(b,n1,h,w,n2,1,3,3,1,conv2Weight);
    conv_block_concat.initialize(b,c,n2,h,w);
    this->sparse = sparse;
    this->index = index;
    this->t1 = t1;
    this->t2 = t2;
}
float * ConvBlk14::forward(float *input) {
    float * conv_block_0_bn_out = conv_block_0_bn.forward(input);
    float * conv_block_0_relu_out = conv_block_0_relu.forward(conv_block_0_bn_out);
    float * conv_block_1_conv_out = conv_block_1_conv.forward(conv_block_0_relu_out);
    float * conv_block_1_bn_out = conv_block_1_bn.forward(conv_block_1_conv_out);
    float * conv_block_1_relu_out;
    float * conv_block_2_conv_out;
    if(sparse){
        conv_block_1_relu_out = conv_block_1_relu.forward(conv_block_1_bn_out);
        hipEvent_t event_start;
        hipEvent_t event_stop;
        hipEventCreate(&event_start);
        hipEventCreate(&event_stop);

        float temp_time;
        if(MEASUE_CUSNN){
            hipEventRecord(event_start);
            conv_block_2_conv_out = conv_block_2_conv.forward(conv_block_1_relu_out);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t1[index - 1] +=temp_time;
        }else{
            hipEventRecord(event_start);
            conv_block_2_conv_out = conv_block_2_conv_cudnn.forward(conv_block_1_relu_out);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t2[index - 1] +=temp_time;
        }
    }else{
        conv_block_1_relu_out = conv_block_1_relu_cudnn.forward(conv_block_1_bn_out);
        conv_block_2_conv_out = conv_block_2_conv_cudnn.forward(conv_block_1_relu_out);
    }
    float * conv_block_concat_out = conv_block_concat.forward(input,conv_block_2_conv_out);
    return conv_block_concat_out;
}
ConvBlk7::ConvBlk7(unsigned int b, unsigned int c, unsigned int h, unsigned int w, unsigned int n1, unsigned int n2,
                   string bn0Weight, string conv1Weight, string bn1Weight, string conv2Weight,bool sparse, int index, float *t1, float *t2) {
    conv_block_0_bn.initialize(b,c,h,w,bn0Weight);
    conv_block_0_relu.initialize(b,c,h,w);
    conv_block_1_conv.initialize(b,c,h,w,n1,0,1,1,1,conv1Weight);
    conv_block_1_bn.initialize(b,n1,h,w,bn1Weight);
    conv_block_1_relu.initialize(b,n1,h,w);
    conv_block_1_relu_cudnn.initialize(b,n1,h,w);
    conv_block_2_conv_cudnn.initialize(b,n1,h,w,n2,1,3,3,1,conv2Weight);
    conv_block_2_conv.initialize(b,n1,h,w,n2,1,3,3,1,conv2Weight);
    conv_block_concat.initialize(b,c,n2,h,w);
    this->sparse = sparse;
    this->index = index;
    this->t1 = t1;
    this->t2 = t2;
}
float * ConvBlk7::forward(float *input) {
    float * conv_block_0_bn_out = conv_block_0_bn.forward(input);
    float * conv_block_0_relu_out = conv_block_0_relu.forward(conv_block_0_bn_out);
    float * conv_block_1_conv_out = conv_block_1_conv.forward(conv_block_0_relu_out);
    float * conv_block_1_bn_out = conv_block_1_bn.forward(conv_block_1_conv_out);
    float * conv_block_1_relu_out;
    float * conv_block_2_conv_out;
    if(sparse){
        conv_block_1_relu_out = conv_block_1_relu.forward(conv_block_1_bn_out);
        hipEvent_t event_start;
        hipEvent_t event_stop;
        hipEventCreate(&event_start);
        hipEventCreate(&event_stop);

        float temp_time;
        if(MEASUE_CUSNN){
            hipEventRecord(event_start);
            conv_block_2_conv_out = conv_block_2_conv.forward(conv_block_1_relu_out);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t1[index - 1] +=temp_time;
        }else{
            hipEventRecord(event_start);
            conv_block_2_conv_out = conv_block_2_conv_cudnn.forward(conv_block_1_relu_out);
            hipEventRecord(event_stop);
            hipEventSynchronize(event_stop);
            hipEventElapsedTime(&temp_time, event_start, event_stop);
            t2[index - 1] +=temp_time;
        }
    }else{
        conv_block_1_relu_out = conv_block_1_relu_cudnn.forward(conv_block_1_bn_out);
        conv_block_2_conv_out = conv_block_2_conv_cudnn.forward(conv_block_1_relu_out);
    }
    float * conv_block_concat_out = conv_block_concat.forward(input,conv_block_2_conv_out);
    return conv_block_concat_out;
}
