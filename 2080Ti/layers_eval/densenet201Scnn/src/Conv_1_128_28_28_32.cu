#include "../inc/scnnOps.h"
void Conv_1_128_28_28_32::initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                                     unsigned int pad,unsigned int r,unsigned int s,unsigned int stride,string weightFile) {
    this->B = b;
    this->C = c;
    this->H = h;
    this->W = w;
    this->N = n;
    this->hOut = h;
    this->wOut = w;
    unsigned int kernelSize = 3*3*C*N;//kernel + bias
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    try{
        load_input(weightFile,kernelSize,cpuKernel);
    }catch (const char* msg) {
        cerr << msg << endl;
    }
    float *temp_kernel = (float *)malloc(kernelSize*sizeof(float));
    for(unsigned int i=0;i<N;++i){
        for(unsigned int l=0;l<C;++l){
            for(unsigned int j=0;j<3;++j){
                for(unsigned int k=0;k<3;++k){
                    temp_kernel[l*N*3*3+j*3*N+k*N+i] = cpuKernel[i*3*3*C+l*9+j*3+k];
                }
            }
        }
    }
    hipMalloc(&kernel,9*C*N*sizeof(float));
    hipMemcpy(kernel,temp_kernel,9*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(temp_kernel);
    free(cpuKernel);
    TILES_EACH_CHANNEL = ((H-1)/TH+1)*((W-1)/TW+1);
    TCS = (C-1)/TC + 1;
    hipMalloc(&nnz,b*c*TILES_EACH_CHANNEL*(TH+3-1)*(TW+3-1)*sizeof(float));
    hipMalloc(&ids,b*c*TILES_EACH_CHANNEL*(TH+3-1)*(TW+3-1)*sizeof(char));
    hipMalloc(&c_lens,b*c*TILES_EACH_CHANNEL*sizeof(unsigned int));
    hipMalloc(&output,b*n*h*w*sizeof(float));
}
float * Conv_1_128_28_28_32::forward(float *input) {
    hipMemset(output, 0, B*N*hOut*wOut*sizeof(float));
    hipMemset(c_lens, 0, B*C*TILES_EACH_CHANNEL*sizeof(unsigned int));
    conv_1_128_28_28_32_transform<<<(B*C*H*W-1)/512+1,512>>>(input,this->nnz,this->ids,this->c_lens);
    conv_1_128_28_28_32_conv2d<<<B*TCS*TILES_EACH_CHANNEL,N>>>(this->nnz,this->c_lens,this->ids,this->kernel,this->output);

    //chkerr(hipGetLastError());
    //chkerr(hipDeviceSynchronize());
    return output;
}
