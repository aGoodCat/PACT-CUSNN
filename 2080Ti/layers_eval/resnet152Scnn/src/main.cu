#include "../inc/convBlk.h"
#include "../inc/nvml_helper.hpp"

int main(int argc,char *argv[]){
    float *input = (float *)malloc(224*224*3*sizeof(float));
    string imagePath = argv[1];
    //load_input(imagePath,3*224*224,input);
    float *dInput;
    hipMalloc(&dInput,224*224*3*sizeof(float));
    float *t1 = new float[100];
    float *t2 = new float[100];
    Conv conv1_conv;
    conv1_conv.initialize(1,3,224,224,64,3,7,7,2,"../../../weights/resnet152/weights/conv1_conv.bin");
    BatchNorm conv1_bn;
    conv1_bn.initialize(1,64,112,112,"../../../weights/resnet152/weights/conv1_bn.bin");
    Activation conv1_relu;
    conv1_relu.initialize(1,64,112,112);
    Pool conv1_max_pool;
    conv1_max_pool.initialize(1,64,112,112,1,3,3,HIPDNN_POOLING_MAX,2);

    ConvBlk1 conv2_block1(1,64,56,56,"../../../weights/resnet152/weights/conv2_block1_0_conv.bin","../../../weights/resnet152/weights/conv2_block1_0_bn.bin",
                          "../../../weights/resnet152/weights/conv2_block1_1_conv.bin","../../../weights/resnet152/weights/conv2_block1_1_bn.bin",
                          "../../../weights/resnet152/weights/conv2_block1_2_conv.bin","../../../weights/resnet152/weights/conv2_block1_2_bn.bin",
                          "../../../weights/resnet152/weights/conv2_block1_3_conv.bin","../../../weights/resnet152/weights/conv2_block1_3_bn.bin",
                          true,1,t1,t2);
    ConvBlk2 conv2_block2(1,256,56,56,"../../../weights/resnet152/weights/conv2_block2_1_conv.bin","../../../weights/resnet152/weights/conv2_block2_1_bn.bin",
                          "../../../weights/resnet152/weights/conv2_block2_2_conv.bin","../../../weights/resnet152/weights/conv2_block2_2_bn.bin",
                          "../../../weights/resnet152/weights/conv2_block2_3_conv.bin","../../../weights/resnet152/weights/conv2_block2_3_bn.bin",
                          true,2,t1,t2);
    ConvBlk2 conv2_block3(1,256,56,56,"../../../weights/resnet152/weights/conv2_block3_1_conv.bin","../../../weights/resnet152/weights/conv2_block3_1_bn.bin",
                          "../../../weights/resnet152/weights/conv2_block3_2_conv.bin","../../../weights/resnet152/weights/conv2_block3_2_bn.bin",
                          "../../../weights/resnet152/weights/conv2_block3_3_conv.bin","../../../weights/resnet152/weights/conv2_block3_3_bn.bin",
                          true,3,t1,t2);
    ConvBlk3 conv3_block1(1,256,56,56,"../../../weights/resnet152/weights/conv3_block1_0_conv.bin","../../../weights/resnet152/weights/conv3_block1_0_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block1_1_conv.bin","../../../weights/resnet152/weights/conv3_block1_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block1_2_conv.bin","../../../weights/resnet152/weights/conv3_block1_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block1_3_conv.bin","../../../weights/resnet152/weights/conv3_block1_3_bn.bin",
                          true,4,t1,t2);
    ConvBlk4 conv3_block2(1,512,28,28,"../../../weights/resnet152/weights/conv3_block2_1_conv.bin","../../../weights/resnet152/weights/conv3_block2_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block2_2_conv.bin","../../../weights/resnet152/weights/conv3_block2_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block2_3_conv.bin","../../../weights/resnet152/weights/conv3_block2_3_bn.bin",
                          true,5,t1,t2);
    ConvBlk4 conv3_block3(1,512,28,28,"../../../weights/resnet152/weights/conv3_block3_1_conv.bin","../../../weights/resnet152/weights/conv3_block3_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block3_2_conv.bin","../../../weights/resnet152/weights/conv3_block3_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block3_3_conv.bin","../../../weights/resnet152/weights/conv3_block3_3_bn.bin",
                          true,6,t1,t2);
    ConvBlk4 conv3_block4(1,512,28,28,"../../../weights/resnet152/weights/conv3_block4_1_conv.bin","../../../weights/resnet152/weights/conv3_block4_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block4_2_conv.bin","../../../weights/resnet152/weights/conv3_block4_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block4_3_conv.bin","../../../weights/resnet152/weights/conv3_block4_3_bn.bin",
                          true,7,t1,t2);
    ConvBlk4 conv3_block5(1,512,28,28,"../../../weights/resnet152/weights/conv3_block5_1_conv.bin","../../../weights/resnet152/weights/conv3_block5_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block5_2_conv.bin","../../../weights/resnet152/weights/conv3_block5_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block5_3_conv.bin","../../../weights/resnet152/weights/conv3_block5_3_bn.bin",
                          true,8,t1,t2);
    ConvBlk4 conv3_block6(1,512,28,28,"../../../weights/resnet152/weights/conv3_block6_1_conv.bin","../../../weights/resnet152/weights/conv3_block6_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block6_2_conv.bin","../../../weights/resnet152/weights/conv3_block6_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block6_3_conv.bin","../../../weights/resnet152/weights/conv3_block6_3_bn.bin",
                          true,9,t1,t2);
    ConvBlk4 conv3_block7(1,512,28,28,"../../../weights/resnet152/weights/conv3_block7_1_conv.bin","../../../weights/resnet152/weights/conv3_block7_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block7_2_conv.bin","../../../weights/resnet152/weights/conv3_block7_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block7_3_conv.bin","../../../weights/resnet152/weights/conv3_block7_3_bn.bin",
                          true,10,t1,t2);
    ConvBlk4 conv3_block8(1,512,28,28,"../../../weights/resnet152/weights/conv3_block8_1_conv.bin","../../../weights/resnet152/weights/conv3_block8_1_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block8_2_conv.bin","../../../weights/resnet152/weights/conv3_block8_2_bn.bin",
                          "../../../weights/resnet152/weights/conv3_block8_3_conv.bin","../../../weights/resnet152/weights/conv3_block8_3_bn.bin",
                          true,11,t1,t2);

    ConvBlk5 conv4_block1(1,512,28,28,"../../../weights/resnet152/weights/conv4_block1_0_conv.bin","../../../weights/resnet152/weights/conv4_block1_0_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block1_1_conv.bin","../../../weights/resnet152/weights/conv4_block1_1_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block1_2_conv.bin","../../../weights/resnet152/weights/conv4_block1_2_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block1_3_conv.bin","../../../weights/resnet152/weights/conv4_block1_3_bn.bin",
                          true,12,t1,t2);
    ConvBlk6 conv4_block2(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block2_1_conv.bin","../../../weights/resnet152/weights/conv4_block2_1_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block2_2_conv.bin","../../../weights/resnet152/weights/conv4_block2_2_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block2_3_conv.bin","../../../weights/resnet152/weights/conv4_block2_3_bn.bin",
                          true,13,t1,t2);
    ConvBlk6 conv4_block3(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block3_1_conv.bin","../../../weights/resnet152/weights/conv4_block3_1_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block3_2_conv.bin","../../../weights/resnet152/weights/conv4_block3_2_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block3_3_conv.bin","../../../weights/resnet152/weights/conv4_block3_3_bn.bin",
                          true,14,t1,t2);
    ConvBlk6 conv4_block4(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block4_1_conv.bin","../../../weights/resnet152/weights/conv4_block4_1_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block4_2_conv.bin","../../../weights/resnet152/weights/conv4_block4_2_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block4_3_conv.bin","../../../weights/resnet152/weights/conv4_block4_3_bn.bin",
                          true,15,t1,t2);
    ConvBlk6 conv4_block5(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block5_1_conv.bin","../../../weights/resnet152/weights/conv4_block5_1_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block5_2_conv.bin","../../../weights/resnet152/weights/conv4_block5_2_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block5_3_conv.bin","../../../weights/resnet152/weights/conv4_block5_3_bn.bin",
                          true,16,t1,t2);
    ConvBlk6 conv4_block6(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block6_1_conv.bin","../../../weights/resnet152/weights/conv4_block6_1_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block6_2_conv.bin","../../../weights/resnet152/weights/conv4_block6_2_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block6_3_conv.bin","../../../weights/resnet152/weights/conv4_block6_3_bn.bin",
                          true,17,t1,t2);
    ConvBlk6 conv4_block7(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block7_1_conv.bin","../../../weights/resnet152/weights/conv4_block7_1_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block7_2_conv.bin","../../../weights/resnet152/weights/conv4_block7_2_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block7_3_conv.bin","../../../weights/resnet152/weights/conv4_block7_3_bn.bin",
                          true,18,t1,t2);
    ConvBlk6 conv4_block8(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block8_1_conv.bin","../../../weights/resnet152/weights/conv4_block8_1_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block8_2_conv.bin","../../../weights/resnet152/weights/conv4_block8_2_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block8_3_conv.bin","../../../weights/resnet152/weights/conv4_block8_3_bn.bin",
                          true,19,t1,t2);
    ConvBlk6 conv4_block9(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block9_1_conv.bin","../../../weights/resnet152/weights/conv4_block9_1_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block9_2_conv.bin","../../../weights/resnet152/weights/conv4_block9_2_bn.bin",
                          "../../../weights/resnet152/weights/conv4_block9_3_conv.bin","../../../weights/resnet152/weights/conv4_block9_3_bn.bin",
                          true,20,t1,t2);
    ConvBlk6 conv4_block10(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block10_1_conv.bin","../../../weights/resnet152/weights/conv4_block10_1_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block10_2_conv.bin","../../../weights/resnet152/weights/conv4_block10_2_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block10_3_conv.bin","../../../weights/resnet152/weights/conv4_block10_3_bn.bin",
                           true,21,t1,t2);
    ConvBlk6 conv4_block11(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block11_1_conv.bin","../../../weights/resnet152/weights/conv4_block11_1_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block11_2_conv.bin","../../../weights/resnet152/weights/conv4_block11_2_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block11_3_conv.bin","../../../weights/resnet152/weights/conv4_block11_3_bn.bin",
                           true,22,t1,t2);
    ConvBlk6 conv4_block12(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block12_1_conv.bin","../../../weights/resnet152/weights/conv4_block12_1_bn.bin","../../../weights/resnet152/weights/conv4_block12_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block12_2_bn.bin","../../../weights/resnet152/weights/conv4_block12_3_conv.bin","../../../weights/resnet152/weights/conv4_block12_3_bn.bin",
                           true,23,t1,t2);
    ConvBlk6 conv4_block13(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block13_1_conv.bin","../../../weights/resnet152/weights/conv4_block13_1_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block13_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block13_2_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block13_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block13_3_bn.bin",
                           true,24,t1,t2);
    ConvBlk6 conv4_block14(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block14_1_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block14_1_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block14_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block14_2_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block14_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block14_3_bn.bin",
                           true,25,t1,t2);
    ConvBlk6 conv4_block15(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block15_1_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block15_1_bn.bin","../../../weights/resnet152/weights/conv4_block15_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block15_2_bn.bin","../../../weights/resnet152/weights/conv4_block15_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block15_3_bn.bin",
                           true,26,t1,t2);
    ConvBlk6 conv4_block16(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block16_1_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block16_1_bn.bin","../../../weights/resnet152/weights/conv4_block16_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block16_2_bn.bin","../../../weights/resnet152/weights/conv4_block16_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block16_3_bn.bin",
                           true,27,t1,t2);
    ConvBlk6 conv4_block17(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block17_1_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block17_1_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block17_2_conv.bin","../../../weights/resnet152/weights/conv4_block17_2_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block17_3_conv.bin","../../../weights/resnet152/weights/conv4_block17_3_bn.bin",
                           true,28,t1,t2);
    ConvBlk6 conv4_block18(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block18_1_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block18_1_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block18_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block18_2_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block18_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block18_3_bn.bin",
                           true,29,t1,t2);
    ConvBlk6 conv4_block19(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block19_1_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block19_1_bn.bin","../../../weights/resnet152/weights/conv4_block19_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block19_2_bn.bin","../../../weights/resnet152/weights/conv4_block19_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block19_3_bn.bin",
                           true,30,t1,t2);
    ConvBlk6 conv4_block20(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block20_1_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block20_1_bn.bin","../../../weights/resnet152/weights/conv4_block20_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block20_2_bn.bin","../../../weights/resnet152/weights/conv4_block20_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block20_3_bn.bin",
                           true,31,t1,t2);
    ConvBlk6 conv4_block21(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block21_1_conv.bin","../../../weights/resnet152/weights/conv4_block21_1_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block21_2_conv.bin","../../../weights/resnet152/weights/conv4_block21_2_bn.bin","../../../weights/resnet152/weights/conv4_block21_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block21_3_bn.bin",
                           true,32,t1,t2);
    ConvBlk6 conv4_block22(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block22_1_conv.bin","../../../weights/resnet152/weights/conv4_block22_1_bn.bin","../../../weights/resnet152/weights/conv4_block22_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block22_2_bn.bin","../../../weights/resnet152/weights/conv4_block22_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block22_3_bn.bin",
                           true,33,t1,t2);
    ConvBlk6 conv4_block23(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block23_1_conv.bin","../../../weights/resnet152/weights/conv4_block23_1_bn.bin","../../../weights/resnet152/weights/conv4_block23_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block23_2_bn.bin","../../../weights/resnet152/weights/conv4_block23_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block23_3_bn.bin",
                           true,34,t1,t2);
    ConvBlk6 conv4_block24(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block24_1_conv.bin","../../../weights/resnet152/weights/conv4_block24_1_bn.bin","../../../weights/resnet152/weights/conv4_block24_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block24_2_bn.bin","../../../weights/resnet152/weights/conv4_block24_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block24_3_bn.bin",
                           true,35,t1,t2);
    ConvBlk6 conv4_block25(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block25_1_conv.bin","../../../weights/resnet152/weights/conv4_block25_1_bn.bin","../../../weights/resnet152/weights/conv4_block25_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block25_2_bn.bin","../../../weights/resnet152/weights/conv4_block25_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block25_3_bn.bin",
                           true,36,t1,t2);
    ConvBlk6 conv4_block26(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block26_1_conv.bin","../../../weights/resnet152/weights/conv4_block26_1_bn.bin","../../../weights/resnet152/weights/conv4_block26_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block26_2_bn.bin","../../../weights/resnet152/weights/conv4_block26_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block26_3_bn.bin",
                           true,37,t1,t2);
    ConvBlk6 conv4_block27(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block27_1_conv.bin","../../../weights/resnet152/weights/conv4_block27_1_bn.bin","../../../weights/resnet152/weights/conv4_block27_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block27_2_bn.bin","../../../weights/resnet152/weights/conv4_block27_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block27_3_bn.bin",
                           true,38,t1,t2);
    ConvBlk6 conv4_block28(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block28_1_conv.bin","../../../weights/resnet152/weights/conv4_block28_1_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block28_2_conv.bin","../../../weights/resnet152/weights/conv4_block28_2_bn.bin","../../../weights/resnet152/weights/conv4_block28_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block28_3_bn.bin",
                           true,39,t1,t2);
    ConvBlk6 conv4_block29(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block29_1_conv.bin","../../../weights/resnet152/weights/conv4_block29_1_bn.bin","../../../weights/resnet152/weights/conv4_block29_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block29_2_bn.bin","../../../weights/resnet152/weights/conv4_block29_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block29_3_bn.bin",
                           true,40,t1,t2);
    ConvBlk6 conv4_block30(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block30_1_conv.bin","../../../weights/resnet152/weights/conv4_block30_1_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block30_2_conv.bin","../../../weights/resnet152/weights/conv4_block30_2_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block30_3_conv.bin","../../../weights/resnet152/weights/conv4_block30_3_bn.bin",
                           true,41,t1,t2);
    ConvBlk6 conv4_block31(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block31_1_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block31_1_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block31_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block31_2_bn.bin","../../../weights/resnet152/weights/conv4_block31_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block31_3_bn.bin",
                           true,42,t1,t2);
    ConvBlk6 conv4_block32(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block32_1_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block32_1_bn.bin","../../../weights/resnet152/weights/conv4_block32_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block32_2_bn.bin","../../../weights/resnet152/weights/conv4_block32_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block32_3_bn.bin",
                           true,43,t1,t2);
    ConvBlk6 conv4_block33(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block33_1_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block33_1_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block33_2_conv.bin","../../../weights/resnet152/weights/conv4_block33_2_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block33_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block33_3_bn.bin",
                           true,44,t1,t2);
    ConvBlk6 conv4_block34(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block34_1_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block34_1_bn.bin","../../../weights/resnet152/weights/conv4_block34_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block34_2_bn.bin","../../../weights/resnet152/weights/conv4_block34_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block34_3_bn.bin",
                           true,45,t1,t2);
    ConvBlk6 conv4_block35(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block35_1_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block35_1_bn.bin","../../../weights/resnet152/weights/conv4_block35_2_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block35_2_bn.bin","../../../weights/resnet152/weights/conv4_block35_3_conv.bin",
                           "../../../weights/resnet152/weights/conv4_block35_3_bn.bin",
                           true,46,t1,t2);
    ConvBlk6 conv4_block36(1,1024,14,14,"../../../weights/resnet152/weights/conv4_block36_1_conv.bin","../../../weights/resnet152/weights/conv4_block36_1_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block36_2_conv.bin","../../../weights/resnet152/weights/conv4_block36_2_bn.bin",
                           "../../../weights/resnet152/weights/conv4_block36_3_conv.bin","../../../weights/resnet152/weights/conv4_block36_3_bn.bin",
                           true,47,t1,t2);
    ConvBlk7 conv5_block1(1,1024,14,14,"../../../weights/resnet152/weights/conv5_block1_0_conv.bin","../../../weights/resnet152/weights/conv5_block1_0_bn.bin",
                          "../../../weights/resnet152/weights/conv5_block1_1_conv.bin","../../../weights/resnet152/weights/conv5_block1_1_bn.bin",
                          "../../../weights/resnet152/weights/conv5_block1_2_conv.bin","../../../weights/resnet152/weights/conv5_block1_2_bn.bin",
                          "../../../weights/resnet152/weights/conv5_block1_3_conv.bin",
                          "../../../weights/resnet152/weights/conv5_block1_3_bn.bin",
                          true,48,t1,t2);
    ConvBlk8 conv5_block2(1,2048,7,7,"../../../weights/resnet152/weights/conv5_block2_1_conv.bin",
                          "../../../weights/resnet152/weights/conv5_block2_1_bn.bin",
                          "../../../weights/resnet152/weights/conv5_block2_2_conv.bin","../../../weights/resnet152/weights/conv5_block2_2_bn.bin",
                          "../../../weights/resnet152/weights/conv5_block2_3_conv.bin","../../../weights/resnet152/weights/conv5_block2_3_bn.bin",
                          true,49,t1,t2);
    ConvBlk8 conv5_block3(1,2048,7,7,"../../../weights/resnet152/weights/conv5_block3_1_conv.bin","../../../weights/resnet152/weights/conv5_block3_1_bn.bin",
                          "../../../weights/resnet152/weights/conv5_block3_2_conv.bin","../../../weights/resnet152/weights/conv5_block3_2_bn.bin",
                          "../../../weights/resnet152/weights/conv5_block3_3_conv.bin",
                          "../../../weights/resnet152/weights/conv5_block3_3_bn.bin",
                          true,50,t1,t2);

    Pool avg_pool;
    avg_pool.initialize(1,2048,7,7,0,7,7,HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING,1);
    Conv predict;
    predict.initialize(1,2048,1,1,1000,0,1,1,1,"../../../weights/resnet152/weights/predictions.bin");

    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    hipMemcpy(dInput, input, 224 * 224 * 3 * sizeof(float), hipMemcpyHostToDevice);
    float *output;
    output = conv1_conv.forward(dInput);
    output = conv1_bn.forward(output);
    output = conv1_relu.forward(output);
    output = conv1_max_pool.forward(output);
    output = conv2_block1.forward(output);
    output = conv2_block2.forward(output);
    output = conv2_block3.forward(output);
    output = conv3_block1.forward(output);
    output = conv3_block2.forward(output);
    output = conv3_block3.forward(output);
    output = conv3_block4.forward(output);
    output = conv3_block5.forward(output);
    output = conv3_block6.forward(output);
    output = conv3_block7.forward(output);
    output = conv3_block8.forward(output);
    output = conv4_block1.forward(output);
    output = conv4_block2.forward(output);
    output = conv4_block3.forward(output);
    output = conv4_block4.forward(output);
    output = conv4_block5.forward(output);
    output = conv4_block6.forward(output);
    output = conv4_block7.forward(output);
    output = conv4_block8.forward(output);
    output = conv4_block9.forward(output);
    output = conv4_block10.forward(output);
    output = conv4_block11.forward(output);
    output = conv4_block12.forward(output);
    output = conv4_block13.forward(output);
    output = conv4_block14.forward(output);
    output = conv4_block15.forward(output);
    output = conv4_block16.forward(output);
    output = conv4_block17.forward(output);
    output = conv4_block18.forward(output);
    output = conv4_block19.forward(output);
    output = conv4_block20.forward(output);
    output = conv4_block21.forward(output);
    output = conv4_block22.forward(output);
    output = conv4_block23.forward(output);
    output = conv4_block24.forward(output);
    output = conv4_block25.forward(output);
    output = conv4_block26.forward(output);
    output = conv4_block27.forward(output);
    output = conv4_block28.forward(output);
    output = conv4_block29.forward(output);
    output = conv4_block30.forward(output);
    output = conv4_block31.forward(output);
    output = conv4_block32.forward(output);
    output = conv4_block33.forward(output);
    output = conv4_block34.forward(output);
    output = conv4_block35.forward(output);
    output = conv4_block36.forward(output);
    output = conv5_block1.forward(output);
    output = conv5_block2.forward(output);
    output = conv5_block3.forward(output);
    output = avg_pool.forward(output);
    float total = 0.0f;
    for(int i=0;i<100;++i){
        t1[i] = 0.0f;
        t2[i] = 0.0f;
    }
    for(int i=0;i<100;++i) {
        load_input(imagePath + to_string(i) + ".bin", 3 * 224 * 224, input);
        hipMemcpy(dInput,input,224*224*3*sizeof(float),hipMemcpyHostToDevice);
        hipEventRecord(event_start);
        output = conv1_conv.forward(dInput);
        output = conv1_bn.forward(output);
        output = conv1_relu.forward(output);
        output = conv1_max_pool.forward(output);
        output = conv2_block1.forward(output);
        output = conv2_block2.forward(output);
        output = conv2_block3.forward(output);
        output = conv3_block1.forward(output);
        output = conv3_block2.forward(output);
        output = conv3_block3.forward(output);
        output = conv3_block4.forward(output);
        output = conv3_block5.forward(output);
        output = conv3_block6.forward(output);
        output = conv3_block7.forward(output);
        output = conv3_block8.forward(output);
        output = conv4_block1.forward(output);
        output = conv4_block2.forward(output);
        output = conv4_block3.forward(output);
        output = conv4_block4.forward(output);
        output = conv4_block5.forward(output);
        output = conv4_block6.forward(output);
        output = conv4_block7.forward(output);
        output = conv4_block8.forward(output);
        output = conv4_block9.forward(output);
        output = conv4_block10.forward(output);
        output = conv4_block11.forward(output);
        output = conv4_block12.forward(output);
        output = conv4_block13.forward(output);
        output = conv4_block14.forward(output);
        output = conv4_block15.forward(output);
        output = conv4_block16.forward(output);
        output = conv4_block17.forward(output);
        output = conv4_block18.forward(output);
        output = conv4_block19.forward(output);
        output = conv4_block20.forward(output);
        output = conv4_block21.forward(output);
        output = conv4_block22.forward(output);
        output = conv4_block23.forward(output);
        output = conv4_block24.forward(output);
        output = conv4_block25.forward(output);
        output = conv4_block26.forward(output);
        output = conv4_block27.forward(output);
        output = conv4_block28.forward(output);
        output = conv4_block29.forward(output);
        output = conv4_block30.forward(output);
        output = conv4_block31.forward(output);
        output = conv4_block32.forward(output);
        output = conv4_block33.forward(output);
        output = conv4_block34.forward(output);
        output = conv4_block35.forward(output);
        output = conv4_block36.forward(output);
        output = conv5_block1.forward(output);
        output = conv5_block2.forward(output);
        output = conv5_block3.forward(output);
        output = avg_pool.forward(output);
        output = predict.forward(output);
        hipDeviceSynchronize();
        float cuda_time;
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        hipEventElapsedTime(&cuda_time, event_start, event_stop);
        total += cuda_time;
    }
    std::ofstream file_out;
    file_out.open("resnet152_layers.txt", std::ofstream::out);
    for(int i=0;i<50;++i){
        string out = to_string(i+1) + "," + to_string(t1[i]/100);
        file_out << out<<endl;
    }
    file_out.close();
    unsigned int outputSize = 1000;
    float *hOutput = (float *)malloc(outputSize*sizeof(float));
    hipMemcpy(hOutput,output,outputSize*sizeof(float),hipMemcpyDeviceToHost);
    vector<float> out_bin;
    for(int i=0;i<outputSize;++i){
        out_bin.push_back(hOutput[i]);
    }
    std::ofstream ofp("s.bin", std::ios::out | std::ios::binary);
    ofp.write(reinterpret_cast<const char*>(out_bin.data()), out_bin.size() * sizeof(float));
    return 0;
}
