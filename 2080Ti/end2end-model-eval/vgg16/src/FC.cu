#include "../inc/cudnnOps.h"
void FC::initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                    unsigned int pad,unsigned int r,unsigned int s,unsigned int stride,string weightFile){
    this->B = b;
    this->C = c;
    this->H = h;
    this->W = w;
    this->N = n;
    this->R = r;
    this->S = s;
    this->hOut = (H+2*pad - r)/stride + 1;
    this->wOut = (W+2*pad - s)/stride + 1;
    hipMalloc(&kernel,sizeof(float)*C*N*R*S);
    hipMalloc(&bias,sizeof(float)*N);
    hipMalloc(&this->output,sizeof(float)*B*hOut*wOut*N);
    hipdnnCreate(&convCudnn);
    hipdnnCreateTensorDescriptor(&convInputDescriptor);
    hipdnnCreateTensorDescriptor(&biasDescriptor);
    hipdnnSetTensor4dDescriptor(biasDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/1,
            /*image_width=*/1);
    hipdnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    hipdnnCreateFilterDescriptor(&convKernelDescriptor);
    hipdnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    hipdnnCreateConvolutionDescriptor(&convDesc);
    hipdnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/pad,
            /*pad_width=*/pad,
            /*vertical_stride=*/stride,
            /*horizontal_stride=*/stride,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/HIPDNN_CROSS_CORRELATION,
                                    HIPDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    hipdnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    hipdnnCreateTensorDescriptor(&convOutputDescriptor);
    hipdnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/N,
            /*image_height=*/hOut,
            /*image_width=*/wOut);
    hipdnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N + N;//kernel + bias
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    try{
        load_input(weightFile,kernelSize,cpuKernel);
    }catch (const char* msg) {
        cerr << msg << endl;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(bias,&cpuKernel[R*S*C*N],N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * FC::forward(float *input) {
    hipMemset(output, 0, B*N*hOut*wOut*sizeof(float));
    checkCUDNN(hipdnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    checkCUDNN(hipdnnAddTensor(convCudnn,&alpha,biasDescriptor,bias,&beta2,convOutputDescriptor,output));
    return output;
}