
#include "../inc/conv_and_pool.h"
#include "../inc/cudnnOps.h"
Conv_and_pool::Conv_and_pool(unsigned int b, unsigned int c, unsigned int h, unsigned int w,unsigned int n1,unsigned int n2,
                             string conv1Weight, string conv2Weight){
    conv1.initialize(b,c,h,w,n1,1,3,3,1,conv1Weight);
    conv2.initialize(b,n1,h,w,n2,1,3,3,1,conv2Weight);
    relu1.initialize(b,n1,h,w);
    relu2.initialize(b,n2,h,w);
    pool.initialize(1,n2,h,w,1,3,3,HIPDNN_POOLING_MAX,2);
}

float *Conv_and_pool::forward(float *input) {
    float *x = conv1.forward(input);
    x = relu1.forward(x);
    x = conv2.forward(x);
    x = relu2.forward(x);
    x = pool.forward(x);
    return x;
}
Conv_and_pool3::Conv_and_pool3(unsigned int b, unsigned int c, unsigned int h, unsigned int w,
                               unsigned int n1,unsigned int n2,unsigned int n3, string conv1Weight,
                               string conv2Weight, string conv3Weight){
    conv1.initialize(b,c,h,w,n1,1,3,3,1,conv1Weight);
    conv2.initialize(b,n1,h,w,n2,1,3,3,1,conv2Weight);
    conv3.initialize(b,n2,h,w,n3,1,3,3,1,conv3Weight);
    relu1.initialize(b,n1,h,w);
    relu2.initialize(b,n2,h,w);
    relu3.initialize(b,n3,h,w);
    pool.initialize(1,n3,h,w,1,3,3,HIPDNN_POOLING_MAX,2);
}

float *Conv_and_pool3::forward(float *input) {
    float *x = conv1.forward(input);
    x = relu1.forward(x);
    x = conv2.forward(x);
    x = relu2.forward(x);
    x = conv3.forward(x);
    x = relu3.forward(x);
    x = pool.forward(x);
    return x;
}