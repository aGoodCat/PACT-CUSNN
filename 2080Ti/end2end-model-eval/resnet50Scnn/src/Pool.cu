#include "../inc/cudnnOps.h"
void Pool::initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int pad,unsigned int windowH,unsigned int windowW,
                      hipdnnPoolingMode_t mode,unsigned int stride) {
    B = b;
    C = c;
    H = h;
    W = w;
    hOut = (h - windowH + 2*pad)/stride + 1;
    wOut = (w - windowW + 2*pad)/stride + 1;
    checkCUDNN(hipdnnCreate(&poolingCudnn));
    checkCUDNN(hipdnnCreateTensorDescriptor(&poolingInputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(poolingInputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
    checkCUDNN(hipdnnCreateTensorDescriptor(&poolingOutputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(poolingOutputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/hOut,
            /*image_width=*/wOut));
    checkCUDNN(hipdnnCreatePoolingDescriptor(&poolingDesc));
    hipdnnSetPooling2dDescriptor(poolingDesc,mode,HIPDNN_NOT_PROPAGATE_NAN,windowH,windowW,
                                pad,pad,stride,stride);
    hipMalloc(&output,B*C*hOut*wOut*sizeof(float));
}
float * Pool::forward(float *input) {
    checkCUDNN(hipdnnPoolingForward(
            poolingCudnn,
            poolingDesc,
            &alpha,
            poolingInputDescriptor,
            input,
            &beta,
            poolingOutputDescriptor,
            output));
    return output;
}