#include "hip/hip_runtime.h"
#include "../inc/common.h"
#include "../inc/scnn.h"
__global__ void relu(float * __restrict__ values, unsigned int B,unsigned int C,unsigned int H,
                     unsigned int W,unsigned int *counter){
    __shared__ unsigned int block_count[1];
    block_count[0] = 0;
    __syncthreads();
    unsigned int local_count = 0;
    unsigned int lane_id = threadIdx.x % 32;
    for(unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;i<B*C*H*W;i+=gridDim.x*blockDim.x){
        float value = values[i];
        if(value<=0.0f){
            values[i] = 0.0f;
            local_count ++;
        }
    }
    for(int j=16;j>0;j=j/2){
        local_count += __shfl_down_sync(0xFFFFFFFF,local_count,j);
    }
    if(lane_id == 0&&local_count>0){
        atomicAdd(&block_count[0],local_count);
    }
    __syncthreads();
    if(threadIdx.x == 0){
        atomicAdd(&counter[0],block_count[0]);
    }
}