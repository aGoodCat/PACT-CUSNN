#include "../inc/cudnnOps.h"
void BatchNorm::initialize(unsigned int b, unsigned int c, unsigned int h, unsigned int w,string weight) {
    B = b;
    H = h;
    W = w;
    C = c;
    chkerr(hipMalloc(&scaleDev,C*sizeof(float)));
    chkerr(hipMalloc(&shiftDev,C*sizeof(float)));
    chkerr(hipMalloc(&meanDev,C*sizeof(float)));
    chkerr(hipMalloc(&varDev,C*sizeof(float)));
    checkCUDNN(hipdnnCreate(&batchNormCudnn));
    hipMalloc(&output,B*C*H*W*sizeof(float));
    checkCUDNN(hipdnnCreateTensorDescriptor(&batchNormInputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(batchNormInputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
    checkCUDNN(hipdnnCreateTensorDescriptor(&batchNormOutputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(batchNormOutputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
    checkCUDNN(hipdnnCreateTensorDescriptor(&bnScaleBiasMeanVarDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(bnScaleBiasMeanVarDesc,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/1,
            /*image_width=*/1));

    this->cpuKernel = (float *)malloc(4*C*sizeof(float));
    //load_input(weight,4*C,cpuKernel);
    try{
        load_input(weight,4*C,cpuKernel);
    }catch (const char* msg) {
        cerr << msg << endl;
    }
    chkerr(hipMemcpy(scaleDev,cpuKernel,C*sizeof(float),hipMemcpyHostToDevice));
    chkerr(hipMemcpy(shiftDev,&cpuKernel[C],C*sizeof(float),hipMemcpyHostToDevice));
    chkerr(hipMemcpy(meanDev,&cpuKernel[2*C],C*sizeof(float),hipMemcpyHostToDevice));
    chkerr(hipMemcpy(varDev,&cpuKernel[3*C],C*sizeof(float),hipMemcpyHostToDevice));
    free(cpuKernel);
}
float * BatchNorm::forward(float *input) {
    checkCUDNN(hipdnnBatchNormalizationForwardInference(
            batchNormCudnn,
            HIPDNN_BATCHNORM_SPATIAL,
            &alpha,
            &beta,
            batchNormInputDescriptor,
            input, //gpu上的
            batchNormOutputDescriptor,
            output, //gpu上的
            bnScaleBiasMeanVarDesc,
            scaleDev,  //gpu上的
            shiftDev,    //gpu上的
            meanDev,  //gpu上的
            varDev,//gpu上的
            HIPDNN_BN_MIN_EPSILON
    ));
    return output;
}