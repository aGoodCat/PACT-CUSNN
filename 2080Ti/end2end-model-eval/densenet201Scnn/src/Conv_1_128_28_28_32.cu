#include "hip/hip_runtime.h"
#include "../inc/scnnOps.h"

__device__ void conv_1_128_28_28_32_switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
    switch (switch_condition) {
        case 0:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(0-s)] += result;
                }
            }
            break;
        case 1:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(1-s)] += result;
                }
            }
            break;
        case 2:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(2-s)] += result;
                }
            }
            break;
        case 3:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(3-s)] += result;
                }
            }
            break;
        case 4:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(4-s)] += result;
                }
            }
            break;
        case 5:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(5-s)] += result;
                }
            }
            break;
        case 6:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(6-s)] += result;
                }
            }
            break;
        case 7:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(7-s)] += result;
                }
            }
            break;
        case 8:
            for ( int r = 0; r < 1; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(0-r)*7+(8-s)] += result;
                }
            }
            break;
        case 9:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(0-s)] += result;
                }
            }
            break;
        case 10:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(1-s)] += result;
                }
            }
            break;
        case 11:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(2-s)] += result;
                }
            }
            break;
        case 12:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(3-s)] += result;
                }
            }
            break;
        case 13:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(4-s)] += result;
                }
            }
            break;
        case 14:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(5-s)] += result;
                }
            }
            break;
        case 15:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(6-s)] += result;
                }
            }
            break;
        case 16:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(7-s)] += result;
                }
            }
            break;
        case 17:
            for ( int r = 0; r < 2; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(1-r)*7+(8-s)] += result;
                }
            }
            break;
        case 18:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(0-s)] += result;
                }
            }
            break;
        case 19:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(1-s)] += result;
                }
            }
            break;
        case 20:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(2-s)] += result;
                }
            }
            break;
        case 21:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(3-s)] += result;
                }
            }
            break;
        case 22:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(4-s)] += result;
                }
            }
            break;
        case 23:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(5-s)] += result;
                }
            }
            break;
        case 24:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(6-s)] += result;
                }
            }
            break;
        case 25:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(7-s)] += result;
                }
            }
            break;
        case 26:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(2-r)*7+(8-s)] += result;
                }
            }
            break;
        case 27:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(0-s)] += result;
                }
            }
            break;
        case 28:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(1-s)] += result;
                }
            }
            break;
        case 29:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(2-s)] += result;
                }
            }
            break;
        case 30:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(3-s)] += result;
                }
            }
            break;
        case 31:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(4-s)] += result;
                }
            }
            break;
        case 32:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(5-s)] += result;
                }
            }
            break;
        case 33:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(6-s)] += result;
                }
            }
            break;
        case 34:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(7-s)] += result;
                }
            }
            break;
        case 35:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(3-r)*7+(8-s)] += result;
                }
            }
            break;
        case 36:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(0-s)] += result;
                }
            }
            break;
        case 37:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(1-s)] += result;
                }
            }
            break;
        case 38:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(2-s)] += result;
                }
            }
            break;
        case 39:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(3-s)] += result;
                }
            }
            break;
        case 40:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(4-s)] += result;
                }
            }
            break;
        case 41:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(5-s)] += result;
                }
            }
            break;
        case 42:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(6-s)] += result;
                }
            }
            break;
        case 43:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(7-s)] += result;
                }
            }
            break;
        case 44:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(4-r)*7+(8-s)] += result;
                }
            }
            break;
        case 45:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(0-s)] += result;
                }
            }
            break;
        case 46:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(1-s)] += result;
                }
            }
            break;
        case 47:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(2-s)] += result;
                }
            }
            break;
        case 48:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(3-s)] += result;
                }
            }
            break;
        case 49:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(4-s)] += result;
                }
            }
            break;
        case 50:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(5-s)] += result;
                }
            }
            break;
        case 51:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(6-s)] += result;
                }
            }
            break;
        case 52:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(7-s)] += result;
                }
            }
            break;
        case 53:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(5-r)*7+(8-s)] += result;
                }
            }
            break;
        case 54:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(0-s)] += result;
                }
            }
            break;
        case 55:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(1-s)] += result;
                }
            }
            break;
        case 56:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(2-s)] += result;
                }
            }
            break;
        case 57:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(3-s)] += result;
                }
            }
            break;
        case 58:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(4-s)] += result;
                }
            }
            break;
        case 59:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(5-s)] += result;
                }
            }
            break;
        case 60:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(6-s)] += result;
                }
            }
            break;
        case 61:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(7-s)] += result;
                }
            }
            break;
        case 62:
            for ( int r = 0; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(6-r)*7+(8-s)] += result;
                }
            }
            break;
        case 63:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(7-r)*7+(0-s)] += result;
                }
            }
            break;
        case 64:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(7-r)*7+(1-s)] += result;
                }
            }
            break;
        case 65:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(7-r)*7+(2-s)] += result;
                }
            }
            break;
        case 66:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(7-r)*7+(3-s)] += result;
                }
            }
            break;
        case 67:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(7-r)*7+(4-s)] += result;
                }
            }
            break;
        case 68:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(7-r)*7+(5-s)] += result;
                }
            }
            break;
        case 69:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(7-r)*7+(6-s)] += result;
                }
            }
            break;
        case 70:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(7-r)*7+(7-s)] += result;
                }
            }
            break;
        case 71:
            for ( int r = 1; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(7-r)*7+(8-s)] += result;
                }
            }
            break;
        case 72:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 1; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(8-r)*7+(0-s)] += result;
                }
            }
            break;
        case 73:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 2; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(8-r)*7+(1-s)] += result;
                }
            }
            break;
        case 74:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(8-r)*7+(2-s)] += result;
                }
            }
            break;
        case 75:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(8-r)*7+(3-s)] += result;
                }
            }
            break;
        case 76:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(8-r)*7+(4-s)] += result;
                }
            }
            break;
        case 77:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(8-r)*7+(5-s)] += result;
                }
            }
            break;
        case 78:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 0; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(8-r)*7+(6-s)] += result;
                }
            }
            break;
        case 79:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 1; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(8-r)*7+(7-s)] += result;
                }
            }
            break;
        case 80:
            for ( int r = 2; r < 3; r++) {
                for ( int s = 2; s < 3; s++) {
                    float result = v * temp_kernel[r*3+s];
                    temp_result[(8-r)*7+(8-s)] += result;
                }
            }
            break;

    }
}
__global__ void conv_1_128_28_28_32_transform(float *matrix, float *d_nnz, char *ids, unsigned int *c_lens){
    unsigned int global_id = blockIdx.x * 512 + threadIdx.x;
    unsigned int batch_id = global_id/(128*28*28);
    if(global_id >= 1*28*28*128){
        return ;
    }
    const float v = matrix[global_id];
    if(v == 0.0f){
        return;
    }
    unsigned int c = global_id % 128;

    int h = ((global_id - batch_id * 128 * 28 * 28)/128)/28+1;
    int w = ((global_id - batch_id * 128 * 28 * 28)/128)%28+1;
    int th_start = min(h/7,4-1);
    int tw_start = min(w/7,4-1);
    for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
        if((tile_h_id*7+7+3-1)<=h){
            break;
        }
        for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
            if((tile_w_id*7+7+3-1)<=w){
                break;
            }
            unsigned int tile_id = tile_h_id * 4 + tile_w_id;
            unsigned int index = atomicAdd(&c_lens[batch_id*128*16+c*16+tile_id],1);
            unsigned int abs_h = h - tile_h_id*7;
            unsigned int abs_w = w - tile_w_id*7;
            d_nnz[batch_id*128*16*(7+3-1)*(7+3-1)+
                  c*16*(7+3-1)*(7+3-1)+tile_id*(7+3-1)*(7+3-1)+index] = v;
            ids[batch_id*128*16*(7+3-1)*(7+3-1)+
                c*16*(7+3-1)*(7+3-1)+tile_id*(7+3-1)*(7+3-1)+index] = abs_h*(7+3-1)+abs_w;
        }
    }
}
__device__ void conv_1_128_28_28_32_load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<3;++r){
        for(unsigned int s=0;s<3;++s){
            data_array[r*3+s] = kernel[c_index*32*3*3+r*3*32+s*32+n_id];
        }
    }
}
__device__ void conv_1_128_28_28_32_load_input_2_shared_memory(float *values,unsigned int *c_lens,char *ids,
                                           float *shared_input,char *shared_ids,unsigned int *shared_lens,
                                           unsigned int warp_id,unsigned int lane_id,unsigned int batch_id,
                                           unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<2&&tile_c_id+c_id<128;c_id+=1){
        unsigned int end_index = c_lens[batch_id*128*16+(tile_c_id+c_id)*16+tile_id];
        if(lane_id ==0){
            shared_lens[c_id] = end_index;
        }
        for(unsigned int id = lane_id;id<end_index;id+=32){
            shared_input[c_id*(7+3-1)*(7+3-1)+id] = values[batch_id*128*16*(7+3-1)*(7+3-1)+
                                                           (tile_c_id+c_id)*16*(7+3-1)*(7+3-1)+tile_id*(7+3-1)*(7+3-1)+id];
            shared_ids[c_id*(7+3-1)*(7+3-1)+id] = ids[batch_id*128*16*(7+3-1)*(7+3-1)+
                                                      (tile_c_id+c_id)*16*(7+3-1)*(7+3-1)+tile_id*(7+3-1)*(7+3-1)+id];
        }
    }
}
__global__ void conv_1_128_28_28_32_conv2d(float * __restrict__ values, unsigned int * __restrict__ c_lens,
                       char * __restrict__ ids,
                       const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[2*(7+3-1)*(7+3-1)];
    __shared__ char input_ids[2*(7+3-1)*(7+3-1)];
    __shared__ unsigned int channel_lens[(2)];

    const unsigned int batch_id = (blockIdx.x/(64*16));
    const unsigned int t_id = (blockIdx.x - batch_id*64*16)/64;
    const unsigned int tile_h_id = (t_id / 4)*7;
    const unsigned int tile_w_id = (t_id % 4)*7;
    const unsigned int index = blockIdx.x % (64);
    const unsigned int start_channel_index = index*2;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[7*7] = {0.0f};
    conv_1_128_28_28_32_load_input_2_shared_memory(values,c_lens,ids,input,input_ids,channel_lens,warp_id,lane_id,batch_id,t_id,start_channel_index);
    __syncthreads();
    float v;
    unsigned int id;
    for(unsigned int n = threadIdx.x;n<32;n+=32){
        for(unsigned int c=start_channel_index;c<start_channel_index+2&&c<128;c++){
            unsigned int abs_c = c - start_channel_index;
            unsigned int start_index = abs_c*(7+3-1)*(7+3-1);
            unsigned int end_index = start_index+channel_lens[abs_c];
            if(start_index == end_index){
                continue;
            }
            conv_1_128_28_28_32_load_data_2_register(data_array,(c),kernel,n);
            unsigned int iters = end_index - start_index;
            for(unsigned int iter=0;iter<iters;iter++) {
                v = input[iter+start_index];
                id = input_ids[iter+start_index];
                conv_1_128_28_28_32_switch_function(id,data_array,v,temp_result);
            }
        }
        for (unsigned int th = 0; th < 7; ++th) {
            for (unsigned int tw = 0; tw < 7; ++tw) {
                if (tile_h_id + th >= 28 || tile_w_id + tw >= 28) {
                    continue;
                }
                atomicAdd(&outputs[batch_id * 32 * 28 * 28 + (tile_h_id + th) * 28 * 32 + (tile_w_id + tw) * 32 +
                                   n],temp_result[(th * 7 + tw)]);
            }
        }
        for(unsigned int i=0;i<7*7;++i){
            temp_result[i] = 0.0f;
        }
    }
}
void Conv_1_128_28_28_32::initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                                     unsigned int pad,unsigned int r,unsigned int s,unsigned int stride,string weightFile) {
    this->B = b;
    this->C = c;
    this->H = h;
    this->W = w;
    this->N = n;
    this->hOut = h;
    this->wOut = w;
    unsigned int kernelSize = 3*3*C*N;//kernel + bias
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    try{
        load_input(weightFile,kernelSize,cpuKernel);
    }catch (const char* msg) {
        cerr << msg << endl;
    }
    float *temp_kernel = (float *)malloc(kernelSize*sizeof(float));
    for(unsigned int i=0;i<N;++i){
        for(unsigned int l=0;l<C;++l){
            for(unsigned int j=0;j<3;++j){
                for(unsigned int k=0;k<3;++k){
                    temp_kernel[l*N*3*3+j*3*N+k*N+i] = cpuKernel[i*3*3*C+l*9+j*3+k];
                }
            }
        }
    }
    hipMalloc(&kernel,9*C*N*sizeof(float));
    hipMemcpy(kernel,temp_kernel,9*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(temp_kernel);
    free(cpuKernel);
    TILES_EACH_CHANNEL = ((H-1)/TH+1)*((W-1)/TW+1);
    TCS = (C-1)/TC + 1;
    hipMalloc(&nnz,b*c*TILES_EACH_CHANNEL*(TH+3-1)*(TW+3-1)*sizeof(float));
    hipMalloc(&ids,b*c*TILES_EACH_CHANNEL*(TH+3-1)*(TW+3-1)*sizeof(char));
    hipMalloc(&c_lens,b*c*TILES_EACH_CHANNEL*sizeof(unsigned int));
    hipMalloc(&output,b*n*h*w*sizeof(float));
}
float * Conv_1_128_28_28_32::forward(float *input) {
    hipMemset(output, 0, B*N*hOut*wOut*sizeof(float));
    hipMemset(c_lens, 0, B*C*TILES_EACH_CHANNEL*sizeof(unsigned int));
    conv_1_128_28_28_32_transform<<<(B*C*H*W-1)/512+1,512>>>(input,this->nnz,this->ids,this->c_lens);
    conv_1_128_28_28_32_conv2d<<<B*TCS*TILES_EACH_CHANNEL,N>>>(this->nnz,this->c_lens,this->ids,this->kernel,this->output);

    //chkerr(hipGetLastError());
    //chkerr(hipDeviceSynchronize());
    return output;
}
