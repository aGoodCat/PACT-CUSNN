#include "../inc/convBlk.h"
PoolBlk::PoolBlk(unsigned int b, unsigned int c, unsigned int h, unsigned int w, unsigned int n, string bn0Weight,
                 string conv1Weight) {
    pool_bn.initialize(b,c,h,w,bn0Weight);
    pool_relu.initialize(b,c,h,w);
    pool_conv.initialize(b,c,h,w,n,0,1,1,1,conv1Weight);
    pool.initialize(b,n,h,w,0,2,2,HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING,2);
}
float * PoolBlk::forward(float *input) {
    float * pool_bn_out = pool_bn.forward(input);
    float * pool_relu_out = pool_relu.forward(pool_bn_out);
    float * pool_conv_out = pool_conv.forward(pool_relu_out);
    float * pool_out = pool.forward(pool_conv_out);
    return pool_out;
}